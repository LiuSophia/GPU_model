#include "hip/hip_runtime.h"

#include "kernel.cuh"
#include <vector>
#include <stdio.h>
#include <stdlib.h>
using namespace std;



__device__ void findmin1_test(float *data, int num, float* min, int* index) {
	
	int tid = threadIdx.x;
	int blockid = blockIdx.x;
	int couts =0;//�@��block�ݭn��������(�q�Ҧ�data)
	int min_couts=0;//�@��block�ݭn��������(�qmin)

	couts=(num+blockDim.x-1)/blockDim.x;

	
	__shared__ int k;
	__shared__ int m;
	__shared__ int tem_index[NT/2];
	__shared__ float tem_descriptor[NT/2] ;
	k = 0;
	
		while (k<couts)//�P�_k<couts
		{
			if (num - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;
				m = 0;
				while (i != 0)
				{
					if (tid < i) {
						if (m == 0)//�Ĥ@������ishare memory
						{
							if (data[blockid * num + k*blockDim.x + tid] > data[blockid * num + k*blockDim.x + tid + i])
							{
								tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid + i];
								tem_index[tid] = k*blockDim.x + tid + i;
							}
							else
							{
								tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid];
								tem_index[tid] = k*blockDim.x + tid;
							}
							__syncthreads();
						}
						else//�ĤG���qshare memory ���Ȥ�
						{
							if (tem_descriptor[tid+ i ] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + i];
								tem_index[tid]= tem_index[tid + i];
							}
						}
					}
					__syncthreads();
					i /= 2;
					m++;
				}
			}
			else//�B�z�h�l
			{
				int remain = num - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				m = 0;
				while (j != 0)
				{
					if (tid < j) 
					{
						if (m == 0)//�Ĥ@������ishare memory
						{
							if (data[blockid * num + k*blockDim.x + tid] > data[blockid * num + k*blockDim.x + tid + j])
							{
								tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid + j];
								tem_index[tid] = k*blockDim.x + tid + j;
							}
							else
							{
								tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid];
								tem_index[tid] = k*blockDim.x + tid;
							}
							if (tid + j * 2 < tem_j) {
									if (data[blockid * num + k*blockDim.x + tid + j * 2] < tem_descriptor[tid])//�h�l���n��w�g��shared����
									{
										tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid + j * 2];
										tem_index[tid] = k*blockDim.x + remain - 1;
									}
								}
							__syncthreads();
						}
						else//�ĤG���qshare memory ���Ȥ�
						{
							if (tem_descriptor[tid+j] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + j];
								tem_index[tid] = tem_index[tid + j];
							}
							if (tid + j * 2 < tem_j) {
								if (tem_descriptor[tid + j * 2] < tem_descriptor[tid])
								{
									tem_descriptor[tid] = tem_descriptor[tid + j * 2];
									tem_index[tid] = tem_index[tid + j * 2];
								}
							}
						}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid ];
						tem_index[tid] = k*blockDim.x + tid;
					}
				}
			}
			if(tid==0)
			{
				min[blockid * couts + k] = tem_descriptor[0];
				index[blockid * couts + k] = tem_index[0];
			}
			__syncthreads();
			k++;
		}

		min_couts=(couts+blockDim.x-1)/blockDim.x;
		k=0;
		
		while(k<min_couts)
		{
			if(couts - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;
				m = 0;
				while (i != 0)
				{
					if (tid < i) {
						if(m==0)
						{
							if(min[blockid*couts +k*blockDim.x+tid]<min[blockid*couts +k*blockDim.x+tid+i])
							{
								tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid];
								tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
							}
							else
							{
								tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid+i];
								tem_index[tid] = index[blockid*couts + k*blockDim.x + tid+i];
							}
						__syncthreads();
						}
						else
						{
							if (tem_descriptor[tid+ i ] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + i];
								tem_index[tid]= tem_index[tid + i];
							}
						}
						
					}
					__syncthreads();
					i /= 2;
					m++;
				}
			}
			else
			{
				int remain = couts - blockDim.x*k;//7
				int tem_j = remain;
				int j = remain / 2;//3
				m = 0;
				while (j != 0)
				{
					if (tid < j) 
					{
						if(m==0)
						{
							if(min[blockid*couts +k*blockDim.x+tid]<min[blockid*couts +k*blockDim.x+tid+j])
							{
								tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid];
								tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
							}
							else
							{
								tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid+j];
								tem_index[tid] = index[blockid*couts + k*blockDim.x + tid+j];
							}
							if (tid + j * 2 < tem_j) {
								if (min[blockid*couts +k*blockDim.x+tid + j * 2] < tem_descriptor[tid])
								{
									tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid + j * 2];
									tem_index[tid] = index[blockid*couts + k*blockDim.x + tid + j * 2];
								}
							}
						__syncthreads();
						}
						else
						{
							if (tem_descriptor[tid+j] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + j];
								tem_index[tid] = tem_index[tid + j];
							}
							if (tid + j * 2 < tem_j) {
								if (tem_descriptor[tid + j * 2] < tem_descriptor[tid])
								{
									tem_descriptor[tid] = tem_descriptor[tid + j * 2];
									tem_index[tid] = tem_index[tid + j * 2];
								}
							}
						}
							
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						tem_descriptor[tid] = min[blockid * couts + k*blockDim.x + tid ];
						tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
					}
				}
				
			}
			
			if(tid==0)
			{
				min[blockid * couts + k] = tem_descriptor[0];
				index[blockid * couts + k] = tem_index[0];
			}
			k++;

		}
		/* if (tid < couts)
		{
			int i = couts / 2;
			int tem_i = couts;
			while (i != 0)
			{
				if (tid < i)
				{
					if (min[blockid*couts + tid] > min[blockid*couts + tid + i])
					{
						min[blockid*couts + tid] = min[blockid*couts + tid + i];
						index[blockid*couts + tid] = index[blockid*couts + tid + i];
					}
					if (tid + i * 2 < tem_i) {
						if (min[blockid*couts + tid] >min[blockid*couts + tid + 2 * i])
						{
							min[blockid*couts + tid] = min[blockid*couts + tid + 2 * i];
							index[blockid*couts + tid] = index[blockid*couts + tid + 2 * i];
						}
					}//�p�G�h�l���O�_��
				}
				__syncthreads();
				i /= 2;
				tem_i /= 2;
			}
		} */
}

__device__ void findmin_global(float *data, int num, float* min, int* index,int* tem_index)
{
	int tid = threadIdx.x;
	int blockid = blockIdx.x;
	int couts =0;//�@��block�ݭn��������(�q�Ҧ�data)
	int min_couts=0;//�@��block�ݭn��������(�qmin)
	couts=(num+blockDim.x-1)/blockDim.x;
	int k=0,m=0;
	//__shared__ int tem_index[NT/2];//global
	
		while (k<couts)//�P�_k<couts
		{
			if (num - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;
				m=0;
				while (i != 0)
				{
					if (tid < i) {
						if (data[blockid * num + k*blockDim.x + tid] > data[blockid * num + k*blockDim.x + tid + i])
						{
							data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid + i];
							if(m==0)	tem_index[blockid*NT/2+tid] = k*blockDim.x + tid + i;
							else	tem_index[blockid*NT/2+tid] = tem_index[blockid*NT/2+tid+i];
						}
						else
						{
							if(m==0)	tem_index[blockid*NT/2+tid] = k*blockDim.x + tid;
						}	
					}
					__syncthreads();
					i /= 2;
					m++;
				}
			}
			else//�B�z�h�l
			{
				int remain = num - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				m=0;
				while (j != 0)
				{
					if (tid < j) 
					{
						
						if (data[blockid * num + k*blockDim.x + tid] > data[blockid * num + k*blockDim.x + tid + j])
						{
							data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid + j];
							/* if(m==0)	tem_index[tid] = k*blockDim.x + tid + j;
							else	tem_index[tid] = tem_index[tid+j]; */
							if(m==0)	tem_index[blockid*NT/2+tid] = k*blockDim.x + tid + j;
							else	tem_index[blockid*NT/2+tid] = tem_index[blockid*NT/2+tid+j];
						}
						else
						{
							if(m==0)	tem_index[blockid*NT/2+tid] = k*blockDim.x + tid;
						}
						if (tid + j * 2 < tem_j) {
							if (data[blockid * num + k*blockDim.x + tid + j * 2] < data[blockid * num + k*blockDim.x + tid])
							{
								data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid + j * 2];
								/* if(m==0)	tem_index[tid] = k*blockDim.x + remain - 1;
								else	tem_index[tid] = tem_index[tid+j*2]; */
								if(m==0)	tem_index[blockid*NT/2+tid] = k*blockDim.x + remain - 1;
								else	tem_index[blockid*NT/2+tid] = tem_index[blockid*NT/2+tid+j*2];
							}
						}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid ];
						tem_index[blockid*NT/2+tid] = k*blockDim.x + tid;
					}
				}
			}
			if(tid==0)
			{
				min[blockid * couts + k] = data[blockid * num + k*blockDim.x];
				index[blockid * couts + k] = tem_index[blockid*NT/2];
			}
			
			k++;
		}
		
		min_couts=(couts+blockDim.x-1)/blockDim.x;//3
		k=0;
		
		while(k<min_couts)
		{
			if(couts - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;

				while (i != 0)
				{
					if (tid < i) {
							if (min[blockid*couts +k*blockDim.x+tid+ i ] < min[blockid*couts +k*blockDim.x+tid])
							{
								min[blockid*couts +k*blockDim.x+tid] = min[blockid*couts +k*blockDim.x+tid+ i ];
								index[blockid*couts +k*blockDim.x+tid]= index[blockid*couts +k*blockDim.x+tid + i];
							}
					}
					__syncthreads();
					i /= 2;
					//m++;
				}
			}
			else
			{
				int remain = couts - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				
				while (j != 0)
				{
					if (tid < j) 
					{
							if (min[blockid*couts +k*blockDim.x+tid+j] < min[blockid*couts +k*blockDim.x+tid])
							{
								min[blockid*couts +k*blockDim.x+tid] = min[blockid*couts +k*blockDim.x+tid + j];
								index[blockid*couts +k*blockDim.x+tid] = index[blockid*couts +k*blockDim.x+tid + j];
							}
							if (tid + j * 2 < tem_j) {
								if (min[blockid*couts +k*blockDim.x+tid + j * 2] < min[blockid*couts +k*blockDim.x+tid])
								{
									min[blockid*couts +k*blockDim.x+tid] = min[blockid*couts +k*blockDim.x+tid + j * 2];
									index[blockid*couts +k*blockDim.x+tid] = index[blockid*couts +k*blockDim.x+tid + j * 2];
								}
							}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					//m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						min[blockid*couts +k*blockDim.x+tid] = min[blockid * couts + k*blockDim.x + tid ];
						index[blockid*couts +k*blockDim.x+tid] = index[blockid*couts + k*blockDim.x + tid];
					}
				}
				
			}
			
			if(tid==0)
			{
				min[blockid * couts + k] = min[blockid*couts +k*blockDim.x];
				index[blockid * couts + k] = index[blockid*couts +k*blockDim.x];
			}
			k++;

		}
		
}

__device__ void findmin1(float *data, int num, float* min, int* index) {
	
	int tid = threadIdx.x;
	int blockid = blockIdx.x;
	int couts =0;//�@��block�ݭn��������(�q�Ҧ�data)
	int min_couts=0;//�@��block�ݭn��������(�qmin)
	couts=(num+blockDim.x-1)/blockDim.x;

	
	__shared__ int k;//block���F�ĴX��
	//__shared__ int m;
	__shared__ int tem_index[NT];
	__shared__ float tem_descriptor[NT] ;
	
	k = 0; //m = 0;
	
		while (k<couts)//�P�_k<couts
		{
			if (num - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;
				//m = 0;
				//if (m == 0)//�Ĥ@������ishare memory
				//{
					tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid];
					tem_index[tid] = k*blockDim.x + tid;
					__syncthreads();
				//}
				
				while (i != 0)
				{
					if (tid < i) {
							if (tem_descriptor[tid+ i ] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + i];
								tem_index[tid]= tem_index[tid + i];
							}
					}
					__syncthreads();
					i /= 2;
					//m++;
				}
			}
			else//�B�z�h�l
			{
				int remain = num - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				//m = 0;
				if (/*m == 0 && */tid<remain)//�Ĥ@������ishare memory
				{
					tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid];
					tem_index[tid] = k*blockDim.x + tid;
					__syncthreads();
				}
				
				while (j != 0)
				{
					if (tid < j) 
					{
							if (tem_descriptor[tid+j] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + j];
								tem_index[tid] = tem_index[tid + j];
							}
							if (tid + j * 2 < tem_j) {
								if (tem_descriptor[tid + j * 2] < tem_descriptor[tid])
								{
									tem_descriptor[tid] = tem_descriptor[tid + j * 2];
									tem_index[tid] = tem_index[tid + j * 2];
								}
							}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					//m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid ];
						tem_index[tid] = k*blockDim.x + tid;
					}
				}
			}
			
			if(tid==0)
			{
				min[blockid * couts + k] = tem_descriptor[0];
				index[blockid * couts + k] = tem_index[0];
			}
			
			k++;
		}

		min_couts=(couts+blockDim.x-1)/blockDim.x;//3
		k=0;
		
		while(k<min_couts)
		{
			if(couts - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;
				//m = 0;
				//if(m==0)
				//{
					tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid];
					tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
					__syncthreads();
				//}
				while (i != 0)
				{
					if (tid < i) {
							if (tem_descriptor[tid+ i ] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + i];
								tem_index[tid]= tem_index[tid + i];
							}
					}
					__syncthreads();
					i /= 2;
					//m++;
				}
			}
			else
			{
				int remain = couts - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				//m = 0;
				if (/*m == 0 &&*/ tid<remain)//�Ĥ@������ishare memory
				{
					tem_descriptor[tid] = min[blockid*couts + k*blockDim.x + tid];
					tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
					__syncthreads();
				}
				
				while (j != 0)
				{
					if (tid < j) 
					{
							if (tem_descriptor[tid+j] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + j];
								tem_index[tid] = tem_index[tid + j];
							}
							if (tid + j * 2 < tem_j) {
								if (tem_descriptor[tid + j * 2] < tem_descriptor[tid])
								{
									tem_descriptor[tid] = tem_descriptor[tid + j * 2];
									tem_index[tid] = tem_index[tid + j * 2];
								}
							}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					//m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						tem_descriptor[tid] = min[blockid * couts + k*blockDim.x + tid ];
						tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
					}
				}
				
			}
			
			if(tid==0)
			{
				min[blockid * couts + k] = tem_descriptor[0];
				index[blockid * couts + k] = tem_index[0];
			}
			k++;

		}
		

}

__global__ void matching(float *d_map, int *l_map, float *d_img, int *l_img, float *result, int img_num,int map_num,int descriptorDim,float *min, int *index,float* gpu_data_test,int *tem_index)
{
	int tid = threadIdx.x;
	int temp_tid = 0;
	float SAD = 0.0;
	__shared__ float sum[NT] ;
	
		for (int j = 0; j < img_num; j++)
		{
			temp_tid=tid;
			SAD = 0.0;
			if (l_map[blockIdx.x] == l_img[j])
			{
				 for (int i = 0; i < descriptorDim/NT; i++)
				{ 
				//SAD = (d_map[blockIdx.x *descriptorDim+ tid] - d_img[j*descriptorDim + tid])*(d_map[blockIdx.x *descriptorDim+ tid] - d_img[j*descriptorDim + tid]);
					 SAD += (d_map[blockIdx.x *descriptorDim+ temp_tid] - d_img[j*descriptorDim + temp_tid])*(d_map[blockIdx.x *descriptorDim+ temp_tid] - d_img[j*descriptorDim + temp_tid]);
					temp_tid += blockDim.x;
				} 
				sum[tid] = SAD;
				__syncthreads();

				int i = blockDim.x / 2;
				while (i != 0)
				{
					if (tid < i) {
						sum[tid] += sum[tid + i];
					}
					__syncthreads();
					i /= 2;
				}
				
				if(tid==0)
					sum[0] = sqrt(sum[0]);
				__syncthreads();
				
				result[img_num*blockIdx.x + j] = sum[0];
			}
			else
			{
				result[img_num*blockIdx.x + j] = 10.0;
			}

		}

	//findmin1(result, img_num, min, index);//��̤p shared memory=threads��
	findmin_global(gpu_data_test, img_num, min, index,tem_index);//��̤p global memory
	//findmin1_test(result, img_num, min, index); //��̤p shared memory threads�Ƥ@�b
	
		

}


void launchKernel(int map_num,int img_num,int descriptorDim,std::vector<int>&gpu_matcher,std::vector<feature>&feature_map,std::vector<feature>&feature_img,float* gpu_result,float* gpu_min,int* gpu_index,float* gpu_data_test)
{
		
	int blocknum = map_num;
	int thredperblock = NT;
	int couts = 0;
		
	couts=(img_num+thredperblock-1)/thredperblock;
    
    
    float *d_map, *d_img ;
	int  *l_map, *l_img;
	/////////////////// allocate memory on the cpu side//////////////////////
	d_map = (float*)malloc(map_num *descriptorDim* sizeof(float));
	l_map = (int*)malloc(map_num * sizeof(int));
	d_img = (float*)malloc(img_num *descriptorDim* sizeof(float));
	l_img = (int*)malloc(img_num * sizeof(int));
    ////////////////// put data in the host memory////////////////////////
    for (int i = 0; i < map_num; i++)
    {
        l_map[i]=feature_map[i].laplacian;
        for(int j=0;j<descriptorDim;j++)
		    d_map[i*descriptorDim+j]=feature_map[i].descriptor[j];
    }
        
	for (int i = 0; i < img_num; i++)
    {
        l_img[i]=feature_img[i].laplacian;
        for(int j=0;j<descriptorDim;j++)
		    d_img[i*descriptorDim+j]=feature_img[i].descriptor[j];
    }	
	////////////////// put data in the host memory////////////////////////
    
	float *dev_d_map, *dev_d_img;
	int *dev_l_map, *dev_l_img;
	float *dev_result;
	float *dev_min;
	int  *dev_index;
	float *dev_gpu_data_test;
	int *global_tem_index,*dev_tem_index;
	global_tem_index=(int*)malloc(map_num*NT/2* sizeof(int));
	////////////////// allocate the memory on the GPU ///////////////////////
	
	hipMalloc((void **)&dev_d_map, map_num *descriptorDim* sizeof(float));
	hipMalloc((void **)&dev_l_map, map_num * sizeof(int));
	hipMalloc((void **)&dev_d_img, img_num * descriptorDim*sizeof(float));
	hipMalloc((void **)&dev_l_img, img_num * sizeof(int));
	hipMalloc((void **)&dev_result, img_num*map_num * sizeof(float));
	//hipMalloc((void **)&dev_match, 2 * map_num * sizeof(int));
	hipMalloc((void **)&dev_min, couts*map_num * sizeof(float));
	hipMalloc((void **)&dev_index, couts * map_num * sizeof(int));
	hipMalloc((void **)&dev_gpu_data_test,map_num*img_num* sizeof(float));
	hipMalloc((void **)&dev_tem_index,map_num*NT/2* sizeof(int));
	////////////////// copy the arrays 's' and 'p' to the GPU/////////////////////////
	hipMemcpy(dev_d_map, d_map, map_num * descriptorDim*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_l_map, l_map, map_num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_d_img, d_img, img_num *descriptorDim* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_l_img, l_img, img_num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_result, gpu_result, img_num*map_num * sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(dev_match, match, 2 * map_num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_min, gpu_min, couts*map_num * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_index, gpu_index, couts*map_num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_gpu_data_test,gpu_data_test,map_num*img_num* sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_tem_index,global_tem_index,map_num*NT/2* sizeof(int),hipMemcpyHostToDevice);
	// Get start time event
	hipEvent_t start_kernel, stop_kernel;
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
	hipEventRecord(start_kernel, 0);

	matching << < blocknum, thredperblock >> > (dev_d_map, dev_l_map, dev_d_img, dev_l_img, dev_result, img_num, map_num,descriptorDim,dev_min, dev_index,dev_gpu_data_test,dev_tem_index);

	
	// Get stop time event
	hipEventRecord(stop_kernel, 0);
	hipEventSynchronize(stop_kernel);
	// Compute execution time
	float kernelTime;
	hipEventElapsedTime(&kernelTime, start_kernel, stop_kernel);
	printf("GPU time: %13f msec\n", kernelTime);
	hipEventDestroy(start_kernel);
	hipEventDestroy(stop_kernel);
	

	hipError_t cuda_err = hipGetLastError();
	if (hipSuccess != cuda_err) {
		cout << "before kernel call: error = %s\n" << hipGetErrorString(cuda_err) << endl;
		system("pause");
		exit(1);
	}

	hipEvent_t start_tra, stop_tra;
	hipEventCreate(&start_tra);
	hipEventCreate(&stop_tra);
	hipEventRecord(start_tra, 0);
	
	hipMemcpy(gpu_result, dev_result, img_num*map_num * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(gpu_min, dev_min, couts*map_num * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(gpu_index, dev_index, couts*map_num *sizeof(int), hipMemcpyDeviceToHost);
	
	hipEventRecord(stop_tra, 0);
	hipEventSynchronize(stop_tra);

	// Compute execution time
	float transferTime;
	hipEventElapsedTime(&transferTime, start_tra, stop_tra);
	printf("GPU transfer time: %13f msec\n", transferTime);
	hipEventDestroy(start_tra);
	hipEventDestroy(stop_tra);
	
	
		
	int min_couts=0;
	float tem_min=0;
	int tem_index=0;
	
	min_couts=(couts+thredperblock-1)/thredperblock;
	/* for(int j=0;j<map_num;j++)
		for(int i=0;i<min_couts;i++)
			cout<<gpu_min[j*couts+i]<<endl; */
	/* for(int j=0;j<12;j++)
			cout<<gpu_index[j]<<endl;
		cout<<endl; */
	/* for(int i=0;i<img_num;i++)
		cout<<gpu_data_test[(map_num-1)*img_num+i]<<endl;
	cout<<endl; */
	/* for(int j=0;j<couts;j++)
			cout<<gpu_min[couts*(map_num-2)+j]<<endl;
		cout<<endl; */
	
	/* for(int j=0;j<couts*map_num;j++)
		cout<<gpu_min[j]<<endl; */
		
	struct timeval startCPU, endCPU;
    gettimeofday(&startCPU, NULL);
	
	for(int i=0;i<map_num;i++)//////�^�Ǥ�̤p
	{
		tem_min=gpu_min[i*couts];tem_index=gpu_index[i*couts];
		for(int j=1;j<min_couts;j++)
		{
			if(tem_min>gpu_min[i*couts+j])
			{
				tem_min=gpu_min[i*couts+j];
				tem_index=gpu_index[i*couts+j];
			}	
		}
		gpu_min[i*couts]=tem_min;
		gpu_index[i*couts]=tem_index;
		if(gpu_min[i*couts]<0.09)//�p����e�ȧ�X���쪺
		{
			gpu_matcher.push_back(i);
			gpu_matcher.push_back(gpu_index[i*couts]);
		}
	}
	
	
		
	gettimeofday(&endCPU, NULL);
    double executime;
    executime = (endCPU.tv_sec - startCPU.tv_sec) * 1000.0;
    executime += (endCPU.tv_usec - startCPU.tv_usec) / 1000.0;
    printf("GPU_CPU find part min time: %13lf msec\n", executime);
	printf("All time: %13lf msec\n", executime+kernelTime+transferTime);
	/////////////////////////////////���զۤv�гy���Ⱥ�X�����G///////////////////////////////////////
	#ifdef GPU_Debug_Create
	int block_error=0,index_error=0;
	for(int i=0;i<map_num;i++)
	{
		 if(fabs(gpu_min[i*couts]-0.001)>0.00001)
		{
			//cout<<i<<setw(10)<<gpu_min[i*couts]<<endl;
			block_error++;
		} 
		//cout<<i<<setw(10)<<gpu_min[i]<<endl;
		//cout<<min[couts*i]<<endl;
		//cout<<index[i]<<endl;
	}
	for(int j=0;j<map_num;j++)
		if(gpu_index[j*couts]!=743)
		{
			//cout<<j<<setw(10)<<gpu_index[j*couts]<<endl;
			index_error++;
		}
			
	cout<<	"block_error"<<block_error<<endl;
	cout<<	"index_error"<<index_error<<endl;
	#endif
	/////////////////////////////////���զۤv�гy���Ⱥ�X�����G///////////////////////////////////////
    
	// Free device memory
	hipFree(dev_d_map);
	hipFree(dev_l_map);
	hipFree(dev_d_img);
	hipFree(dev_l_img);
	hipFree(dev_result);
	hipFree(dev_min);
	hipFree(dev_index);
	
	//return min;
	
}


