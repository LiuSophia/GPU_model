#include "hip/hip_runtime.h"

#include "kernelP3P.cuh"
//#include "P3P.h
#include <vector>
#include <stdio.h>
#include <stdlib.h>
using namespace std;



__global__ void getCameraPose_dub(int *index, double *WL_all, double *Rs_all, double *CameraPose)
{
	int tid=threadIdx.x;
	int pose_index=index[0];
	CameraPose[tid]=WL_all[pose_index*3];
	CameraPose[tid+1]=WL_all[pose_index*3+1];
	CameraPose[tid+2]=WL_all[pose_index*3+2];
	CameraPose[tid+3]=Rs_all[pose_index*9];
	CameraPose[tid+4]=Rs_all[pose_index*9+1];
	CameraPose[tid+5]=Rs_all[pose_index*9+2];
	CameraPose[tid+6]=Rs_all[pose_index*9+3];
	CameraPose[tid+7]=Rs_all[pose_index*9+4];
	CameraPose[tid+8]=Rs_all[pose_index*9+5];
	CameraPose[tid+9]=Rs_all[pose_index*9+6];
	CameraPose[tid+10]=Rs_all[pose_index*9+7];
	CameraPose[tid+11]=Rs_all[pose_index*9+8];
}


__global__ void getCameraPose(int *index, float *WL_all, float *Rs_all, float *CameraPose)
{
	int tid=threadIdx.x;
	int pose_index=index[0];
	CameraPose[tid]=WL_all[pose_index*3];
	CameraPose[tid+1]=WL_all[pose_index*3+1];
	CameraPose[tid+2]=WL_all[pose_index*3+2];
	CameraPose[tid+3]=Rs_all[pose_index*9];
	CameraPose[tid+4]=Rs_all[pose_index*9+1];
	CameraPose[tid+5]=Rs_all[pose_index*9+2];
	CameraPose[tid+6]=Rs_all[pose_index*9+3];
	CameraPose[tid+7]=Rs_all[pose_index*9+4];
	CameraPose[tid+8]=Rs_all[pose_index*9+5];
	CameraPose[tid+9]=Rs_all[pose_index*9+6];
	CameraPose[tid+10]=Rs_all[pose_index*9+7];
	CameraPose[tid+11]=Rs_all[pose_index*9+8];
}


__global__ void findmax_global_dub(int *data, double* para, int *ransac_match_max,int *index, int *index_max)
{
	int sol_num=para[4];
	int tid = threadIdx.x;
	int blockid=blockIdx.x;
	if(blockid*blockDim.x+tid<sol_num)
	{
	
		if(sol_num - blockDim.x*blockid >= blockDim.x)
		{
			int i = blockDim.x / 2;
			
						
			while (i != 0)
			{
				if (tid < i) {
						if (data[blockid*blockDim.x+tid+ i ] > data[blockid*blockDim.x+tid])
						{
							data[blockid*blockDim.x+tid] = data[blockid*blockDim.x+tid+ i ];
							index[blockid*blockDim.x+tid]= index[blockid*blockDim.x+tid+i];
						}
				}
				__syncthreads();
				i /= 2;
				//m++;
				}
		}
		else
		{
			int remain = sol_num - blockDim.x*blockid;
			int tem_j = remain;
			int j = remain / 2;
			
						
			while (j != 0)
			{
				if (tid < j) 
				{
						if (data[blockid*blockDim.x+tid+j] > data[blockid*blockDim.x+tid])
						{
							data[blockid*blockDim.x+tid] = data[blockid*blockDim.x+tid + j];
							index[blockid*blockDim.x+tid] = index[blockid*blockDim.x+tid+j];
						}
						if (tid + j * 2 < tem_j) {
							if (data[blockid*blockDim.x+tid + j * 2] > data[blockid*blockDim.x+tid])
							{
								data[blockid*blockDim.x+tid] = data[blockid*blockDim.x+tid + j * 2];
								index[blockid*blockDim.x+tid] = index[blockid*blockDim.x+tid + j * 2];
							}
						}
				}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					//m++;
			}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						data[blockid*blockDim.x+tid] = data[blockid*blockDim.x+tid];
						index[blockid*blockDim.x+tid] = index[blockid*blockDim.x+tid];
					}
				}
				
		}

		if(tid==0)
		{
			ransac_match_max[blockid] = data[blockid*blockDim.x];
			index_max[blockid] = index[blockid*blockDim.x];
		}
	}
	
	
}


__global__ void findmax_global(int *data, float* para, int *ransac_match_max,int *index, int *index_max)
{
	int sol_num=para[4];
	int tid = threadIdx.x;
	int blockid=blockIdx.x;
	if(blockid*blockDim.x+tid<sol_num)
	{
	
		if(sol_num - blockDim.x*blockid >= blockDim.x)
		{
			int i = blockDim.x / 2;
			
						
			while (i != 0)
			{
				if (tid < i) {
						if (data[blockid*blockDim.x+tid+ i ] > data[blockid*blockDim.x+tid])
						{
							data[blockid*blockDim.x+tid] = data[blockid*blockDim.x+tid+ i ];
							index[blockid*blockDim.x+tid]= index[blockid*blockDim.x+tid+i];
						}
				}
				__syncthreads();
				i /= 2;
				//m++;
				}
		}
		else
		{
			int remain = sol_num - blockDim.x*blockid;
			int tem_j = remain;
			int j = remain / 2;
			
						
			while (j != 0)
			{
				if (tid < j) 
				{
						if (data[blockid*blockDim.x+tid+j] > data[blockid*blockDim.x+tid])
						{
							data[blockid*blockDim.x+tid] = data[blockid*blockDim.x+tid + j];
							index[blockid*blockDim.x+tid] = index[blockid*blockDim.x+tid+j];
						}
						if (tid + j * 2 < tem_j) {
							if (data[blockid*blockDim.x+tid + j * 2] > data[blockid*blockDim.x+tid])
							{
								data[blockid*blockDim.x+tid] = data[blockid*blockDim.x+tid + j * 2];
								index[blockid*blockDim.x+tid] = index[blockid*blockDim.x+tid + j * 2];
							}
						}
				}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					//m++;
			}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						data[blockid*blockDim.x+tid] = data[blockid*blockDim.x+tid];
						index[blockid*blockDim.x+tid] = index[blockid*blockDim.x+tid];
					}
				}
				
		}

		if(tid==0)
		{
			ransac_match_max[blockid] = data[blockid*blockDim.x];
			index_max[blockid] = index[blockid*blockDim.x];
		}
	}
	
	
}



__device__ void Ransac(float *mapft_info, float *WL ,float *Rs,float *parm, int *ransac_match_num,int *index_num)
{
	int tid =threadIdx.x;
	int sol_num=parm[4];//u0,vo,fu,fv
	int map_num=parm[5];
	float range=25.0;
	if(blockIdx.x*blockDim.x+tid<sol_num)
	{
		float rx = WL[blockIdx.x*3*blockDim.x+tid*3];
		float ry = WL[blockIdx.x*3*blockDim.x+tid*3+1];
		float rz = WL[blockIdx.x*3*blockDim.x+tid*3+2];
		
		int setsize=0;
		float dist=0;
		for(int i=0;i<map_num;i++)
		{
			float feature[3], h[3];//, h2[3];
			feature[0] = (mapft_info[i*6+3] - parm[0]) / parm[2] * mapft_info[i*6+5];
			feature[1] = (mapft_info[i*6+4] - parm[1]) / parm[3] * mapft_info[i*6+5];
			feature[2] = mapft_info[i*6+5];
			
			h[0] = Rs[blockIdx.x*9*blockDim.x+tid*9] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+3] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+6] * feature[2];
			h[1] = Rs[blockIdx.x*9*blockDim.x+tid*9+1] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+4] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+7] * feature[2];
			h[2] = Rs[blockIdx.x*9*blockDim.x+tid*9+2] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+5] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+8] * feature[2];
			
			dist=0;
			dist = (h[0] - mapft_info[i*6] + rx)*(h[0] - mapft_info[i*6] + rx) +
					(h[1] - mapft_info[i*6+1] + ry)*(h[1] - mapft_info[i*6+1] + ry) +
					(h[2] - mapft_info[i*6+2] + rz)*(h[2] - mapft_info[i*6+2] + rz);
			dist = sqrt(dist);
			if (dist < range)
			{
				setsize++;//�ŦX���󪺯S�x�I�Ӽ�
			}

		}
		ransac_match_num[blockIdx.x*blockDim.x+tid]=setsize;
		index_num[blockIdx.x*blockDim.x+tid]=blockIdx.x*blockDim.x+tid;
	}
}

__device__ void Ransac_dub(double *mapft_info, double *WL ,double *Rs,double *parm, int *ransac_match_num,int *index_num)
{
	int tid =threadIdx.x;
	int sol_num=parm[4];//u0,vo,fu,fv
	int map_num=parm[5];
	double range=25.0;
	if(blockIdx.x*blockDim.x+tid<sol_num)
	{
		double rx = WL[blockIdx.x*3*blockDim.x+tid*3];
		double ry = WL[blockIdx.x*3*blockDim.x+tid*3+1];
		double rz = WL[blockIdx.x*3*blockDim.x+tid*3+2];
		
		int setsize=0;
		double dist=0;
		for(int i=0;i<map_num;i++)
		{
			double feature[3], h[3];//, h2[3];
			feature[0] = (mapft_info[i*6+3] - parm[0]) / parm[2] * mapft_info[i*6+5];
			feature[1] = (mapft_info[i*6+4] - parm[1]) / parm[3] * mapft_info[i*6+5];
			feature[2] = mapft_info[i*6+5];
			
			h[0] = Rs[blockIdx.x*9*blockDim.x+tid*9] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+3] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+6] * feature[2];
			h[1] = Rs[blockIdx.x*9*blockDim.x+tid*9+1] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+4] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+7] * feature[2];
			h[2] = Rs[blockIdx.x*9*blockDim.x+tid*9+2] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+5] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+8] * feature[2];
			
			dist=0;
			dist = (h[0] - mapft_info[i*6] + rx)*(h[0] - mapft_info[i*6] + rx) +
					(h[1] - mapft_info[i*6+1] + ry)*(h[1] - mapft_info[i*6+1] + ry) +
					(h[2] - mapft_info[i*6+2] + rz)*(h[2] - mapft_info[i*6+2] + rz);
			dist = sqrt(dist);
			if (dist < range)
			{
				setsize++;//�ŦX���󪺯S�x�I�Ӽ�
			}

		}
		ransac_match_num[blockIdx.x*blockDim.x+tid]=setsize;
		index_num[blockIdx.x*blockDim.x+tid]=blockIdx.x*blockDim.x+tid;
	}
}


__global__ void GPU_P3P(int *matft_NO, int *mapft_NO, float *matft_info, float *mapft_info, int *P3P_NO, float *parm, float *WL ,float *Rs,int *ransac_match_num, int *ransac_match_max, int *index_num)
{
	int tid =threadIdx.x;
	//int blockIdx=blockIdx.x;
	int sol_num=parm[4];//u0,vo,fu,fv
	
	if(blockIdx.x*blockDim.x+tid<sol_num)
	{
	
	float ptw[9],pti[6],dep[3];//�C��thread���ۤv��
	int p1,p2,p3;
	
	p1=P3P_NO[blockIdx.x*3*blockDim.x+tid*3];//�C��thread���T�ӽs��
	p2=P3P_NO[blockIdx.x*3*blockDim.x+tid*3+1];
	p3=P3P_NO[blockIdx.x*3*blockDim.x+tid*3+2];
	
	ptw[0]=matft_info[p1*6];//�Ĥ@���I
	ptw[1]=matft_info[p1*6+1];
	ptw[2]=matft_info[p1*6+2];
	pti[0]=matft_info[p1*6+3];
	pti[1]=matft_info[p1*6+4];
	dep[0]=matft_info[p1*6+5]*matft_info[p1*6+5];
	
	ptw[3]=matft_info[p2*6];//�ĤG���I
	ptw[4]=matft_info[p2*6+1];
	ptw[5]=matft_info[p2*6+2];
	pti[2]=matft_info[p2*6+3];
	pti[3]=matft_info[p2*6+4];
	dep[1]=matft_info[p2*6+5]*matft_info[p2*6+5];
	
	ptw[6]=matft_info[p3*6];//�ĤT���I
	ptw[7]=matft_info[p3*6+1];
	ptw[8]=matft_info[p3*6+2];
	pti[4]=matft_info[p3*6+3];
	pti[5]=matft_info[p3*6+4];
	dep[2]=matft_info[p3*6+5]*matft_info[p3*6+5]; 
	
	 float aa,bb,cc;
	aa= sqrt( ((pti[0] - parm[0])/parm[2])*((pti[0] - parm[0])/parm[2])*dep[0] + ((pti[1]-parm[1])/parm[3])*((pti[1]-parm[1])/parm[3])*dep[0] + dep[0]);
	bb = sqrt( ((pti[2] - parm[0])/parm[2])*((pti[2] - parm[0])/parm[2])*dep[1] + ((pti[3]-parm[1])/parm[3])*((pti[3]-parm[1])/parm[3])*dep[1] + dep[1]);
	cc = sqrt( ((pti[4] - parm[0])/parm[2])*((pti[4] - parm[0])/parm[2])*dep[2] + ((pti[5]-parm[1])/parm[3])*((pti[5]-parm[1])/parm[3])*dep[2] + dep[2]);
	 
	/* Rs[blockIdx.x*9*blockDim.x+tid*9]=ptw[0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+1]=ptw[1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+2]=ptw[2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+3]=ptw[3];
	Rs[blockIdx.x*9*blockDim.x+tid*9+4]=ptw[4];
	Rs[blockIdx.x*9*blockDim.x+tid*9+5]=ptw[5];
	Rs[blockIdx.x*9*blockDim.x+tid*9+6]=ptw[6];
	Rs[blockIdx.x*9*blockDim.x+tid*9+7]=ptw[7];
	Rs[blockIdx.x*9*blockDim.x+tid*9+8]=ptw[8];  */
	
	
	//////////////////////////////////////P3P�p��///////////////////////////////////
	
	///////////////////////////////����v����m//////////////////////////
	float VAB[3],VAC[3],VBC[3];
		for(int i=0;i<3;i++)
		{
			VAB[i]=ptw[i+3]-ptw[i];//2-1
			VAC[i]=ptw[i+6]-ptw[i];//3-1
			VBC[i]=ptw[i+6]-ptw[i+3];//3-1
		}
		
	//Length of edge between control points
	float Rab,Rac,Rbc;
	Rab=sqrt(VAB[0]*VAB[0]+VAB[1]*VAB[1]+VAB[2]*VAB[2]);
	Rac=sqrt(VAC[0]*VAC[0]+VAC[1]*VAC[1]+VAC[2]*VAC[2]);
	Rbc=sqrt(VBC[0]*VBC[0]+VBC[1]*VBC[1]+VBC[2]*VBC[2]);//�᭱�Τ���
	
	float CA[3],CB[3],CC[3];
		CA[0]=(pti[0]-parm[0])/parm[2];	CA[1]=(pti[1]-parm[1])/parm[3];	CA[2]=1;
		CB[0]=(pti[2]-parm[0])/parm[2];	CB[1]=(pti[3]-parm[1])/parm[3];	CB[2]=1;
		CC[0]=(pti[4]-parm[0])/parm[2];	CC[1]=(pti[5]-parm[1])/parm[3];	CC[2]=1;
	
	float RCA,RCB,RCC;
		RCA=sqrt(CA[0]*CA[0]+CA[1]*CA[1]+CA[2]*CA[2]);
		RCB=sqrt(CB[0]*CB[0]+CB[1]*CB[1]+CB[2]*CB[2]);
		RCC=sqrt(CC[0]*CC[0]+CC[1]*CC[1]+CC[2]*CC[2]);
	
	//Normalize
	CA[0]=CA[0]/RCA;	CA[1]=CA[1]/RCA;	CA[2]=CA[2]/RCA;
	CB[0]=CB[0]/RCB;	CB[1]=CB[1]/RCB;	CB[2]=CB[2]/RCB;
	CC[0]=CC[0]/RCC;	CC[1]=CC[1]/RCC;	CC[2]=CC[2]/RCC;
	
	float Rab1,Rac1,Rbc1;//�V�q���׭p�� unit mm
		Rab1=sqrt(CA[0]*CA[0]+CA[1]*CA[1]+CA[2]*CA[2]);
		Rac1=sqrt(CB[0]*CB[0]+CB[1]*CB[1]+CB[2]*CB[2]);
		Rbc1=sqrt(CC[0]*CC[0]+CC[1]*CC[1]+CC[2]*CC[2]);
	
	//Cosine of angles//�᭱�S�Ψ�
	float Calb, Calc, Cblc;
	//Compute Calb Calc Cblc using Law of Cosine
	 Calb = (2 - Rab1*Rab1) / 2;
	Calc = (2 - Rac1*Rac1) / 2;
	Cblc = (2 - Rbc1*Rbc1) / 2; 
	
	//Get cosine of the angles
	float Clab = (aa*aa + Rab*Rab - bb*bb) / (2 * aa*Rab);
	float Clac = (aa*aa + Rac*Rac - cc*cc) / (2 * aa*Rac);
	
	//Get scale along norm vector
	float Raq = aa*Clab;
	float Rap = aa*Clac;
	
	//Get norm vector of plane P1 P2
	float VAB_norm = Rab;
	float VAC_norm = Rac;
	
	float WQ[3],WP[3];
	for (int i = 0; i<3; i++)
	{
		WQ[i]=ptw[i]+ Raq*VAB[i] / VAB_norm;
		WP[i]=ptw[i]+ Rap*VAC[i] / VAC_norm;
	}
	
	//Compute Plane P1 P2 P3
	float NP1[3],NP2[3],NP3[3];
	for (int i = 0; i<3; i++)
	{
		NP1[i]=VAB[i] / VAB_norm;
		NP2[i]=VAC[i] / VAC_norm;
	}
	
	float DP1, DP2, DP3;
	DP1=NP1[0]*WQ[0]+NP1[1]*WQ[1]+NP1[2]*WQ[2];
	DP2=NP2[0]*WP[0]+NP2[1]*WP[1]+NP2[2]*WP[2];
	
	float P1[4],P2[4],P3[4];
	P1[0]=NP1[0];	P1[1]=NP1[1];	P1[2]=NP1[2];	P1[3]=-DP1;
	P2[0]=NP2[0];	P2[1]=NP2[1];	P2[2]=NP2[2];	P2[3]=-DP2;
	
	float VCX[3],VCY[3],VCZ[3];
	for (int i = 0; i<3; i++)
	{
		VCX[i]=CB[i]-CA[i];
		VCY[i]=CC[i]-CA[i];
	}
	
	//�p����u�V�q�~�nZ��V�����q�P�_�ϥΦ�رƦC(�t��AC X AB)
	if ( (VCX[0] * VCY[1] - VCY[0] * VCX[1]) > 0 )	
	{
		//cvCrossProduct(VAC, VAB, NP3);
		NP3[0]=VAC[1]*VAB[2]-VAC[2]*VAB[1];
		NP3[1]=VAC[2]*VAB[0]-VAC[0]*VAB[2];
		NP3[2]=VAC[0]*VAB[1]-VAC[1]*VAB[0];
	}
	else
	{
		//cvCrossProduct(VAB, VAC, NP3);
		NP3[0]=VAB[1]*VAC[2]-VAB[2]*VAC[1];
		NP3[1]=VAB[2]*VAC[0]-VAB[0]*VAC[2];
		NP3[2]=VAB[0]*VAC[1]-VAB[1]*VAC[0];
	}
	
	float NP3_norm=sqrt(NP3[0]*NP3[0]+NP3[1]*NP3[1]+NP3[2]*NP3[2]);
	//Normalize
	NP3[0]=NP3[0]/NP3_norm;	NP3[1]=NP3[1]/NP3_norm;	NP3[2]=NP3[2]/NP3_norm;
	
	DP3=NP3[0]*ptw[0]+NP3[1]*ptw[1]+NP3[2]*ptw[2];//DP3 = cvDotProduct(NP3, W_one);
	
	P3[0]=NP3[0];	P3[1]=NP3[1];	P3[2]=NP3[2];	P3[3]=-DP3;
	
	//�J�԰������D�T������ P1 P2 P3
	float delta, delta_x, delta_y, delta_z;
	delta=P1[0]*P2[1]*P3[2]+P2[0]*P3[1]*P1[2]+P3[0]*P1[1]*P2[2]-P3[0]*P2[1]*P1[2]-P1[0]*P3[1]*P2[2]-P2[0]*P1[1]*P3[2];
	delta_x=P2[1]*P3[2]*P1[3]+P1[1]*P2[2]*P3[3]+P3[1]*P2[3]*P1[2]-P2[1]*P1[2]*P3[3]-P1[1]*P2[3]*P3[2]-P3[1]*P1[3]*P2[2];
	delta_y=P1[0]*P3[2]*P2[3]+P2[0]*P1[2]*P3[3]+P3[0]*P2[2]*P1[3]-P3[0]*P1[2]*P2[3]-P1[0]*P2[2]*P3[3]-P2[0]*P3[2]*P1[3];
	delta_z=P1[0]*P2[1]*P3[3]+P2[0]*P3[1]*P1[3]+P3[0]*P1[1]*P2[3]-P3[0]*P2[1]*P1[3]-P1[0]*P3[1]*P2[3]-P2[0]*P1[1]*P3[3];
	
	float WR[3];
	if(delta!=0)
	{
		//�W���Y�Ʋ����A�]���t�@�ӭt��
		WR[0]=-delta_x/delta;	WR[1]=-delta_y/delta;	WR[2]=-delta_z/delta;
	}
	else
	{
		WR[0]=1000000;	WR[1]=1000000;	WR[2]=1000000;
	}
	
	
	//Get length of LR
	float Rar, Rlr;
	Rar=sqrt((ptw[0]-WR[0])*(ptw[0]-WR[0])+(ptw[1]-WR[1])*(ptw[1]-WR[1])+(ptw[2]-WR[2])*(ptw[2]-WR[2]));
	if(aa*aa - Rar*Rar>0)
		Rlr = sqrt(aa*aa - Rar*Rar);//��aa*aa - Rar*Rar<0�ɡARlr�h��nan
	else
		Rlr = 10e6;
	//Get Position of L in world frame
	//WL=WR+NP3*Rlr;
	float WL_test[3];
	WL_test[0]=WR[0]+NP3[0]*Rlr;
	WL_test[1]=WR[1]+NP3[1]*Rlr;
	WL_test[2]=WR[2]+NP3[2]*Rlr;
	
	WL[blockIdx.x*3*blockDim.x+tid*3]=WL_test[0];
	WL[blockIdx.x*3*blockDim.x+tid*3+1]=WL_test[1];
	WL[blockIdx.x*3*blockDim.x+tid*3+2]=WL_test[2];
	
	///////////////////////////////////////�ا��Шt�����x�}/////////////////////////////////////////////
	VCZ[0]=VCX[1]*VCY[2]-VCX[2]*VCY[1];	//cvCrossProduct(VCX, VCY, VCZ);
	VCZ[1]=VCX[2]*VCY[0]-VCX[0]*VCY[2];
	VCZ[2]=VCX[0]*VCY[1]-VCX[1]*VCY[0];
	
	VCY[0]=VCZ[1]*VCX[2]-VCZ[2]*VCX[1];	//cvCrossProduct(VCZ, VCX, VCY);
	VCY[1]=VCZ[2]*VCX[0]-VCZ[0]*VCX[2];
	VCY[2]=VCZ[0]*VCX[1]-VCZ[1]*VCX[0];
	
	//Normalize
	float VCX_norm,VCY_norm,VCZ_norm;
	VCX_norm=sqrt(VCX[0]*VCX[0]+VCX[1]*VCX[1]+VCX[2]*VCX[2]);
	VCY_norm=sqrt(VCY[0]*VCY[0]+VCY[1]*VCY[1]+VCY[2]*VCY[2]);
	VCZ_norm=sqrt(VCZ[0]*VCZ[0]+VCZ[1]*VCZ[1]+VCZ[2]*VCZ[2]);
	for(int i=0;i<3;i++)
	{
		VCX[i]=VCX[i]/VCX_norm;	VCY[i]=VCY[i]/VCY_norm;	VCZ[i]=VCZ[i]/VCZ_norm;
	}
	
	//%Get ray in the world frame
	float Vla[3],Vlb[3],Vlc[3];
	for(int i=0;i<3;i++)
	{
		Vla[i]=ptw[i]-WL_test[i];	//Vla=WA-WL;
		Vlb[i]=ptw[i+3]-WL_test[i];	//Vlb=WB-WL;
		Vlc[i]=ptw[i+6]-WL_test[i];	//Vlc=WC-WL;
	}
	
	//Normalize
	float Vla_norm,Vlb_norm,Vlc_norm;
	Vla_norm=sqrt(Vla[0]*Vla[0]+Vla[1]*Vla[1]+Vla[2]*Vla[2]);
	Vlb_norm=sqrt(Vlb[0]*Vlb[0]+Vlb[1]*Vlb[1]+Vlb[2]*Vlb[2]);
	Vlc_norm=sqrt(Vlc[0]*Vlc[0]+Vlc[1]*Vlc[1]+Vlc[2]*Vlc[2]);
	for(int i=0;i<3;i++)
	{
		Vla[i]=Vla[i]/Vla_norm;	Vlb[i]=Vlb[i]/Vlb_norm;	Vlc[i]=Vlc[i]/Vlc_norm;
	}
	
	float WA1[3],WB1[3],WC1[3];
	for(int i=0;i<3;i++)
	{
		WA1[i]=WL_test[i]+Vla[i];	//WA1=WL+1*Vla;
		WB1[i]=WL_test[i]+Vlb[i];	//WB1=WL+1*Vlb;
		WC1[i]=WL_test[i]+Vlc[i];	//WC1=WL+1*Vlc;
	}
	
	float vcx[3],vcy[3],vcz[3];
	for(int i=0;i<3;i++)
	{
		vcx[i]=WB1[i]-WA1[i];	//vcx=WB1-WA1;
		vcy[i]=WC1[i]-WA1[i];	//vcy=WC1-WA1;
	}
	
	
	vcz[0]=vcx[1]*vcy[2]-vcx[2]*vcy[1];	//cvCrossProduct(vcx, vcy, vcz);
	vcz[1]=vcx[2]*vcy[0]-vcx[0]*vcy[2];
	vcz[2]=vcx[0]*vcy[1]-vcx[1]*vcy[0];
	
	vcy[0]=vcz[1]*vcx[2]-vcz[2]*vcx[1];	//cvCrossProduct(vcz, vcx, vcy);
	vcy[1]=vcz[2]*vcx[0]-vcz[0]*vcx[2];
	vcy[2]=vcz[0]*vcx[1]-vcz[1]*vcx[0];
	
	
	
	//Normalize
	//float vcx_norm_test,vcy_norm_test,vcz_norm_test;
	float vcx_norm,vcy_norm,vcz_norm;
	
	/* vcx_norm_test=vcx[0]*vcx[0]+vcx[1]*vcx[1]+vcx[2]*vcx[2];
	vcy_norm_test=vcy[0]*vcy[0]+vcy[1]*vcy[1]+vcy[2]*vcy[2];
	vcz_norm_test=vcz[0]*vcz[0]+vcz[1]*vcz[1]+vcz[2]*vcz[2];
	
	if(vcx_norm_test==0)
		vcx_norm=sqrt(10e-12);
	else	vcx_norm=sqrt(vcx_norm_test);
	
	if(vcy_norm_test==0)
		vcy_norm_test=sqrt(10e-12);
	else	vcy_norm=sqrt(vcy_norm_test);
	
	if(vcz_norm_test==0)
		vcz_norm_test=sqrt(10e-12);
	else	vcz_norm=sqrt(vcz_norm_test); */
	
	vcx_norm=sqrt(vcx[0]*vcx[0]+vcx[1]*vcx[1]+vcx[2]*vcx[2]);
	vcy_norm=sqrt(vcy[0]*vcy[0]+vcy[1]*vcy[1]+vcy[2]*vcy[2]);
	vcz_norm=sqrt(vcz[0]*vcz[0]+vcz[1]*vcz[1]+vcz[2]*vcz[2]);
	
	for(int i=0;i<3;i++)
	{
		vcx[i]=vcx[i]/vcx_norm;	vcy[i]=vcy[i]/vcy_norm;	vcz[i]=vcz[i]/vcz_norm;
	}
	
	//R=[VCx VCy VCz]*inv([vcx vcy vcz]);
	float R1[3][3],R2[3][3]/*,R3[3][3]*/;
	for(int i=0;i<3;i++)
	{
		R1[i][0]=VCX[i];	R1[i][1]=VCY[i];	R1[i][2]=VCZ[i];
		R2[0][i]=vcx[i];	R2[1][i]=vcy[i];	R2[2][i]=vcz[i];	//�]������ncvTranspose(R2, R2);�ҥH�񪺮ɭԪ���
	}
	
	//�x�}�ۭ�(3*3)R1*R2
	float Rext[3][3];
	for(int i=0;i<3;i++)
	{
		for(int j=0;j<3;j++)
		{
			float sum=0.0;
			for(int k=0;k<3;k++)
				sum+=R1[i][k]*R2[k][j];
			Rext[i][j]=sum;
		}
	}
		
	
	Rs[blockIdx.x*9*blockDim.x+tid*9]=Rext[0][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+1]=Rext[0][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+2]=Rext[0][2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+3]=Rext[1][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+4]=Rext[1][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+5]=Rext[1][2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+6]=Rext[2][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+7]=Rext[2][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+8]=Rext[2][2]; 
		
	//for (int k = 0; k< 3; k++)
	//{
	//	cvmSet(text, k, 0, -cvmGet(WL, k, 0));
	//}
	//cvMatMul(Rext, text, text);			
	/* for(int i=0;i<3;i++)		text�ثe�S�Ψ�
	{
		float sum=0.0;
		for(int k=0;k<3;k++)
			sum+=Rext[i][k]*(-WL_test[k]);
		text[i]=sum;
	} */

	}
	
	Ransac(mapft_info, WL ,Rs,parm, ransac_match_num,index_num);

}

__global__ void GPU_P3P_dub(int *matft_NO, int *mapft_NO, double *matft_info, double *mapft_info, int *P3P_NO, double *parm, double *WL ,double *Rs,int *ransac_match_num, int *ransac_match_max, int *index_num)
{
	int tid =threadIdx.x;
	//int blockIdx=blockIdx.x;
	int sol_num=parm[4];//u0,vo,fu,fv
	
	if(blockIdx.x*blockDim.x+tid<sol_num)
	{
	
	double ptw[9],pti[6],dep[3];//�C��thread���ۤv��
	int p1,p2,p3;
	
	p1=P3P_NO[blockIdx.x*3*blockDim.x+tid*3];//�C��thread���T�ӽs��
	p2=P3P_NO[blockIdx.x*3*blockDim.x+tid*3+1];
	p3=P3P_NO[blockIdx.x*3*blockDim.x+tid*3+2];
	
	ptw[0]=matft_info[p1*6];//�Ĥ@���I
	ptw[1]=matft_info[p1*6+1];
	ptw[2]=matft_info[p1*6+2];
	pti[0]=matft_info[p1*6+3];
	pti[1]=matft_info[p1*6+4];
	dep[0]=matft_info[p1*6+5]*matft_info[p1*6+5];
	
	ptw[3]=matft_info[p2*6];//�ĤG���I
	ptw[4]=matft_info[p2*6+1];
	ptw[5]=matft_info[p2*6+2];
	pti[2]=matft_info[p2*6+3];
	pti[3]=matft_info[p2*6+4];
	dep[1]=matft_info[p2*6+5]*matft_info[p2*6+5];
	
	ptw[6]=matft_info[p3*6];//�ĤT���I
	ptw[7]=matft_info[p3*6+1];
	ptw[8]=matft_info[p3*6+2];
	pti[4]=matft_info[p3*6+3];
	pti[5]=matft_info[p3*6+4];
	dep[2]=matft_info[p3*6+5]*matft_info[p3*6+5]; 
	
	 double aa,bb,cc;
	aa= sqrt( ((pti[0] - parm[0])/parm[2])*((pti[0] - parm[0])/parm[2])*dep[0] + ((pti[1]-parm[1])/parm[3])*((pti[1]-parm[1])/parm[3])*dep[0] + dep[0]);
	bb = sqrt( ((pti[2] - parm[0])/parm[2])*((pti[2] - parm[0])/parm[2])*dep[1] + ((pti[3]-parm[1])/parm[3])*((pti[3]-parm[1])/parm[3])*dep[1] + dep[1]);
	cc = sqrt( ((pti[4] - parm[0])/parm[2])*((pti[4] - parm[0])/parm[2])*dep[2] + ((pti[5]-parm[1])/parm[3])*((pti[5]-parm[1])/parm[3])*dep[2] + dep[2]);
	 
	/* Rs[blockIdx.x*9*blockDim.x+tid*9]=ptw[0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+1]=ptw[1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+2]=ptw[2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+3]=ptw[3];
	Rs[blockIdx.x*9*blockDim.x+tid*9+4]=ptw[4];
	Rs[blockIdx.x*9*blockDim.x+tid*9+5]=ptw[5];
	Rs[blockIdx.x*9*blockDim.x+tid*9+6]=ptw[6];
	Rs[blockIdx.x*9*blockDim.x+tid*9+7]=ptw[7];
	Rs[blockIdx.x*9*blockDim.x+tid*9+8]=ptw[8];  */
	
	
	//////////////////////////////////////P3P�p��///////////////////////////////////
	
	///////////////////////////////����v����m//////////////////////////
	double VAB[3],VAC[3],VBC[3];
		for(int i=0;i<3;i++)
		{
			VAB[i]=ptw[i+3]-ptw[i];//2-1
			VAC[i]=ptw[i+6]-ptw[i];//3-1
			VBC[i]=ptw[i+6]-ptw[i+3];//3-1
		}
		
	//Length of edge between control points
	double Rab,Rac,Rbc;
	Rab=sqrt(VAB[0]*VAB[0]+VAB[1]*VAB[1]+VAB[2]*VAB[2]);
	Rac=sqrt(VAC[0]*VAC[0]+VAC[1]*VAC[1]+VAC[2]*VAC[2]);
	Rbc=sqrt(VBC[0]*VBC[0]+VBC[1]*VBC[1]+VBC[2]*VBC[2]);//�᭱�Τ���
	
	double CA[3],CB[3],CC[3];
		CA[0]=(pti[0]-parm[0])/parm[2];	CA[1]=(pti[1]-parm[1])/parm[3];	CA[2]=1;
		CB[0]=(pti[2]-parm[0])/parm[2];	CB[1]=(pti[3]-parm[1])/parm[3];	CB[2]=1;
		CC[0]=(pti[4]-parm[0])/parm[2];	CC[1]=(pti[5]-parm[1])/parm[3];	CC[2]=1;
	
	double RCA,RCB,RCC;
		RCA=sqrt(CA[0]*CA[0]+CA[1]*CA[1]+CA[2]*CA[2]);
		RCB=sqrt(CB[0]*CB[0]+CB[1]*CB[1]+CB[2]*CB[2]);
		RCC=sqrt(CC[0]*CC[0]+CC[1]*CC[1]+CC[2]*CC[2]);
	
	//Normalize
	CA[0]=CA[0]/RCA;	CA[1]=CA[1]/RCA;	CA[2]=CA[2]/RCA;
	CB[0]=CB[0]/RCB;	CB[1]=CB[1]/RCB;	CB[2]=CB[2]/RCB;
	CC[0]=CC[0]/RCC;	CC[1]=CC[1]/RCC;	CC[2]=CC[2]/RCC;
	
		
	double Rab1,Rac1,Rbc1;//�V�q���׭p�� unit mm
		Rab1=sqrt(CA[0]*CA[0]+CA[1]*CA[1]+CA[2]*CA[2]);
		Rac1=sqrt(CB[0]*CB[0]+CB[1]*CB[1]+CB[2]*CB[2]);
		Rbc1=sqrt(CC[0]*CC[0]+CC[1]*CC[1]+CC[2]*CC[2]);
	
	
	//Cosine of angles//�᭱�S�Ψ�
	double Calb, Calc, Cblc;
	//Compute Calb Calc Cblc using Law of Cosine
	 Calb = (2 - Rab1*Rab1) / 2;
	Calc = (2 - Rac1*Rac1) / 2;
	Cblc = (2 - Rbc1*Rbc1) / 2; 
	
	//Get cosine of the angles
	double Clab = (aa*aa + Rab*Rab - bb*bb) / (2 * aa*Rab);
	double Clac = (aa*aa + Rac*Rac - cc*cc) / (2 * aa*Rac);
	
	//Get scale along norm vector
	double Raq = aa*Clab;
	double Rap = aa*Clac;
	
		
	//Get norm vector of plane P1 P2
	double VAB_norm = Rab;
	double VAC_norm = Rac;
	
	double WQ[3],WP[3];
	for (int i = 0; i<3; i++)
	{
		WQ[i]=ptw[i]+ Raq*VAB[i] / VAB_norm;
		WP[i]=ptw[i]+ Rap*VAC[i] / VAC_norm;
	}
	
	//Compute Plane P1 P2 P3
	double NP1[3],NP2[3],NP3[3];
	for (int i = 0; i<3; i++)
	{
		NP1[i]=VAB[i] / VAB_norm;
		NP2[i]=VAC[i] / VAC_norm;
	}
	
	double DP1, DP2, DP3;
	DP1=NP1[0]*WQ[0]+NP1[1]*WQ[1]+NP1[2]*WQ[2];
	DP2=NP2[0]*WP[0]+NP2[1]*WP[1]+NP2[2]*WP[2];
	
	double P1[4],P2[4],P3[4];
	P1[0]=NP1[0];	P1[1]=NP1[1];	P1[2]=NP1[2];	P1[3]=-DP1;
	P2[0]=NP2[0];	P2[1]=NP2[1];	P2[2]=NP2[2];	P2[3]=-DP2;
	
	double VCX[3],VCY[3],VCZ[3];
	for (int i = 0; i<3; i++)
	{
		VCX[i]=CB[i]-CA[i];
		VCY[i]=CC[i]-CA[i];
	}
	
	//�p����u�V�q�~�nZ��V�����q�P�_�ϥΦ�رƦC(�t��AC X AB)
	if ( (VCX[0] * VCY[1] - VCY[0] * VCX[1]) > 0 )	
	{
		//cvCrossProduct(VAC, VAB, NP3);
		NP3[0]=VAC[1]*VAB[2]-VAC[2]*VAB[1];
		NP3[1]=VAC[2]*VAB[0]-VAC[0]*VAB[2];
		NP3[2]=VAC[0]*VAB[1]-VAC[1]*VAB[0];
	}
	else
	{
		//cvCrossProduct(VAB, VAC, NP3);
		NP3[0]=VAB[1]*VAC[2]-VAB[2]*VAC[1];
		NP3[1]=VAB[2]*VAC[0]-VAB[0]*VAC[2];
		NP3[2]=VAB[0]*VAC[1]-VAB[1]*VAC[0];
	}
	
	double NP3_norm=sqrt(NP3[0]*NP3[0]+NP3[1]*NP3[1]+NP3[2]*NP3[2]);
	//Normalize
	NP3[0]=NP3[0]/NP3_norm;	NP3[1]=NP3[1]/NP3_norm;	NP3[2]=NP3[2]/NP3_norm;
	
	DP3=NP3[0]*ptw[0]+NP3[1]*ptw[1]+NP3[2]*ptw[2];//DP3 = cvDotProduct(NP3, W_one);
	
	P3[0]=NP3[0];	P3[1]=NP3[1];	P3[2]=NP3[2];	P3[3]=-DP3;
	
	//�J�԰������D�T������ P1 P2 P3
	double delta, delta_x, delta_y, delta_z;
	delta=P1[0]*P2[1]*P3[2]+P2[0]*P3[1]*P1[2]+P3[0]*P1[1]*P2[2]-P3[0]*P2[1]*P1[2]-P1[0]*P3[1]*P2[2]-P2[0]*P1[1]*P3[2];
	delta_x=P2[1]*P3[2]*P1[3]+P1[1]*P2[2]*P3[3]+P3[1]*P2[3]*P1[2]-P2[1]*P1[2]*P3[3]-P1[1]*P2[3]*P3[2]-P3[1]*P1[3]*P2[2];
	delta_y=P1[0]*P3[2]*P2[3]+P2[0]*P1[2]*P3[3]+P3[0]*P2[2]*P1[3]-P3[0]*P1[2]*P2[3]-P1[0]*P2[2]*P3[3]-P2[0]*P3[2]*P1[3];
	delta_z=P1[0]*P2[1]*P3[3]+P2[0]*P3[1]*P1[3]+P3[0]*P1[1]*P2[3]-P3[0]*P2[1]*P1[3]-P1[0]*P3[1]*P2[3]-P2[0]*P1[1]*P3[3];
	
	double WR[3];
	if(delta!=0)
	{
		//�W���Y�Ʋ����A�]���t�@�ӭt��
		WR[0]=-delta_x/delta;	WR[1]=-delta_y/delta;	WR[2]=-delta_z/delta;
	}
	else
	{
		WR[0]=1000000;	WR[1]=1000000;	WR[2]=1000000;
	}
	
	
	//Get length of LR
	double Rar, Rlr;
	Rar=sqrt((ptw[0]-WR[0])*(ptw[0]-WR[0])+(ptw[1]-WR[1])*(ptw[1]-WR[1])+(ptw[2]-WR[2])*(ptw[2]-WR[2]));
	if(aa*aa - Rar*Rar>0)
		Rlr = sqrt(aa*aa - Rar*Rar);//��aa*aa - Rar*Rar<0�ɡARlr�h��nan
	else
		Rlr = 10e6;
	//Get Position of L in world frame
	//WL=WR+NP3*Rlr;
	double WL_test[3];
	WL_test[0]=WR[0]+NP3[0]*Rlr;
	WL_test[1]=WR[1]+NP3[1]*Rlr;
	WL_test[2]=WR[2]+NP3[2]*Rlr;
	
	WL[blockIdx.x*3*blockDim.x+tid*3]=WL_test[0];
	WL[blockIdx.x*3*blockDim.x+tid*3+1]=WL_test[1];
	WL[blockIdx.x*3*blockDim.x+tid*3+2]=WL_test[2];
	
	///////////////////////////////////////�ا��Шt�����x�}/////////////////////////////////////////////
	VCZ[0]=VCX[1]*VCY[2]-VCX[2]*VCY[1];	//cvCrossProduct(VCX, VCY, VCZ);
	VCZ[1]=VCX[2]*VCY[0]-VCX[0]*VCY[2];
	VCZ[2]=VCX[0]*VCY[1]-VCX[1]*VCY[0];
	
	VCY[0]=VCZ[1]*VCX[2]-VCZ[2]*VCX[1];	//cvCrossProduct(VCZ, VCX, VCY);
	VCY[1]=VCZ[2]*VCX[0]-VCZ[0]*VCX[2];
	VCY[2]=VCZ[0]*VCX[1]-VCZ[1]*VCX[0];
	
	//Normalize
	double VCX_norm,VCY_norm,VCZ_norm;
	VCX_norm=sqrt(VCX[0]*VCX[0]+VCX[1]*VCX[1]+VCX[2]*VCX[2]);
	VCY_norm=sqrt(VCY[0]*VCY[0]+VCY[1]*VCY[1]+VCY[2]*VCY[2]);
	VCZ_norm=sqrt(VCZ[0]*VCZ[0]+VCZ[1]*VCZ[1]+VCZ[2]*VCZ[2]);
	for(int i=0;i<3;i++)
	{
		VCX[i]=VCX[i]/VCX_norm;	VCY[i]=VCY[i]/VCY_norm;	VCZ[i]=VCZ[i]/VCZ_norm;
	}
	
	//%Get ray in the world frame
	double Vla[3],Vlb[3],Vlc[3];
	for(int i=0;i<3;i++)
	{
		Vla[i]=ptw[i]-WL_test[i];	//Vla=WA-WL;
		Vlb[i]=ptw[i+3]-WL_test[i];	//Vlb=WB-WL;
		Vlc[i]=ptw[i+6]-WL_test[i];	//Vlc=WC-WL;
	}
	
	//Normalize
	double Vla_norm,Vlb_norm,Vlc_norm;
	Vla_norm=sqrt(Vla[0]*Vla[0]+Vla[1]*Vla[1]+Vla[2]*Vla[2]);
	Vlb_norm=sqrt(Vlb[0]*Vlb[0]+Vlb[1]*Vlb[1]+Vlb[2]*Vlb[2]);
	Vlc_norm=sqrt(Vlc[0]*Vlc[0]+Vlc[1]*Vlc[1]+Vlc[2]*Vlc[2]);
	for(int i=0;i<3;i++)
	{
		Vla[i]=Vla[i]/Vla_norm;	Vlb[i]=Vlb[i]/Vlb_norm;	Vlc[i]=Vlc[i]/Vlc_norm;
	}
	
	double WA1[3],WB1[3],WC1[3];
	for(int i=0;i<3;i++)
	{
		WA1[i]=WL_test[i]+Vla[i];	//WA1=WL+1*Vla;
		WB1[i]=WL_test[i]+Vlb[i];	//WB1=WL+1*Vlb;
		WC1[i]=WL_test[i]+Vlc[i];	//WC1=WL+1*Vlc;
	}
	
	double vcx[3],vcy[3],vcz[3];
	for(int i=0;i<3;i++)
	{
		vcx[i]=WB1[i]-WA1[i];	//vcx=WB1-WA1;
		vcy[i]=WC1[i]-WA1[i];	//vcy=WC1-WA1;
	}
	
	
	vcz[0]=vcx[1]*vcy[2]-vcx[2]*vcy[1];	//cvCrossProduct(vcx, vcy, vcz);
	vcz[1]=vcx[2]*vcy[0]-vcx[0]*vcy[2];
	vcz[2]=vcx[0]*vcy[1]-vcx[1]*vcy[0];
	
	vcy[0]=vcz[1]*vcx[2]-vcz[2]*vcx[1];	//cvCrossProduct(vcz, vcx, vcy);
	vcy[1]=vcz[2]*vcx[0]-vcz[0]*vcx[2];
	vcy[2]=vcz[0]*vcx[1]-vcz[1]*vcx[0];
	
	
	
	//Normalize
	//float vcx_norm_test,vcy_norm_test,vcz_norm_test;
	double vcx_norm,vcy_norm,vcz_norm;
	
	/* vcx_norm_test=vcx[0]*vcx[0]+vcx[1]*vcx[1]+vcx[2]*vcx[2];
	vcy_norm_test=vcy[0]*vcy[0]+vcy[1]*vcy[1]+vcy[2]*vcy[2];
	vcz_norm_test=vcz[0]*vcz[0]+vcz[1]*vcz[1]+vcz[2]*vcz[2];
	
	if(vcx_norm_test==0)
		vcx_norm=sqrt(10e-12);
	else	vcx_norm=sqrt(vcx_norm_test);
	
	if(vcy_norm_test==0)
		vcy_norm_test=sqrt(10e-12);
	else	vcy_norm=sqrt(vcy_norm_test);
	
	if(vcz_norm_test==0)
		vcz_norm_test=sqrt(10e-12);
	else	vcz_norm=sqrt(vcz_norm_test); */
	
	vcx_norm=sqrt(vcx[0]*vcx[0]+vcx[1]*vcx[1]+vcx[2]*vcx[2]);
	vcy_norm=sqrt(vcy[0]*vcy[0]+vcy[1]*vcy[1]+vcy[2]*vcy[2]);
	vcz_norm=sqrt(vcz[0]*vcz[0]+vcz[1]*vcz[1]+vcz[2]*vcz[2]);
	
	for(int i=0;i<3;i++)
	{
		vcx[i]=vcx[i]/vcx_norm;	vcy[i]=vcy[i]/vcy_norm;	vcz[i]=vcz[i]/vcz_norm;
	}
	
	//R=[VCx VCy VCz]*inv([vcx vcy vcz]);
	double R1[3][3],R2[3][3]/*,R3[3][3]*/;
	for(int i=0;i<3;i++)
	{
		R1[i][0]=VCX[i];	R1[i][1]=VCY[i];	R1[i][2]=VCZ[i];
		R2[0][i]=vcx[i];	R2[1][i]=vcy[i];	R2[2][i]=vcz[i];	//�]������ncvTranspose(R2, R2);�ҥH�񪺮ɭԪ���
	}
	
	//�x�}�ۭ�(3*3)R1*R2
	double Rext[3][3];
	for(int i=0;i<3;i++)
	{
		for(int j=0;j<3;j++)
		{
			double sum=0.0;
			for(int k=0;k<3;k++)
				sum+=R1[i][k]*R2[k][j];
			Rext[i][j]=sum;
		}
	}
		
	
	Rs[blockIdx.x*9*blockDim.x+tid*9]=Rext[0][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+1]=Rext[0][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+2]=Rext[0][2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+3]=Rext[1][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+4]=Rext[1][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+5]=Rext[1][2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+6]=Rext[2][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+7]=Rext[2][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+8]=Rext[2][2]; 
		
	//for (int k = 0; k< 3; k++)
	//{
	//	cvmSet(text, k, 0, -cvmGet(WL, k, 0));
	//}
	//cvMatMul(Rext, text, text);			
	/* for(int i=0;i<3;i++)		text�ثe�S�Ψ�
	{
		float sum=0.0;
		for(int k=0;k<3;k++)
			sum+=Rext[i][k]*(-WL_test[k]);
		text[i]=sum;
	} */

	}
	
	Ransac_dub(mapft_info, WL ,Rs,parm, ransac_match_num,index_num);
	//findmax_device(ransac_match_num, sol_num, WL ,Rs, WL_max, Rs_max, ransac_match_max, index, tem_index,WL_test);
	
	//__shared__ int sol_level;
	
	//sol_level=(sol_num+NT-1)/NT;
	
	//while(sol_level!=1)
	//{
		//findmax_device(ransac_match_max,sol_level,WL_max,Rs_max,WL_max,Rs_max,ransac_match_max,index,tem_index,WL_test1);
		//findmax(ransac_match_num, sol_num, WL ,Rs, WL_max, Rs_max, ransac_match_max, index, tem_index);
		//sol_level=(sol_level+NT-1)/NT;
	//}
	/* if(sol_level==2)
	{
		findmax(ransac_match_max,sol_level,WL_max,Rs_max,WL_max,Rs_max,ransac_match_max,index,tem_index);
		//findmax(ransac_match_num, sol_num, WL ,Rs, WL_max, Rs_max, ransac_match_max, index, tem_index);
	} */
	
	
}


void launchVOKernel(std::vector<keepfeature> matchfeature , std::vector<keepfeature> map_feature,std::vector<Ransac_Pos> &ransac_match,float *GPU_WL,float *GPU_Rs,int *ransac_match_num_all,double l_u0,double l_v0,double l_fu,double l_fv)
{
	int thredperblock = NT;
	int sol_num=(matchfeature.size()*(matchfeature.size()-1)*(matchfeature.size()-2))/6; //n��3
	int blocknum=(sol_num+thredperblock-1)/thredperblock;
	/////////////////// allocate memory on the cpu side//////////////////////
	int *matchfeature_NO,*map_feature_NO;//�s��
	float *matchfeature_info, *map_feature_info;
	int *P3P_NO;//��model���S�x�I�s��
	float *parm, *CameraPose;
	int *ransac_match_max,*index_num,*index_max;//�ŦX�@�Ѷ��X���ƶq
	matchfeature_NO = (int*)malloc(matchfeature.size()* sizeof(int));
	map_feature_NO = (int*)malloc(map_feature.size()* sizeof(int));
	matchfeature_info = (float*)malloc(6 *matchfeature.size()* sizeof(float));//hx hy hz ix iy
	map_feature_info = (float*)malloc(6 *map_feature.size()* sizeof(float));//hx hy hz ix iy
	P3P_NO=(int*)malloc(3 *sol_num* sizeof(int));
	parm=(float*)malloc(6 * sizeof(float));//l_u0,l_v0,l_fu,l_fv,map_feature.size()
	ransac_match_max=(int*)malloc(blocknum* sizeof(int));
	index_num=(int*)malloc(sol_num* sizeof(int));
	index_max=(int*)malloc(blocknum* sizeof(int));
	CameraPose=(float*)malloc(12* sizeof(float));//WL*3 Rs*9
	////////////////// put data in the host memory////////////////////////
	int times=0;
	for(int g=0;g<matchfeature.size()-2;g++)
	{
		for(int g_two=g+1 ;  g_two< matchfeature.size()-1 ; g_two++ )
		{
			for(int g_three=g_two+1 ;  g_three< matchfeature.size() ; g_three++ )
			{
				P3P_NO[times]=g;
				P3P_NO[times+1]=g_two;
				P3P_NO[times+2]=g_three;
				times+=3;
			}
		}
	}
	
	parm[0]=l_u0;parm[1]=l_v0;parm[2]=l_fu;parm[3]=l_fv;parm[4]=sol_num;parm[5]=map_feature.size();
	
	for(int i=0;i<matchfeature.size();i++)
	{
		matchfeature_NO[i]=matchfeature[i].num;
		matchfeature_info[i*6]=matchfeature[i].hx;
		matchfeature_info[i*6+1]=matchfeature[i].hy;
		matchfeature_info[i*6+2]=matchfeature[i].hz;
		matchfeature_info[i*6+3]=matchfeature[i].l_ix;
		matchfeature_info[i*6+4]=matchfeature[i].l_iy;
		matchfeature_info[i*6+5]=matchfeature[i].depthvalue;
	}

	for(int j=0;j<map_feature.size();j++)
	{
		map_feature_NO[j]=map_feature[j].num;
		map_feature_info[j*6]=map_feature[j].hx;
		map_feature_info[j*6+1]=map_feature[j].hy;
		map_feature_info[j*6+2]=map_feature[j].hz;
		map_feature_info[j*6+3]=map_feature[j].l_ix;
		map_feature_info[j*6+4]=map_feature[j].l_iy;
		map_feature_info[j*6+5]=map_feature[j].depthvalue;
	}
		
	////////////////// allocate the memory on the GPU ///////////////////////
	
	int *dev_matchfeature_NO,*dev_map_feature_NO;//�s��
	float *dev_matchfeature_info, *dev_map_feature_info;
	float *dev_WL_all,*dev_Rs_all;
	int *dev_P3P_NO;//��model���S�x�I�s��
	float *dev_parm, *dev_CameraPose;
	int *dev_ransac_match_num_all,*dev_ransac_match_max;//�ŦX�@�Ѷ��X���ƶq
	int *dev_index_num,*dev_index_max;
	hipMalloc((void **)&dev_matchfeature_NO, matchfeature.size()* sizeof(int));
	hipMalloc((void **)&dev_map_feature_NO, map_feature.size()* sizeof(int));
	hipMalloc((void **)&dev_matchfeature_info, 6 *matchfeature.size()* sizeof(float));
	hipMalloc((void **)&dev_map_feature_info, 6 *map_feature.size()* sizeof(float));
	hipMalloc((void **)&dev_WL_all, 3 *sol_num* sizeof(float));
	hipMalloc((void **)&dev_Rs_all, 9 *sol_num* sizeof(float));
	hipMalloc((void **)&dev_P3P_NO, 3 *sol_num* sizeof(int));
	hipMalloc((void **)&dev_parm, 6*sizeof(float));
	hipMalloc((void **)&dev_ransac_match_num_all, sol_num* sizeof(int));
	hipMalloc((void **)&dev_ransac_match_max, blocknum* sizeof(int));//����
	hipMalloc((void **)&dev_index_max, blocknum* sizeof(int));
	hipMalloc((void **)&dev_index_num, sol_num* sizeof(int));
	hipMalloc((void **)&dev_CameraPose, 12* sizeof(float));
	////////////////// copy the arrays to the GPU/////////////////////////
	hipMemcpy(dev_matchfeature_NO, matchfeature_NO, matchfeature.size()* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_map_feature_NO, map_feature_NO, map_feature.size()* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_matchfeature_info, matchfeature_info, 6 *matchfeature.size()* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_map_feature_info, map_feature_info, 6 *map_feature.size()* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_WL_all, GPU_WL, 3 *sol_num* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Rs_all, GPU_Rs, 9 *sol_num* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_P3P_NO, P3P_NO, 3 *sol_num* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_parm, parm, 6* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_ransac_match_num_all, ransac_match_num_all, sol_num* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_ransac_match_max, ransac_match_max, blocknum* sizeof(int), hipMemcpyHostToDevice);//������
	hipMemcpy(dev_index_max, index_max, blocknum* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_index_num, index_num, sol_num* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_CameraPose, CameraPose, 12* sizeof(float), hipMemcpyHostToDevice);
	cout<<"called launchVOKernel_dub ok"<<endl;
	cout<<"blocknum"<<setw(10)<<blocknum<<endl;
	cout<<"thredperblock"<<setw(10)<<thredperblock<<endl;
	
	// Get start time event
	hipEvent_t start_kernel, stop_kernel;
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
	hipEventRecord(start_kernel, 0);
	cout<<"sol_num"<<sol_num<<endl;
	cout<<"blocknum"<<blocknum<<endl;
	int tem_sol_num=sol_num;//�ΨӧP�_��̤j�ȼh��
	int tem_blonum=blocknum;//�ΨӧP�_��̤j�ȼh��
	int sol_times=0;
	
	GPU_P3P << < blocknum, thredperblock >> > (dev_matchfeature_NO,dev_map_feature_NO,dev_matchfeature_info,dev_map_feature_info,dev_P3P_NO,dev_parm,dev_WL_all,dev_Rs_all,dev_ransac_match_num_all,dev_ransac_match_max,dev_index_num);
	
	while(tem_sol_num!=1)
	{
		if(sol_times==0)
			findmax_global << < blocknum, thredperblock >> > (dev_ransac_match_num_all, dev_parm, dev_ransac_match_max, dev_index_num, dev_index_max);
		else
			findmax_global << < tem_blonum, thredperblock >> > (dev_ransac_match_max, dev_parm, dev_ransac_match_max, dev_index_max, dev_index_max);
		
		sol_times++;
		tem_sol_num=tem_blonum;
		tem_blonum=(tem_blonum+NT-1)/NT;
		/* cout<<"sol_num"<<tem_sol_num<<endl;
		cout<<"blonum"<<tem_blonum<<endl; */
		parm[4]=tem_sol_num;
		hipMemcpy(dev_parm, parm, 6* sizeof(float), hipMemcpyHostToDevice);
	}
	
	getCameraPose << < 1,1 >> >(dev_index_max,dev_WL_all,dev_Rs_all,dev_CameraPose);
	//findmax_global << < blocknum, thredperblock >> > (dev_ransac_match_num_all, dev_parm, dev_WL_all ,dev_Rs_all, dev_WL, dev_Rs, dev_ransac_match_max, dev_index, dev_tem_index,dev_WL_test);
	
	/* parm[4]=(sol_num+thredperblock-1)/thredperblock;
	hipMemcpy(dev_parm, parm, 6* sizeof(float), hipMemcpyHostToDevice);
	
	findmax_global << < blocknum, thredperblock >> > (dev_ransac_match_max, dev_parm, dev_WL ,dev_Rs, dev_WL, dev_Rs, dev_ransac_match_max, dev_index, dev_tem_index,dev_WL_test1); */
	
	
	// Get stop time event
	hipEventRecord(stop_kernel, 0);
	hipEventSynchronize(stop_kernel);
	// Compute execution time
	float kernelTime;
	hipEventElapsedTime(&kernelTime, start_kernel, stop_kernel);
	printf("GPU float time: %13f msec\n", kernelTime);
	hipEventDestroy(start_kernel);
	hipEventDestroy(stop_kernel);
	
	
	hipError_t cuda_err = hipGetLastError();
	if (hipSuccess != cuda_err) {
		cout << "before kernel call: error = %s\n" << hipGetErrorString(cuda_err) << endl;
		system("pause");
		exit(1);
	}
	
	hipEvent_t start_tra, stop_tra;
	hipEventCreate(&start_tra);
	hipEventCreate(&stop_tra);
	hipEventRecord(start_tra, 0);
	
	//hipMemcpy(GPU_WL, dev_WL_all, 3 *sol_num* sizeof(float), hipMemcpyDeviceToHost);//test
	//hipMemcpy(GPU_Rs, dev_Rs_all, 9 *sol_num* sizeof(float), hipMemcpyDeviceToHost);//test
	//hipMemcpy(ransac_match_num_all, dev_ransac_match_num_all, sol_num* sizeof(int), hipMemcpyDeviceToHost);//test
	//hipMemcpy(index_max, dev_index_max, blocknum* sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(index_num, dev_index_num, sol_num* sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(ransac_match_max, dev_ransac_match_max, blocknum* sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(CameraPose, dev_CameraPose, 12* sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop_tra, 0);
	hipEventSynchronize(stop_tra);

	// Compute execution time
	float transferTime;
	hipEventElapsedTime(&transferTime, start_tra, stop_tra);
	printf("GPU transfer time: %13f msec\n", transferTime);
	cout<<"---------------------------------------------------"<<endl;
	hipEventDestroy(start_tra);
	hipEventDestroy(stop_tra);
	

	/* for(int i=0;i<1 ;i++)
		cout<<ransac_match_max[i]<<endl;
		//printf("%.13f\n",ransac_match_max[i]);
	cout<<"---------------------------------------------------"<<endl;
	for(int i=0;i<1 ;i++)
		cout<<index_max[i]<<endl;
	cout<<"---------------------------------------------------"<<endl; */
	
	printf("GPU float WL: %.13f\t",CameraPose[0]);
	printf("%.13f\t",CameraPose[1]);
	printf("%.13f\n",CameraPose[2]);
	printf("GPU float Rs: %.13f\t",atan2(CameraPose[3+5], CameraPose[3+8])* 180 / PI + 90);
	printf("%.13f\t",asin(-CameraPose[3+2])* 180 / PI );
	printf("%.13f\n",atan2(CameraPose[3+1], CameraPose[3])* 180 / PI );
	cout<<"---------------------------------------------------"<<endl;
	
	/* for(int i=0;i<sol_num;i++)
		if(ransac_match_num_all[i]==22)
		{
			cout<<ransac_match_num_all[i]<<endl;
			cout<<GPU_WL[i*3]<<endl;
		}
			 */
	/* for(int i=0;i<blocknum  ;i++)
		cout<<ransac_match_max[i]<<endl; */
		//printf("%.13f\n",index[i]);
	
	/* cout<<"-----------------------GPU_ransac_match--------------------------"<<endl;
	for(int i=0;i<sol_num;i++)
	{
		cout<<ransac_match_num[i]<<endl;
	}
	cout<<"-----------------------GPU_ransac_match--------------------------"<<endl;
	cout<<"sol num "<<setw(5)<<sol_num<<endl; */
	
	/* cout<<"-----------------------GPU sol camera pose----------------------------"<<endl; 
	for(int i=0;i<sol_num;i++)
	{
		cout<<GPU_WL[i*3]<<setw(15)<<GPU_WL[i*3+1]<<setw(15)<<GPU_WL[i*3+2]<<endl;
	}
	cout<<"-----------------------GPU sol rotation matrix----------------------------"<<endl;  */
	/* for(int i=0;i<sol_num;i++)
	{
		cout<<Rs[i*9]<<setw(15)<<Rs[i*9+1]<<setw(15)<<Rs[i*9+2]<<endl;
		cout<<Rs[i*9+3]<<setw(15)<<Rs[i*9+4]<<setw(15)<<Rs[i*9+5]<<endl;
		cout<<Rs[i*9+6]<<setw(15)<<Rs[i*9+7]<<setw(15)<<Rs[i*9+8]<<endl;
	} */
	
	/* for(int i=4086;i<4095;i++)
		cout<<Rs[i]<<endl; */
	// Free device memory
	hipFree(dev_matchfeature_NO);
	hipFree(dev_map_feature_NO);
	hipFree(dev_matchfeature_info);
	hipFree(dev_map_feature_info);
	hipFree(dev_WL_all);
	hipFree(dev_Rs_all);
	hipFree(dev_parm);
	hipFree(dev_P3P_NO);
	hipFree(dev_ransac_match_num_all);
	hipFree(dev_ransac_match_max);
	hipFree(dev_index_num);
	hipFree(dev_index_max);
	
}

void launchVOKernel_dub(std::vector<keepfeature> matchfeature , std::vector<keepfeature> map_feature,std::vector<Ransac_Pos> &ransac_match,double *GPU_WL,double *GPU_Rs,int *ransac_match_num_all,double l_u0,double l_v0,double l_fu,double l_fv)
{
	int thredperblock = NT;
	int sol_num=(matchfeature.size()*(matchfeature.size()-1)*(matchfeature.size()-2))/6; //n��3
	int blocknum=(sol_num+thredperblock-1)/thredperblock;
	/////////////////// allocate memory on the cpu side//////////////////////
	int *matchfeature_NO,*map_feature_NO;//�s��
	double *matchfeature_info, *map_feature_info;
	int *P3P_NO;//��model���S�x�I�s��
	double *parm, *CameraPose;
	int *ransac_match_max,*index_num,*index_max;//�ŦX�@�Ѷ��X���ƶq
	matchfeature_NO = (int*)malloc(matchfeature.size()* sizeof(int));
	map_feature_NO = (int*)malloc(map_feature.size()* sizeof(int));
	matchfeature_info = (double*)malloc(6 *matchfeature.size()* sizeof(double));//hx hy hz ix iy
	map_feature_info = (double*)malloc(6 *map_feature.size()* sizeof(double));//hx hy hz ix iy
	P3P_NO=(int*)malloc(3 *sol_num* sizeof(int));
	parm=(double*)malloc(6 * sizeof(double));//l_u0,l_v0,l_fu,l_fv,map_feature.size()
	ransac_match_max=(int*)malloc(blocknum* sizeof(int));
	index_num=(int*)malloc(sol_num* sizeof(int));
	index_max=(int*)malloc(blocknum* sizeof(int));
	CameraPose=(double*)malloc(12* sizeof(double));//WL*3 Rs*9
	////////////////// put data in the host memory////////////////////////
	int times=0;
	for(int g=0;g<matchfeature.size()-2;g++)
	{
		for(int g_two=g+1 ;  g_two< matchfeature.size()-1 ; g_two++ )
		{
			for(int g_three=g_two+1 ;  g_three< matchfeature.size() ; g_three++ )
			{
				P3P_NO[times]=g;
				P3P_NO[times+1]=g_two;
				P3P_NO[times+2]=g_three;
				times+=3;
			}
		}
	}
	
	parm[0]=l_u0;parm[1]=l_v0;parm[2]=l_fu;parm[3]=l_fv;parm[4]=sol_num;parm[5]=map_feature.size();
	
	for(int i=0;i<matchfeature.size();i++)
	{
		matchfeature_NO[i]=matchfeature[i].num;
		matchfeature_info[i*6]=matchfeature[i].hx;
		matchfeature_info[i*6+1]=matchfeature[i].hy;
		matchfeature_info[i*6+2]=matchfeature[i].hz;
		matchfeature_info[i*6+3]=matchfeature[i].l_ix;
		matchfeature_info[i*6+4]=matchfeature[i].l_iy;
		matchfeature_info[i*6+5]=matchfeature[i].depthvalue;
	}

	for(int j=0;j<map_feature.size();j++)
	{
		map_feature_NO[j]=map_feature[j].num;
		map_feature_info[j*6]=map_feature[j].hx;
		map_feature_info[j*6+1]=map_feature[j].hy;
		map_feature_info[j*6+2]=map_feature[j].hz;
		map_feature_info[j*6+3]=map_feature[j].l_ix;
		map_feature_info[j*6+4]=map_feature[j].l_iy;
		map_feature_info[j*6+5]=map_feature[j].depthvalue;
	}
		
	////////////////// allocate the memory on the GPU ///////////////////////
	
	int *dev_matchfeature_NO,*dev_map_feature_NO;//�s��
	double *dev_matchfeature_info, *dev_map_feature_info;
	double *dev_WL_all,*dev_Rs_all;
	int *dev_P3P_NO;//��model���S�x�I�s��
	double *dev_parm, *dev_CameraPose;
	int *dev_ransac_match_num_all,*dev_ransac_match_max;//�ŦX�@�Ѷ��X���ƶq
	int *dev_index_num,*dev_index_max;
	hipMalloc((void **)&dev_matchfeature_NO, matchfeature.size()* sizeof(int));
	hipMalloc((void **)&dev_map_feature_NO, map_feature.size()* sizeof(int));
	hipMalloc((void **)&dev_matchfeature_info, 6 *matchfeature.size()* sizeof(double));
	hipMalloc((void **)&dev_map_feature_info, 6 *map_feature.size()* sizeof(double));
	hipMalloc((void **)&dev_WL_all, 3 *sol_num* sizeof(double));
	hipMalloc((void **)&dev_Rs_all, 9 *sol_num* sizeof(double));
	hipMalloc((void **)&dev_P3P_NO, 3 *sol_num* sizeof(int));
	hipMalloc((void **)&dev_parm, 6*sizeof(double));
	hipMalloc((void **)&dev_ransac_match_num_all, sol_num* sizeof(int));
	hipMalloc((void **)&dev_ransac_match_max, blocknum* sizeof(int));//����
	hipMalloc((void **)&dev_index_max, blocknum* sizeof(int));
	hipMalloc((void **)&dev_index_num, sol_num* sizeof(int));
	hipMalloc((void **)&dev_CameraPose, 12* sizeof(double));
	////////////////// copy the arrays to the GPU/////////////////////////
	hipMemcpy(dev_matchfeature_NO, matchfeature_NO, matchfeature.size()* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_map_feature_NO, map_feature_NO, map_feature.size()* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_matchfeature_info, matchfeature_info, 6 *matchfeature.size()* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_map_feature_info, map_feature_info, 6 *map_feature.size()* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_WL_all, GPU_WL, 3 *sol_num* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_Rs_all, GPU_Rs, 9 *sol_num* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_P3P_NO, P3P_NO, 3 *sol_num* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_parm, parm, 6* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_ransac_match_num_all, ransac_match_num_all, sol_num* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_ransac_match_max, ransac_match_max, blocknum* sizeof(int), hipMemcpyHostToDevice);//������
	hipMemcpy(dev_index_max, index_max, blocknum* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_index_num, index_num, sol_num* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_CameraPose, CameraPose, 12* sizeof(double), hipMemcpyHostToDevice);
	cout<<"called launchVOKernel_dub ok"<<endl;
	cout<<"blocknum"<<setw(10)<<blocknum<<endl;
	cout<<"thredperblock"<<setw(10)<<thredperblock<<endl;
	
	// Get start time event
	hipEvent_t start_kernel, stop_kernel;
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
	hipEventRecord(start_kernel, 0);
	cout<<"sol_num"<<sol_num<<endl;
	cout<<"blocknum"<<blocknum<<endl;
	int tem_sol_num=sol_num;//�ΨӧP�_��̤j�ȼh��
	int tem_blonum=blocknum;//�ΨӧP�_��̤j�ȼh��
	int sol_times=0;
	
	GPU_P3P_dub << < blocknum, thredperblock >> > (dev_matchfeature_NO,dev_map_feature_NO,dev_matchfeature_info,dev_map_feature_info,dev_P3P_NO,dev_parm,dev_WL_all,dev_Rs_all,dev_ransac_match_num_all,dev_ransac_match_max,dev_index_num);
	
	while(tem_sol_num!=1)
	{
		if(sol_times==0)
			findmax_global_dub << < blocknum, thredperblock >> > (dev_ransac_match_num_all, dev_parm, dev_ransac_match_max, dev_index_num, dev_index_max);
		else
			findmax_global_dub << < tem_blonum, thredperblock >> > (dev_ransac_match_max, dev_parm, dev_ransac_match_max, dev_index_max, dev_index_max);
		
		sol_times++;
		tem_sol_num=tem_blonum;
		tem_blonum=(tem_blonum+NT-1)/NT;
		/* cout<<"sol_num"<<tem_sol_num<<endl;
		cout<<"blonum"<<tem_blonum<<endl; */
		parm[4]=tem_sol_num;
		hipMemcpy(dev_parm, parm, 6* sizeof(double), hipMemcpyHostToDevice);
	}
	
	getCameraPose_dub << < 1,1 >> >(dev_index_max,dev_WL_all,dev_Rs_all,dev_CameraPose);
	//findmax_global << < blocknum, thredperblock >> > (dev_ransac_match_num_all, dev_parm, dev_WL_all ,dev_Rs_all, dev_WL, dev_Rs, dev_ransac_match_max, dev_index, dev_tem_index,dev_WL_test);
	
	/* parm[4]=(sol_num+thredperblock-1)/thredperblock;
	hipMemcpy(dev_parm, parm, 6* sizeof(double), hipMemcpyHostToDevice);
	
	findmax_global << < blocknum, thredperblock >> > (dev_ransac_match_max, dev_parm, dev_WL ,dev_Rs, dev_WL, dev_Rs, dev_ransac_match_max, dev_index, dev_tem_index,dev_WL_test1); */
	
	
	// Get stop time event
	hipEventRecord(stop_kernel, 0);
	hipEventSynchronize(stop_kernel);
	// Compute execution time
	float kernelTime;
	hipEventElapsedTime(&kernelTime, start_kernel, stop_kernel);
	printf("GPU double time: %13f msec\n", kernelTime);
	hipEventDestroy(start_kernel);
	hipEventDestroy(stop_kernel);
	
	
	hipError_t cuda_err = hipGetLastError();
	if (hipSuccess != cuda_err) {
		cout << "before kernel call: error = %s\n" << hipGetErrorString(cuda_err) << endl;
		system("pause");
		exit(1);
	}
	
	hipEvent_t start_tra, stop_tra;
	hipEventCreate(&start_tra);
	hipEventCreate(&stop_tra);
	hipEventRecord(start_tra, 0);
	
	//hipMemcpy(GPU_WL, dev_WL_all, 3 *sol_num* sizeof(double), hipMemcpyDeviceToHost);//test
	//hipMemcpy(GPU_Rs, dev_Rs_all, 9 *sol_num* sizeof(double), hipMemcpyDeviceToHost);//test
	//hipMemcpy(ransac_match_num_all, dev_ransac_match_num_all, sol_num* sizeof(int), hipMemcpyDeviceToHost);//test
	//hipMemcpy(index_max, dev_index_max, blocknum* sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(index_num, dev_index_num, sol_num* sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(ransac_match_max, dev_ransac_match_max, blocknum* sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(CameraPose, dev_CameraPose, 12* sizeof(double), hipMemcpyDeviceToHost);
	hipEventRecord(stop_tra, 0);
	hipEventSynchronize(stop_tra);

	// Compute execution time
	float transferTime;
	hipEventElapsedTime(&transferTime, start_tra, stop_tra);
	printf("GPU transfer time: %13f msec\n", transferTime);
	cout<<"---------------------------------------------------"<<endl;
	hipEventDestroy(start_tra);
	hipEventDestroy(stop_tra);
	

	/* for(int i=0;i<1 ;i++)
		cout<<ransac_match_max[i]<<endl;
		//printf("%.13f\n",ransac_match_max[i]);
	cout<<"---------------------------------------------------"<<endl;
	for(int i=0;i<1 ;i++)
		cout<<index_max[i]<<endl;
	cout<<"---------------------------------------------------"<<endl; */
	
	printf("GPU double WL: %.13f\t",CameraPose[0]);
	printf("%.13f\t",CameraPose[1]);
	printf("%.13f\n",CameraPose[2]);
	printf("GPU double Rs: %.13f\t",atan2(CameraPose[3+5], CameraPose[3+8])* 180 / PI + 90);
	printf("%.13f\t",asin(-CameraPose[3+2])* 180 / PI );
	printf("%.13f\n",atan2(CameraPose[3+1], CameraPose[3])* 180 / PI );
	cout<<"---------------------------------------------------"<<endl;
	
	/* for(int i=0;i<sol_num;i++)
		if(ransac_match_num_all[i]==22)
		{
			cout<<ransac_match_num_all[i]<<endl;
			cout<<GPU_WL[i*3]<<endl;
		}
			 */
	/* for(int i=0;i<blocknum  ;i++)
		cout<<ransac_match_max[i]<<endl; */
		//printf("%.13f\n",index[i]);
	
	/* cout<<"-----------------------GPU_ransac_match--------------------------"<<endl;
	for(int i=0;i<sol_num;i++)
	{
		cout<<ransac_match_num[i]<<endl;
	}
	cout<<"-----------------------GPU_ransac_match--------------------------"<<endl;
	cout<<"sol num "<<setw(5)<<sol_num<<endl; */
	
	/* cout<<"-----------------------GPU sol camera pose----------------------------"<<endl; 
	for(int i=0;i<sol_num;i++)
	{
		cout<<GPU_WL[i*3]<<setw(15)<<GPU_WL[i*3+1]<<setw(15)<<GPU_WL[i*3+2]<<endl;
	}
	cout<<"-----------------------GPU sol rotation matrix----------------------------"<<endl;  */
	/* for(int i=0;i<sol_num;i++)
	{
		cout<<Rs[i*9]<<setw(15)<<Rs[i*9+1]<<setw(15)<<Rs[i*9+2]<<endl;
		cout<<Rs[i*9+3]<<setw(15)<<Rs[i*9+4]<<setw(15)<<Rs[i*9+5]<<endl;
		cout<<Rs[i*9+6]<<setw(15)<<Rs[i*9+7]<<setw(15)<<Rs[i*9+8]<<endl;
	} */
	
	/* for(int i=4086;i<4095;i++)
		cout<<Rs[i]<<endl; */
	// Free device memory
	hipFree(dev_matchfeature_NO);
	hipFree(dev_map_feature_NO);
	hipFree(dev_matchfeature_info);
	hipFree(dev_map_feature_info);
	hipFree(dev_WL_all);
	hipFree(dev_Rs_all);
	hipFree(dev_parm);
	hipFree(dev_P3P_NO);
	hipFree(dev_ransac_match_num_all);
	hipFree(dev_ransac_match_max);
	hipFree(dev_index_num);
	hipFree(dev_index_max);
	
}


