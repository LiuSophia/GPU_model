#include "hip/hip_runtime.h"

#include "kernelP3P.cuh"
//#include "P3P.h
#include <vector>
#include <stdio.h>
#include <stdlib.h>
using namespace std;



__device__ void findmin1_test(float *data, int num, float* min, int* index) {
	
	int tid = threadIdx.x;
	int blockid = blockIdx.x;
	int couts =0;//�@��block�ݭn��������(�q�Ҧ�data)
	int min_couts=0;//�@��block�ݭn��������(�qmin)

	couts=(num+blockDim.x-1)/blockDim.x;

	
	__shared__ int k;
	__shared__ int m;
	__shared__ int tem_index[NT/2];
	__shared__ float tem_descriptor[NT/2] ;
	k = 0;
	
		while (k<couts)//�P�_k<couts
		{
			if (num - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;
				m = 0;
				while (i != 0)
				{
					if (tid < i) {
						if (m == 0)//�Ĥ@������ishare memory
						{
							if (data[blockid * num + k*blockDim.x + tid] > data[blockid * num + k*blockDim.x + tid + i])
							{
								tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid + i];
								tem_index[tid] = k*blockDim.x + tid + i;
							}
							else
							{
								tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid];
								tem_index[tid] = k*blockDim.x + tid;
							}
							__syncthreads();
						}
						else//�ĤG���qshare memory ���Ȥ�
						{
							if (tem_descriptor[tid+ i ] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + i];
								tem_index[tid]= tem_index[tid + i];
							}
						}
					}
					__syncthreads();
					i /= 2;
					m++;
				}
			}
			else//�B�z�h�l
			{
				int remain = num - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				m = 0;
				while (j != 0)
				{
					if (tid < j) 
					{
						if (m == 0)//�Ĥ@������ishare memory
						{
							if (data[blockid * num + k*blockDim.x + tid] > data[blockid * num + k*blockDim.x + tid + j])
							{
								tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid + j];
								tem_index[tid] = k*blockDim.x + tid + j;
							}
							else
							{
								tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid];
								tem_index[tid] = k*blockDim.x + tid;
							}
							if (tid + j * 2 < tem_j) {
									if (data[blockid * num + k*blockDim.x + tid + j * 2] < tem_descriptor[tid])//�h�l���n��w�g��shared����
									{
										tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid + j * 2];
										tem_index[tid] = k*blockDim.x + remain - 1;
									}
								}
							__syncthreads();
						}
						else//�ĤG���qshare memory ���Ȥ�
						{
							if (tem_descriptor[tid+j] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + j];
								tem_index[tid] = tem_index[tid + j];
							}
							if (tid + j * 2 < tem_j) {
								if (tem_descriptor[tid + j * 2] < tem_descriptor[tid])
								{
									tem_descriptor[tid] = tem_descriptor[tid + j * 2];
									tem_index[tid] = tem_index[tid + j * 2];
								}
							}
						}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid ];
						tem_index[tid] = k*blockDim.x + tid;
					}
				}
			}
			if(tid==0)
			{
				min[blockid * couts + k] = tem_descriptor[0];
				index[blockid * couts + k] = tem_index[0];
			}
			__syncthreads();
			k++;
		}

		min_couts=(couts+blockDim.x-1)/blockDim.x;
		k=0;
		
		while(k<min_couts)
		{
			if(couts - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;
				m = 0;
				while (i != 0)
				{
					if (tid < i) {
						if(m==0)
						{
							if(min[blockid*couts +k*blockDim.x+tid]<min[blockid*couts +k*blockDim.x+tid+i])
							{
								tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid];
								tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
							}
							else
							{
								tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid+i];
								tem_index[tid] = index[blockid*couts + k*blockDim.x + tid+i];
							}
						__syncthreads();
						}
						else
						{
							if (tem_descriptor[tid+ i ] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + i];
								tem_index[tid]= tem_index[tid + i];
							}
						}
						
					}
					__syncthreads();
					i /= 2;
					m++;
				}
			}
			else
			{
				int remain = couts - blockDim.x*k;//7
				int tem_j = remain;
				int j = remain / 2;//3
				m = 0;
				while (j != 0)
				{
					if (tid < j) 
					{
						if(m==0)
						{
							if(min[blockid*couts +k*blockDim.x+tid]<min[blockid*couts +k*blockDim.x+tid+j])
							{
								tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid];
								tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
							}
							else
							{
								tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid+j];
								tem_index[tid] = index[blockid*couts + k*blockDim.x + tid+j];
							}
							if (tid + j * 2 < tem_j) {
								if (min[blockid*couts +k*blockDim.x+tid + j * 2] < tem_descriptor[tid])
								{
									tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid + j * 2];
									tem_index[tid] = index[blockid*couts + k*blockDim.x + tid + j * 2];
								}
							}
						__syncthreads();
						}
						else
						{
							if (tem_descriptor[tid+j] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + j];
								tem_index[tid] = tem_index[tid + j];
							}
							if (tid + j * 2 < tem_j) {
								if (tem_descriptor[tid + j * 2] < tem_descriptor[tid])
								{
									tem_descriptor[tid] = tem_descriptor[tid + j * 2];
									tem_index[tid] = tem_index[tid + j * 2];
								}
							}
						}
							
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						tem_descriptor[tid] = min[blockid * couts + k*blockDim.x + tid ];
						tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
					}
				}
				
			}
			
			if(tid==0)
			{
				min[blockid * couts + k] = tem_descriptor[0];
				index[blockid * couts + k] = tem_index[0];
			}
			k++;

		}
		/* if (tid < couts)
		{
			int i = couts / 2;
			int tem_i = couts;
			while (i != 0)
			{
				if (tid < i)
				{
					if (min[blockid*couts + tid] > min[blockid*couts + tid + i])
					{
						min[blockid*couts + tid] = min[blockid*couts + tid + i];
						index[blockid*couts + tid] = index[blockid*couts + tid + i];
					}
					if (tid + i * 2 < tem_i) {
						if (min[blockid*couts + tid] >min[blockid*couts + tid + 2 * i])
						{
							min[blockid*couts + tid] = min[blockid*couts + tid + 2 * i];
							index[blockid*couts + tid] = index[blockid*couts + tid + 2 * i];
						}
					}//�p�G�h�l���O�_��
				}
				__syncthreads();
				i /= 2;
				tem_i /= 2;
			}
		} */
}

__device__ void findmin_global(float *data, int num, float* min, int* index,int* tem_index)
{
	int tid = threadIdx.x;
	int blockid = blockIdx.x;
	int couts =0;//�@��block�ݭn��������(�q�Ҧ�data)
	int min_couts=0;//�@��block�ݭn��������(�qmin)
	couts=(num+blockDim.x-1)/blockDim.x;
	int k=0,m=0;
	//__shared__ int tem_index[NT/2];//global
	
		while (k<couts)//�P�_k<couts
		{
			if (num - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;
				m=0;
				while (i != 0)
				{
					if (tid < i) {
						if (data[blockid * num + k*blockDim.x + tid] > data[blockid * num + k*blockDim.x + tid + i])
						{
							data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid + i];
							if(m==0)	tem_index[blockid*NT/2+tid] = k*blockDim.x + tid + i;
							else	tem_index[blockid*NT/2+tid] = tem_index[blockid*NT/2+tid+i];
						}
						else
						{
							if(m==0)	tem_index[blockid*NT/2+tid] = k*blockDim.x + tid;
						}	
					}
					__syncthreads();
					i /= 2;
					m++;
				}
			}
			else//�B�z�h�l
			{
				int remain = num - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				m=0;
				while (j != 0)
				{
					if (tid < j) 
					{
						
						if (data[blockid * num + k*blockDim.x + tid] > data[blockid * num + k*blockDim.x + tid + j])
						{
							data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid + j];
							/* if(m==0)	tem_index[tid] = k*blockDim.x + tid + j;
							else	tem_index[tid] = tem_index[tid+j]; */
							if(m==0)	tem_index[blockid*NT/2+tid] = k*blockDim.x + tid + j;
							else	tem_index[blockid*NT/2+tid] = tem_index[blockid*NT/2+tid+j];
						}
						else
						{
							if(m==0)	tem_index[blockid*NT/2+tid] = k*blockDim.x + tid;
						}
						if (tid + j * 2 < tem_j) {
							if (data[blockid * num + k*blockDim.x + tid + j * 2] < data[blockid * num + k*blockDim.x + tid])
							{
								data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid + j * 2];
								/* if(m==0)	tem_index[tid] = k*blockDim.x + remain - 1;
								else	tem_index[tid] = tem_index[tid+j*2]; */
								if(m==0)	tem_index[blockid*NT/2+tid] = k*blockDim.x + remain - 1;
								else	tem_index[blockid*NT/2+tid] = tem_index[blockid*NT/2+tid+j*2];
							}
						}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid ];
						tem_index[blockid*NT/2+tid] = k*blockDim.x + tid;
					}
				}
			}
			if(tid==0)
			{
				min[blockid * couts + k] = data[blockid * num + k*blockDim.x];
				index[blockid * couts + k] = tem_index[blockid*NT/2];
			}
			
			k++;
		}
		
		min_couts=(couts+blockDim.x-1)/blockDim.x;//3
		k=0;
		
		while(k<min_couts)
		{
			if(couts - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;

				while (i != 0)
				{
					if (tid < i) {
							if (min[blockid*couts +k*blockDim.x+tid+ i ] < min[blockid*couts +k*blockDim.x+tid])
							{
								min[blockid*couts +k*blockDim.x+tid] = min[blockid*couts +k*blockDim.x+tid+ i ];
								index[blockid*couts +k*blockDim.x+tid]= index[blockid*couts +k*blockDim.x+tid + i];
							}
					}
					__syncthreads();
					i /= 2;
					//m++;
				}
			}
			else
			{
				int remain = couts - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				
				while (j != 0)
				{
					if (tid < j) 
					{
							if (min[blockid*couts +k*blockDim.x+tid+j] < min[blockid*couts +k*blockDim.x+tid])
							{
								min[blockid*couts +k*blockDim.x+tid] = min[blockid*couts +k*blockDim.x+tid + j];
								index[blockid*couts +k*blockDim.x+tid] = index[blockid*couts +k*blockDim.x+tid + j];
							}
							if (tid + j * 2 < tem_j) {
								if (min[blockid*couts +k*blockDim.x+tid + j * 2] < min[blockid*couts +k*blockDim.x+tid])
								{
									min[blockid*couts +k*blockDim.x+tid] = min[blockid*couts +k*blockDim.x+tid + j * 2];
									index[blockid*couts +k*blockDim.x+tid] = index[blockid*couts +k*blockDim.x+tid + j * 2];
								}
							}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					//m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						min[blockid*couts +k*blockDim.x+tid] = min[blockid * couts + k*blockDim.x + tid ];
						index[blockid*couts +k*blockDim.x+tid] = index[blockid*couts + k*blockDim.x + tid];
					}
				}
				
			}
			
			if(tid==0)
			{
				min[blockid * couts + k] = min[blockid*couts +k*blockDim.x];
				index[blockid * couts + k] = index[blockid*couts +k*blockDim.x];
			}
			k++;

		}
		
}

__device__ void findmin1(float *data, int num, float* min, int* index) {
	
	int tid = threadIdx.x;
	int blockid = blockIdx.x;
	int couts =0;//�@��block�ݭn��������(�q�Ҧ�data)
	int min_couts=0;//�@��block�ݭn��������(�qmin)
	couts=(num+blockDim.x-1)/blockDim.x;

	
	__shared__ int k;//block���F�ĴX��
	//__shared__ int m;
	__shared__ int tem_index[NT];
	__shared__ float tem_descriptor[NT] ;
	
	k = 0; //m = 0;
	
		while (k<couts)//�P�_k<couts
		{
			if (num - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;
				//m = 0;
				//if (m == 0)//�Ĥ@������ishare memory
				//{
					tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid];
					tem_index[tid] = k*blockDim.x + tid;
					__syncthreads();
				//}
				
				while (i != 0)
				{
					if (tid < i) {
							if (tem_descriptor[tid+ i ] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + i];
								tem_index[tid]= tem_index[tid + i];
							}
					}
					__syncthreads();
					i /= 2;
					//m++;
				}
			}
			else//�B�z�h�l
			{
				int remain = num - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				//m = 0;
				if (/*m == 0 && */tid<remain)//�Ĥ@������ishare memory
				{
					tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid];
					tem_index[tid] = k*blockDim.x + tid;
					__syncthreads();
				}
				
				while (j != 0)
				{
					if (tid < j) 
					{
							if (tem_descriptor[tid+j] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + j];
								tem_index[tid] = tem_index[tid + j];
							}
							if (tid + j * 2 < tem_j) {
								if (tem_descriptor[tid + j * 2] < tem_descriptor[tid])
								{
									tem_descriptor[tid] = tem_descriptor[tid + j * 2];
									tem_index[tid] = tem_index[tid + j * 2];
								}
							}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					//m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						tem_descriptor[tid] = data[blockid * num + k*blockDim.x + tid ];
						tem_index[tid] = k*blockDim.x + tid;
					}
				}
			}
			
			if(tid==0)
			{
				min[blockid * couts + k] = tem_descriptor[0];
				index[blockid * couts + k] = tem_index[0];
			}
			
			k++;
		}

		min_couts=(couts+blockDim.x-1)/blockDim.x;//3
		k=0;
		
		while(k<min_couts)
		{
			if(couts - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;
				//m = 0;
				//if(m==0)
				//{
					tem_descriptor[tid] = min[blockid*couts +k*blockDim.x+tid];
					tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
					__syncthreads();
				//}
				while (i != 0)
				{
					if (tid < i) {
							if (tem_descriptor[tid+ i ] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + i];
								tem_index[tid]= tem_index[tid + i];
							}
					}
					__syncthreads();
					i /= 2;
					//m++;
				}
			}
			else
			{
				int remain = couts - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				//m = 0;
				if (/*m == 0 &&*/ tid<remain)//�Ĥ@������ishare memory
				{
					tem_descriptor[tid] = min[blockid*couts + k*blockDim.x + tid];
					tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
					__syncthreads();
				}
				
				while (j != 0)
				{
					if (tid < j) 
					{
							if (tem_descriptor[tid+j] < tem_descriptor[tid])
							{
								tem_descriptor[tid] = tem_descriptor[tid + j];
								tem_index[tid] = tem_index[tid + j];
							}
							if (tid + j * 2 < tem_j) {
								if (tem_descriptor[tid + j * 2] < tem_descriptor[tid])
								{
									tem_descriptor[tid] = tem_descriptor[tid + j * 2];
									tem_index[tid] = tem_index[tid + j * 2];
								}
							}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					//m++;
				}
				if(remain==1)//�p�G�Ѥ@��
				{
					if (tid < remain)
					{
						tem_descriptor[tid] = min[blockid * couts + k*blockDim.x + tid ];
						tem_index[tid] = index[blockid*couts + k*blockDim.x + tid];
					}
				}
				
			}
			
			if(tid==0)
			{
				min[blockid * couts + k] = tem_descriptor[0];
				index[blockid * couts + k] = tem_index[0];
			}
			k++;

		}
		

}

__global__ void matching(float *d_map, int *l_map, float *d_img, int *l_img, float *result, int img_num,int map_num,int descriptorDim,float *min, int *index,float* gpu_data_test,int *tem_index)
{
	int tid = threadIdx.x;
	int temp_tid = 0;
	float SAD = 0.0;
	__shared__ float sum[NT] ;
	
		for (int j = 0; j < img_num; j++)
		{
			temp_tid=tid;
			SAD = 0.0;
			if (l_map[blockIdx.x] == l_img[j])
			{
				 for (int i = 0; i < descriptorDim/NT; i++)
				{ 
				//SAD = (d_map[blockIdx.x *descriptorDim+ tid] - d_img[j*descriptorDim + tid])*(d_map[blockIdx.x *descriptorDim+ tid] - d_img[j*descriptorDim + tid]);
					 SAD += (d_map[blockIdx.x *descriptorDim+ temp_tid] - d_img[j*descriptorDim + temp_tid])*(d_map[blockIdx.x *descriptorDim+ temp_tid] - d_img[j*descriptorDim + temp_tid]);
					temp_tid += blockDim.x;
				} 
				sum[tid] = SAD;
				__syncthreads();

				int i = blockDim.x / 2;
				while (i != 0)
				{
					if (tid < i) {
						sum[tid] += sum[tid + i];
					}
					__syncthreads();
					i /= 2;
				}
				
				if(tid==0)
					sum[0] = sqrt(sum[0]);
				__syncthreads();
				
				result[img_num*blockIdx.x + j] = sum[0];
			}
			else
			{
				result[img_num*blockIdx.x + j] = 10.0;
			}

		}

	//findmin1(result, img_num, min, index);//��̤p shared memory=threads��
	findmin_global(gpu_data_test, img_num, min, index,tem_index);//��̤p global memory
	//findmin1_test(result, img_num, min, index); //��̤p shared memory threads�Ƥ@�b
	
		

}

__device__ void Ransac(float *mapft_info, float *WL ,float *Rs,float *parm, int *ransac_match_num)
{
	int tid =threadIdx.x;
	int sol_num=parm[4];//u0,vo,fu,fv
	int map_num=parm[5];
	float range=25.0;
	if(blockIdx.x*blockDim.x+tid<sol_num)
	{
		float rx = WL[blockIdx.x*3*blockDim.x+tid*3];
		float ry = WL[blockIdx.x*3*blockDim.x+tid*3+1];
		float rz = WL[blockIdx.x*3*blockDim.x+tid*3+2];
		
		int setsize=0;
		float dist=0;
		for(int i=0;i<map_num;i++)
		{
			float feature[3], h[3];//, h2[3];
			feature[0] = (mapft_info[i*6+3] - parm[0]) / parm[2] * mapft_info[i*6+5];
			feature[1] = (mapft_info[i*6+4] - parm[1]) / parm[3] * mapft_info[i*6+5];
			feature[2] = mapft_info[i*6+5];
			
			h[0] = Rs[blockIdx.x*9*blockDim.x+tid*9] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+3] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+6] * feature[2];
			h[1] = Rs[blockIdx.x*9*blockDim.x+tid*9+1] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+4] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+7] * feature[2];
			h[2] = Rs[blockIdx.x*9*blockDim.x+tid*9+2] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+5] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+8] * feature[2];
			
			dist=0;
			dist = (h[0] - mapft_info[i*6] + rx)*(h[0] - mapft_info[i*6] + rx) +
					(h[1] - mapft_info[i*6+1] + ry)*(h[1] - mapft_info[i*6+1] + ry) +
					(h[2] - mapft_info[i*6+2] + rz)*(h[2] - mapft_info[i*6+2] + rz);
			dist = sqrt(dist);
			if (dist < range)
			{
				setsize++;//�ŦX���󪺯S�x�I�Ӽ�
			}

		}
		ransac_match_num[blockIdx.x*blockDim.x+tid]=setsize;
	}
}

__device__ void Ransac_dub(double *mapft_info, double *WL ,double *Rs,double *parm, int *ransac_match_num)
{
	int tid =threadIdx.x;
	int sol_num=parm[4];//u0,vo,fu,fv
	int map_num=parm[5];
	double range=25.0;
	if(blockIdx.x*blockDim.x+tid<sol_num)
	{
		double rx = WL[blockIdx.x*3*blockDim.x+tid*3];
		double ry = WL[blockIdx.x*3*blockDim.x+tid*3+1];
		double rz = WL[blockIdx.x*3*blockDim.x+tid*3+2];
		
		int setsize=0;
		double dist=0;
		for(int i=0;i<map_num;i++)
		{
			double feature[3], h[3];//, h2[3];
			feature[0] = (mapft_info[i*6+3] - parm[0]) / parm[2] * mapft_info[i*6+5];
			feature[1] = (mapft_info[i*6+4] - parm[1]) / parm[3] * mapft_info[i*6+5];
			feature[2] = mapft_info[i*6+5];
			
			h[0] = Rs[blockIdx.x*9*blockDim.x+tid*9] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+3] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+6] * feature[2];
			h[1] = Rs[blockIdx.x*9*blockDim.x+tid*9+1] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+4] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+7] * feature[2];
			h[2] = Rs[blockIdx.x*9*blockDim.x+tid*9+2] * feature[0] + Rs[blockIdx.x*9*blockDim.x+tid*9+5] * feature[1] + Rs[blockIdx.x*9*blockDim.x+tid*9+8] * feature[2];
			
			dist=0;
			dist = (h[0] - mapft_info[i*6] + rx)*(h[0] - mapft_info[i*6] + rx) +
					(h[1] - mapft_info[i*6+1] + ry)*(h[1] - mapft_info[i*6+1] + ry) +
					(h[2] - mapft_info[i*6+2] + rz)*(h[2] - mapft_info[i*6+2] + rz);
			dist = sqrt(dist);
			if (dist < range)
			{
				setsize++;//�ŦX���󪺯S�x�I�Ӽ�
			}

		}
		ransac_match_num[blockIdx.x*blockDim.x+tid]=setsize;
	}
}


__global__ void GPU_P3P(int *matft_NO, int *mapft_NO, float *matft_info, float *mapft_info, int *P3P_NO, float *parm, float *WL ,float *Rs,int *ransac_match_num)
{
	int tid =threadIdx.x;
	//int blockIdx=blockIdx.x;
	int sol_num=parm[4];//u0,vo,fu,fv
	
	if(blockIdx.x*blockDim.x+tid<sol_num)
	{
	
	float ptw[9],pti[6],dep[3];//�C��thread���ۤv��
	int p1,p2,p3;
	
	p1=P3P_NO[blockIdx.x*3*blockDim.x+tid*3];//�C��thread���T�ӽs��
	p2=P3P_NO[blockIdx.x*3*blockDim.x+tid*3+1];
	p3=P3P_NO[blockIdx.x*3*blockDim.x+tid*3+2];
	
	ptw[0]=matft_info[p1*6];//�Ĥ@���I
	ptw[1]=matft_info[p1*6+1];
	ptw[2]=matft_info[p1*6+2];
	pti[0]=matft_info[p1*6+3];
	pti[1]=matft_info[p1*6+4];
	dep[0]=matft_info[p1*6+5]*matft_info[p1*6+5];
	
	ptw[3]=matft_info[p2*6];//�ĤG���I
	ptw[4]=matft_info[p2*6+1];
	ptw[5]=matft_info[p2*6+2];
	pti[2]=matft_info[p2*6+3];
	pti[3]=matft_info[p2*6+4];
	dep[1]=matft_info[p2*6+5]*matft_info[p2*6+5];
	
	ptw[6]=matft_info[p3*6];//�ĤT���I
	ptw[7]=matft_info[p3*6+1];
	ptw[8]=matft_info[p3*6+2];
	pti[4]=matft_info[p3*6+3];
	pti[5]=matft_info[p3*6+4];
	dep[2]=matft_info[p3*6+5]*matft_info[p3*6+5]; 
	
	 float aa,bb,cc;
	aa= sqrt( ((pti[0] - parm[0])/parm[2])*((pti[0] - parm[0])/parm[2])*dep[0] + ((pti[1]-parm[1])/parm[3])*((pti[1]-parm[1])/parm[3])*dep[0] + dep[0]);
	bb = sqrt( ((pti[2] - parm[0])/parm[2])*((pti[2] - parm[0])/parm[2])*dep[1] + ((pti[3]-parm[1])/parm[3])*((pti[3]-parm[1])/parm[3])*dep[1] + dep[1]);
	cc = sqrt( ((pti[4] - parm[0])/parm[2])*((pti[4] - parm[0])/parm[2])*dep[2] + ((pti[5]-parm[1])/parm[3])*((pti[5]-parm[1])/parm[3])*dep[2] + dep[2]);
	 
	/* Rs[blockIdx.x*9*blockDim.x+tid*9]=ptw[0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+1]=ptw[1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+2]=ptw[2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+3]=ptw[3];
	Rs[blockIdx.x*9*blockDim.x+tid*9+4]=ptw[4];
	Rs[blockIdx.x*9*blockDim.x+tid*9+5]=ptw[5];
	Rs[blockIdx.x*9*blockDim.x+tid*9+6]=ptw[6];
	Rs[blockIdx.x*9*blockDim.x+tid*9+7]=ptw[7];
	Rs[blockIdx.x*9*blockDim.x+tid*9+8]=ptw[8];  */
	
	
	//////////////////////////////////////P3P�p��///////////////////////////////////
	
	///////////////////////////////����v����m//////////////////////////
	float VAB[3],VAC[3],VBC[3];
		for(int i=0;i<3;i++)
		{
			VAB[i]=ptw[i+3]-ptw[i];//2-1
			VAC[i]=ptw[i+6]-ptw[i];//3-1
			VBC[i]=ptw[i+6]-ptw[i+3];//3-1
		}
		
	//Length of edge between control points
	float Rab,Rac,Rbc;
	Rab=sqrt(VAB[0]*VAB[0]+VAB[1]*VAB[1]+VAB[2]*VAB[2]);
	Rac=sqrt(VAC[0]*VAC[0]+VAC[1]*VAC[1]+VAC[2]*VAC[2]);
	Rbc=sqrt(VBC[0]*VBC[0]+VBC[1]*VBC[1]+VBC[2]*VBC[2]);//�᭱�Τ���
	
	float CA[3],CB[3],CC[3];
		CA[0]=(pti[0]-parm[0])/parm[2];	CA[1]=(pti[1]-parm[1])/parm[3];	CA[2]=1;
		CB[0]=(pti[2]-parm[0])/parm[2];	CB[1]=(pti[3]-parm[1])/parm[3];	CB[2]=1;
		CC[0]=(pti[4]-parm[0])/parm[2];	CC[1]=(pti[5]-parm[1])/parm[3];	CC[2]=1;
	
	float RCA,RCB,RCC;
		RCA=sqrt(CA[0]*CA[0]+CA[1]*CA[1]+CA[2]*CA[2]);
		RCB=sqrt(CB[0]*CB[0]+CB[1]*CB[1]+CB[2]*CB[2]);
		RCC=sqrt(CC[0]*CC[0]+CC[1]*CC[1]+CC[2]*CC[2]);
	
	//Normalize
	CA[0]=CA[0]/RCA;	CA[1]=CA[1]/RCA;	CA[2]=CA[2]/RCA;
	CB[0]=CB[0]/RCB;	CB[1]=CB[1]/RCB;	CB[2]=CB[2]/RCB;
	CC[0]=CC[0]/RCC;	CC[1]=CC[1]/RCC;	CC[2]=CC[2]/RCC;
	
	float Rab1,Rac1,Rbc1;//�V�q���׭p�� unit mm
		Rab1=sqrt(CA[0]*CA[0]+CA[1]*CA[1]+CA[2]*CA[2]);
		Rac1=sqrt(CB[0]*CB[0]+CB[1]*CB[1]+CB[2]*CB[2]);
		Rbc1=sqrt(CC[0]*CC[0]+CC[1]*CC[1]+CC[2]*CC[2]);
	
	//Cosine of angles//�᭱�S�Ψ�
	float Calb, Calc, Cblc;
	//Compute Calb Calc Cblc using Law of Cosine
	 Calb = (2 - Rab1*Rab1) / 2;
	Calc = (2 - Rac1*Rac1) / 2;
	Cblc = (2 - Rbc1*Rbc1) / 2; 
	
	//Get cosine of the angles
	float Clab = (aa*aa + Rab*Rab - bb*bb) / (2 * aa*Rab);
	float Clac = (aa*aa + Rac*Rac - cc*cc) / (2 * aa*Rac);
	
	//Get scale along norm vector
	float Raq = aa*Clab;
	float Rap = aa*Clac;
	
	//Get norm vector of plane P1 P2
	float VAB_norm = Rab;
	float VAC_norm = Rac;
	
	float WQ[3],WP[3];
	for (int i = 0; i<3; i++)
	{
		WQ[i]=ptw[i]+ Raq*VAB[i] / VAB_norm;
		WP[i]=ptw[i]+ Rap*VAC[i] / VAC_norm;
	}
	
	//Compute Plane P1 P2 P3
	float NP1[3],NP2[3],NP3[3];
	for (int i = 0; i<3; i++)
	{
		NP1[i]=VAB[i] / VAB_norm;
		NP2[i]=VAC[i] / VAC_norm;
	}
	
	float DP1, DP2, DP3;
	DP1=NP1[0]*WQ[0]+NP1[1]*WQ[1]+NP1[2]*WQ[2];
	DP2=NP2[0]*WP[0]+NP2[1]*WP[1]+NP2[2]*WP[2];
	
	float P1[4],P2[4],P3[4];
	P1[0]=NP1[0];	P1[1]=NP1[1];	P1[2]=NP1[2];	P1[3]=-DP1;
	P2[0]=NP2[0];	P2[1]=NP2[1];	P2[2]=NP2[2];	P2[3]=-DP2;
	
	float VCX[3],VCY[3],VCZ[3];
	for (int i = 0; i<3; i++)
	{
		VCX[i]=CB[i]-CA[i];
		VCY[i]=CC[i]-CA[i];
	}
	
	//�p����u�V�q�~�nZ��V�����q�P�_�ϥΦ�رƦC(�t��AC X AB)
	if ( (VCX[0] * VCY[1] - VCY[0] * VCX[1]) > 0 )	
	{
		//cvCrossProduct(VAC, VAB, NP3);
		NP3[0]=VAC[1]*VAB[2]-VAC[2]*VAB[1];
		NP3[1]=VAC[2]*VAB[0]-VAC[0]*VAB[2];
		NP3[2]=VAC[0]*VAB[1]-VAC[1]*VAB[0];
	}
	else
	{
		//cvCrossProduct(VAB, VAC, NP3);
		NP3[0]=VAB[1]*VAC[2]-VAB[2]*VAC[1];
		NP3[1]=VAB[2]*VAC[0]-VAB[0]*VAC[2];
		NP3[2]=VAB[0]*VAC[1]-VAB[1]*VAC[0];
	}
	
	float NP3_norm=sqrt(NP3[0]*NP3[0]+NP3[1]*NP3[1]+NP3[2]*NP3[2]);
	//Normalize
	NP3[0]=NP3[0]/NP3_norm;	NP3[1]=NP3[1]/NP3_norm;	NP3[2]=NP3[2]/NP3_norm;
	
	DP3=NP3[0]*ptw[0]+NP3[1]*ptw[1]+NP3[2]*ptw[2];//DP3 = cvDotProduct(NP3, W_one);
	
	P3[0]=NP3[0];	P3[1]=NP3[1];	P3[2]=NP3[2];	P3[3]=-DP3;
	
	//�J�԰������D�T������ P1 P2 P3
	float delta, delta_x, delta_y, delta_z;
	delta=P1[0]*P2[1]*P3[2]+P2[0]*P3[1]*P1[2]+P3[0]*P1[1]*P2[2]-P3[0]*P2[1]*P1[2]-P1[0]*P3[1]*P2[2]-P2[0]*P1[1]*P3[2];
	delta_x=P2[1]*P3[2]*P1[3]+P1[1]*P2[2]*P3[3]+P3[1]*P2[3]*P1[2]-P2[1]*P1[2]*P3[3]-P1[1]*P2[3]*P3[2]-P3[1]*P1[3]*P2[2];
	delta_y=P1[0]*P3[2]*P2[3]+P2[0]*P1[2]*P3[3]+P3[0]*P2[2]*P1[3]-P3[0]*P1[2]*P2[3]-P1[0]*P2[2]*P3[3]-P2[0]*P3[2]*P1[3];
	delta_z=P1[0]*P2[1]*P3[3]+P2[0]*P3[1]*P1[3]+P3[0]*P1[1]*P2[3]-P3[0]*P2[1]*P1[3]-P1[0]*P3[1]*P2[3]-P2[0]*P1[1]*P3[3];
	
	float WR[3];
	if(delta!=0)
	{
		//�W���Y�Ʋ����A�]���t�@�ӭt��
		WR[0]=-delta_x/delta;	WR[1]=-delta_y/delta;	WR[2]=-delta_z/delta;
	}
	else
	{
		WR[0]=1000000;	WR[1]=1000000;	WR[2]=1000000;
	}
	
	
	//Get length of LR
	float Rar, Rlr;
	Rar=sqrt((ptw[0]-WR[0])*(ptw[0]-WR[0])+(ptw[1]-WR[1])*(ptw[1]-WR[1])+(ptw[2]-WR[2])*(ptw[2]-WR[2]));
	if(aa*aa - Rar*Rar>0)
		Rlr = sqrt(aa*aa - Rar*Rar);//��aa*aa - Rar*Rar<0�ɡARlr�h��nan
	else
		Rlr = 10e6;
	//Get Position of L in world frame
	//WL=WR+NP3*Rlr;
	float WL_test[3];
	WL_test[0]=WR[0]+NP3[0]*Rlr;
	WL_test[1]=WR[1]+NP3[1]*Rlr;
	WL_test[2]=WR[2]+NP3[2]*Rlr;
	
	WL[blockIdx.x*3*blockDim.x+tid*3]=WL_test[0];
	WL[blockIdx.x*3*blockDim.x+tid*3+1]=WL_test[1];
	WL[blockIdx.x*3*blockDim.x+tid*3+2]=WL_test[2];
	
	///////////////////////////////////////�ا��Шt�����x�}/////////////////////////////////////////////
	VCZ[0]=VCX[1]*VCY[2]-VCX[2]*VCY[1];	//cvCrossProduct(VCX, VCY, VCZ);
	VCZ[1]=VCX[2]*VCY[0]-VCX[0]*VCY[2];
	VCZ[2]=VCX[0]*VCY[1]-VCX[1]*VCY[0];
	
	VCY[0]=VCZ[1]*VCX[2]-VCZ[2]*VCX[1];	//cvCrossProduct(VCZ, VCX, VCY);
	VCY[1]=VCZ[2]*VCX[0]-VCZ[0]*VCX[2];
	VCY[2]=VCZ[0]*VCX[1]-VCZ[1]*VCX[0];
	
	//Normalize
	float VCX_norm,VCY_norm,VCZ_norm;
	VCX_norm=sqrt(VCX[0]*VCX[0]+VCX[1]*VCX[1]+VCX[2]*VCX[2]);
	VCY_norm=sqrt(VCY[0]*VCY[0]+VCY[1]*VCY[1]+VCY[2]*VCY[2]);
	VCZ_norm=sqrt(VCZ[0]*VCZ[0]+VCZ[1]*VCZ[1]+VCZ[2]*VCZ[2]);
	for(int i=0;i<3;i++)
	{
		VCX[i]=VCX[i]/VCX_norm;	VCY[i]=VCY[i]/VCY_norm;	VCZ[i]=VCZ[i]/VCZ_norm;
	}
	
	//%Get ray in the world frame
	float Vla[3],Vlb[3],Vlc[3];
	for(int i=0;i<3;i++)
	{
		Vla[i]=ptw[i]-WL_test[i];	//Vla=WA-WL;
		Vlb[i]=ptw[i+3]-WL_test[i];	//Vlb=WB-WL;
		Vlc[i]=ptw[i+6]-WL_test[i];	//Vlc=WC-WL;
	}
	
	//Normalize
	float Vla_norm,Vlb_norm,Vlc_norm;
	Vla_norm=sqrt(Vla[0]*Vla[0]+Vla[1]*Vla[1]+Vla[2]*Vla[2]);
	Vlb_norm=sqrt(Vlb[0]*Vlb[0]+Vlb[1]*Vlb[1]+Vlb[2]*Vlb[2]);
	Vlc_norm=sqrt(Vlc[0]*Vlc[0]+Vlc[1]*Vlc[1]+Vlc[2]*Vlc[2]);
	for(int i=0;i<3;i++)
	{
		Vla[i]=Vla[i]/Vla_norm;	Vlb[i]=Vlb[i]/Vlb_norm;	Vlc[i]=Vlc[i]/Vlc_norm;
	}
	
	float WA1[3],WB1[3],WC1[3];
	for(int i=0;i<3;i++)
	{
		WA1[i]=WL_test[i]+Vla[i];	//WA1=WL+1*Vla;
		WB1[i]=WL_test[i]+Vlb[i];	//WB1=WL+1*Vlb;
		WC1[i]=WL_test[i]+Vlc[i];	//WC1=WL+1*Vlc;
	}
	
	float vcx[3],vcy[3],vcz[3];
	for(int i=0;i<3;i++)
	{
		vcx[i]=WB1[i]-WA1[i];	//vcx=WB1-WA1;
		vcy[i]=WC1[i]-WA1[i];	//vcy=WC1-WA1;
	}
	
	
	vcz[0]=vcx[1]*vcy[2]-vcx[2]*vcy[1];	//cvCrossProduct(vcx, vcy, vcz);
	vcz[1]=vcx[2]*vcy[0]-vcx[0]*vcy[2];
	vcz[2]=vcx[0]*vcy[1]-vcx[1]*vcy[0];
	
	vcy[0]=vcz[1]*vcx[2]-vcz[2]*vcx[1];	//cvCrossProduct(vcz, vcx, vcy);
	vcy[1]=vcz[2]*vcx[0]-vcz[0]*vcx[2];
	vcy[2]=vcz[0]*vcx[1]-vcz[1]*vcx[0];
	
	
	
	//Normalize
	//float vcx_norm_test,vcy_norm_test,vcz_norm_test;
	float vcx_norm,vcy_norm,vcz_norm;
	
	/* vcx_norm_test=vcx[0]*vcx[0]+vcx[1]*vcx[1]+vcx[2]*vcx[2];
	vcy_norm_test=vcy[0]*vcy[0]+vcy[1]*vcy[1]+vcy[2]*vcy[2];
	vcz_norm_test=vcz[0]*vcz[0]+vcz[1]*vcz[1]+vcz[2]*vcz[2];
	
	if(vcx_norm_test==0)
		vcx_norm=sqrt(10e-12);
	else	vcx_norm=sqrt(vcx_norm_test);
	
	if(vcy_norm_test==0)
		vcy_norm_test=sqrt(10e-12);
	else	vcy_norm=sqrt(vcy_norm_test);
	
	if(vcz_norm_test==0)
		vcz_norm_test=sqrt(10e-12);
	else	vcz_norm=sqrt(vcz_norm_test); */
	
	vcx_norm=sqrt(vcx[0]*vcx[0]+vcx[1]*vcx[1]+vcx[2]*vcx[2]);
	vcy_norm=sqrt(vcy[0]*vcy[0]+vcy[1]*vcy[1]+vcy[2]*vcy[2]);
	vcz_norm=sqrt(vcz[0]*vcz[0]+vcz[1]*vcz[1]+vcz[2]*vcz[2]);
	
	for(int i=0;i<3;i++)
	{
		vcx[i]=vcx[i]/vcx_norm;	vcy[i]=vcy[i]/vcy_norm;	vcz[i]=vcz[i]/vcz_norm;
	}
	
	//R=[VCx VCy VCz]*inv([vcx vcy vcz]);
	float R1[3][3],R2[3][3]/*,R3[3][3]*/;
	for(int i=0;i<3;i++)
	{
		R1[i][0]=VCX[i];	R1[i][1]=VCY[i];	R1[i][2]=VCZ[i];
		R2[0][i]=vcx[i];	R2[1][i]=vcy[i];	R2[2][i]=vcz[i];	//�]������ncvTranspose(R2, R2);�ҥH�񪺮ɭԪ���
	}
	
	//�x�}�ۭ�(3*3)R1*R2
	float Rext[3][3];
	for(int i=0;i<3;i++)
	{
		for(int j=0;j<3;j++)
		{
			float sum=0.0;
			for(int k=0;k<3;k++)
				sum+=R1[i][k]*R2[k][j];
			Rext[i][j]=sum;
		}
	}
		
	
	Rs[blockIdx.x*9*blockDim.x+tid*9]=Rext[0][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+1]=Rext[0][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+2]=Rext[0][2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+3]=Rext[1][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+4]=Rext[1][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+5]=Rext[1][2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+6]=Rext[2][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+7]=Rext[2][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+8]=Rext[2][2]; 
		
	//for (int k = 0; k< 3; k++)
	//{
	//	cvmSet(text, k, 0, -cvmGet(WL, k, 0));
	//}
	//cvMatMul(Rext, text, text);			
	/* for(int i=0;i<3;i++)		text�ثe�S�Ψ�
	{
		float sum=0.0;
		for(int k=0;k<3;k++)
			sum+=Rext[i][k]*(-WL_test[k]);
		text[i]=sum;
	} */

	}
	
	Ransac(mapft_info, WL ,Rs,parm, ransac_match_num);
}

__global__ void GPU_P3P_dub(int *matft_NO, int *mapft_NO, double *matft_info, double *mapft_info, int *P3P_NO, double *parm, double *WL ,double *Rs,int *ransac_match_num)
{
	int tid =threadIdx.x;
	//int blockIdx=blockIdx.x;
	int sol_num=parm[4];//u0,vo,fu,fv
	
	if(blockIdx.x*blockDim.x+tid<sol_num)
	{
	
	double ptw[9],pti[6],dep[3];//�C��thread���ۤv��
	int p1,p2,p3;
	
	p1=P3P_NO[blockIdx.x*3*blockDim.x+tid*3];//�C��thread���T�ӽs��
	p2=P3P_NO[blockIdx.x*3*blockDim.x+tid*3+1];
	p3=P3P_NO[blockIdx.x*3*blockDim.x+tid*3+2];
	
	ptw[0]=matft_info[p1*6];//�Ĥ@���I
	ptw[1]=matft_info[p1*6+1];
	ptw[2]=matft_info[p1*6+2];
	pti[0]=matft_info[p1*6+3];
	pti[1]=matft_info[p1*6+4];
	dep[0]=matft_info[p1*6+5]*matft_info[p1*6+5];
	
	ptw[3]=matft_info[p2*6];//�ĤG���I
	ptw[4]=matft_info[p2*6+1];
	ptw[5]=matft_info[p2*6+2];
	pti[2]=matft_info[p2*6+3];
	pti[3]=matft_info[p2*6+4];
	dep[1]=matft_info[p2*6+5]*matft_info[p2*6+5];
	
	ptw[6]=matft_info[p3*6];//�ĤT���I
	ptw[7]=matft_info[p3*6+1];
	ptw[8]=matft_info[p3*6+2];
	pti[4]=matft_info[p3*6+3];
	pti[5]=matft_info[p3*6+4];
	dep[2]=matft_info[p3*6+5]*matft_info[p3*6+5]; 
	
	 double aa,bb,cc;
	aa= sqrt( ((pti[0] - parm[0])/parm[2])*((pti[0] - parm[0])/parm[2])*dep[0] + ((pti[1]-parm[1])/parm[3])*((pti[1]-parm[1])/parm[3])*dep[0] + dep[0]);
	bb = sqrt( ((pti[2] - parm[0])/parm[2])*((pti[2] - parm[0])/parm[2])*dep[1] + ((pti[3]-parm[1])/parm[3])*((pti[3]-parm[1])/parm[3])*dep[1] + dep[1]);
	cc = sqrt( ((pti[4] - parm[0])/parm[2])*((pti[4] - parm[0])/parm[2])*dep[2] + ((pti[5]-parm[1])/parm[3])*((pti[5]-parm[1])/parm[3])*dep[2] + dep[2]);
	 
	/* Rs[blockIdx.x*9*blockDim.x+tid*9]=ptw[0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+1]=ptw[1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+2]=ptw[2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+3]=ptw[3];
	Rs[blockIdx.x*9*blockDim.x+tid*9+4]=ptw[4];
	Rs[blockIdx.x*9*blockDim.x+tid*9+5]=ptw[5];
	Rs[blockIdx.x*9*blockDim.x+tid*9+6]=ptw[6];
	Rs[blockIdx.x*9*blockDim.x+tid*9+7]=ptw[7];
	Rs[blockIdx.x*9*blockDim.x+tid*9+8]=ptw[8];  */
	
	
	//////////////////////////////////////P3P�p��///////////////////////////////////
	
	///////////////////////////////����v����m//////////////////////////
	double VAB[3],VAC[3],VBC[3];
		for(int i=0;i<3;i++)
		{
			VAB[i]=ptw[i+3]-ptw[i];//2-1
			VAC[i]=ptw[i+6]-ptw[i];//3-1
			VBC[i]=ptw[i+6]-ptw[i+3];//3-1
		}
		
	//Length of edge between control points
	double Rab,Rac,Rbc;
	Rab=sqrt(VAB[0]*VAB[0]+VAB[1]*VAB[1]+VAB[2]*VAB[2]);
	Rac=sqrt(VAC[0]*VAC[0]+VAC[1]*VAC[1]+VAC[2]*VAC[2]);
	Rbc=sqrt(VBC[0]*VBC[0]+VBC[1]*VBC[1]+VBC[2]*VBC[2]);//�᭱�Τ���
	
	double CA[3],CB[3],CC[3];
		CA[0]=(pti[0]-parm[0])/parm[2];	CA[1]=(pti[1]-parm[1])/parm[3];	CA[2]=1;
		CB[0]=(pti[2]-parm[0])/parm[2];	CB[1]=(pti[3]-parm[1])/parm[3];	CB[2]=1;
		CC[0]=(pti[4]-parm[0])/parm[2];	CC[1]=(pti[5]-parm[1])/parm[3];	CC[2]=1;
	
	double RCA,RCB,RCC;
		RCA=sqrt(CA[0]*CA[0]+CA[1]*CA[1]+CA[2]*CA[2]);
		RCB=sqrt(CB[0]*CB[0]+CB[1]*CB[1]+CB[2]*CB[2]);
		RCC=sqrt(CC[0]*CC[0]+CC[1]*CC[1]+CC[2]*CC[2]);
	
	//Normalize
	CA[0]=CA[0]/RCA;	CA[1]=CA[1]/RCA;	CA[2]=CA[2]/RCA;
	CB[0]=CB[0]/RCB;	CB[1]=CB[1]/RCB;	CB[2]=CB[2]/RCB;
	CC[0]=CC[0]/RCC;	CC[1]=CC[1]/RCC;	CC[2]=CC[2]/RCC;
	
		
	double Rab1,Rac1,Rbc1;//�V�q���׭p�� unit mm
		Rab1=sqrt(CA[0]*CA[0]+CA[1]*CA[1]+CA[2]*CA[2]);
		Rac1=sqrt(CB[0]*CB[0]+CB[1]*CB[1]+CB[2]*CB[2]);
		Rbc1=sqrt(CC[0]*CC[0]+CC[1]*CC[1]+CC[2]*CC[2]);
	
	
	//Cosine of angles//�᭱�S�Ψ�
	double Calb, Calc, Cblc;
	//Compute Calb Calc Cblc using Law of Cosine
	 Calb = (2 - Rab1*Rab1) / 2;
	Calc = (2 - Rac1*Rac1) / 2;
	Cblc = (2 - Rbc1*Rbc1) / 2; 
	
	//Get cosine of the angles
	double Clab = (aa*aa + Rab*Rab - bb*bb) / (2 * aa*Rab);
	double Clac = (aa*aa + Rac*Rac - cc*cc) / (2 * aa*Rac);
	
	//Get scale along norm vector
	double Raq = aa*Clab;
	double Rap = aa*Clac;
	
		
	//Get norm vector of plane P1 P2
	double VAB_norm = Rab;
	double VAC_norm = Rac;
	
	double WQ[3],WP[3];
	for (int i = 0; i<3; i++)
	{
		WQ[i]=ptw[i]+ Raq*VAB[i] / VAB_norm;
		WP[i]=ptw[i]+ Rap*VAC[i] / VAC_norm;
	}
	
	//Compute Plane P1 P2 P3
	double NP1[3],NP2[3],NP3[3];
	for (int i = 0; i<3; i++)
	{
		NP1[i]=VAB[i] / VAB_norm;
		NP2[i]=VAC[i] / VAC_norm;
	}
	
	double DP1, DP2, DP3;
	DP1=NP1[0]*WQ[0]+NP1[1]*WQ[1]+NP1[2]*WQ[2];
	DP2=NP2[0]*WP[0]+NP2[1]*WP[1]+NP2[2]*WP[2];
	
	double P1[4],P2[4],P3[4];
	P1[0]=NP1[0];	P1[1]=NP1[1];	P1[2]=NP1[2];	P1[3]=-DP1;
	P2[0]=NP2[0];	P2[1]=NP2[1];	P2[2]=NP2[2];	P2[3]=-DP2;
	
	double VCX[3],VCY[3],VCZ[3];
	for (int i = 0; i<3; i++)
	{
		VCX[i]=CB[i]-CA[i];
		VCY[i]=CC[i]-CA[i];
	}
	
	//�p����u�V�q�~�nZ��V�����q�P�_�ϥΦ�رƦC(�t��AC X AB)
	if ( (VCX[0] * VCY[1] - VCY[0] * VCX[1]) > 0 )	
	{
		//cvCrossProduct(VAC, VAB, NP3);
		NP3[0]=VAC[1]*VAB[2]-VAC[2]*VAB[1];
		NP3[1]=VAC[2]*VAB[0]-VAC[0]*VAB[2];
		NP3[2]=VAC[0]*VAB[1]-VAC[1]*VAB[0];
	}
	else
	{
		//cvCrossProduct(VAB, VAC, NP3);
		NP3[0]=VAB[1]*VAC[2]-VAB[2]*VAC[1];
		NP3[1]=VAB[2]*VAC[0]-VAB[0]*VAC[2];
		NP3[2]=VAB[0]*VAC[1]-VAB[1]*VAC[0];
	}
	
	double NP3_norm=sqrt(NP3[0]*NP3[0]+NP3[1]*NP3[1]+NP3[2]*NP3[2]);
	//Normalize
	NP3[0]=NP3[0]/NP3_norm;	NP3[1]=NP3[1]/NP3_norm;	NP3[2]=NP3[2]/NP3_norm;
	
	DP3=NP3[0]*ptw[0]+NP3[1]*ptw[1]+NP3[2]*ptw[2];//DP3 = cvDotProduct(NP3, W_one);
	
	P3[0]=NP3[0];	P3[1]=NP3[1];	P3[2]=NP3[2];	P3[3]=-DP3;
	
	//�J�԰������D�T������ P1 P2 P3
	double delta, delta_x, delta_y, delta_z;
	delta=P1[0]*P2[1]*P3[2]+P2[0]*P3[1]*P1[2]+P3[0]*P1[1]*P2[2]-P3[0]*P2[1]*P1[2]-P1[0]*P3[1]*P2[2]-P2[0]*P1[1]*P3[2];
	delta_x=P2[1]*P3[2]*P1[3]+P1[1]*P2[2]*P3[3]+P3[1]*P2[3]*P1[2]-P2[1]*P1[2]*P3[3]-P1[1]*P2[3]*P3[2]-P3[1]*P1[3]*P2[2];
	delta_y=P1[0]*P3[2]*P2[3]+P2[0]*P1[2]*P3[3]+P3[0]*P2[2]*P1[3]-P3[0]*P1[2]*P2[3]-P1[0]*P2[2]*P3[3]-P2[0]*P3[2]*P1[3];
	delta_z=P1[0]*P2[1]*P3[3]+P2[0]*P3[1]*P1[3]+P3[0]*P1[1]*P2[3]-P3[0]*P2[1]*P1[3]-P1[0]*P3[1]*P2[3]-P2[0]*P1[1]*P3[3];
	
	double WR[3];
	if(delta!=0)
	{
		//�W���Y�Ʋ����A�]���t�@�ӭt��
		WR[0]=-delta_x/delta;	WR[1]=-delta_y/delta;	WR[2]=-delta_z/delta;
	}
	else
	{
		WR[0]=1000000;	WR[1]=1000000;	WR[2]=1000000;
	}
	
	
	//Get length of LR
	double Rar, Rlr;
	Rar=sqrt((ptw[0]-WR[0])*(ptw[0]-WR[0])+(ptw[1]-WR[1])*(ptw[1]-WR[1])+(ptw[2]-WR[2])*(ptw[2]-WR[2]));
	if(aa*aa - Rar*Rar>0)
		Rlr = sqrt(aa*aa - Rar*Rar);//��aa*aa - Rar*Rar<0�ɡARlr�h��nan
	else
		Rlr = 10e6;
	//Get Position of L in world frame
	//WL=WR+NP3*Rlr;
	double WL_test[3];
	WL_test[0]=WR[0]+NP3[0]*Rlr;
	WL_test[1]=WR[1]+NP3[1]*Rlr;
	WL_test[2]=WR[2]+NP3[2]*Rlr;
	
	WL[blockIdx.x*3*blockDim.x+tid*3]=WL_test[0];
	WL[blockIdx.x*3*blockDim.x+tid*3+1]=WL_test[1];
	WL[blockIdx.x*3*blockDim.x+tid*3+2]=WL_test[2];
	
	///////////////////////////////////////�ا��Шt�����x�}/////////////////////////////////////////////
	VCZ[0]=VCX[1]*VCY[2]-VCX[2]*VCY[1];	//cvCrossProduct(VCX, VCY, VCZ);
	VCZ[1]=VCX[2]*VCY[0]-VCX[0]*VCY[2];
	VCZ[2]=VCX[0]*VCY[1]-VCX[1]*VCY[0];
	
	VCY[0]=VCZ[1]*VCX[2]-VCZ[2]*VCX[1];	//cvCrossProduct(VCZ, VCX, VCY);
	VCY[1]=VCZ[2]*VCX[0]-VCZ[0]*VCX[2];
	VCY[2]=VCZ[0]*VCX[1]-VCZ[1]*VCX[0];
	
	//Normalize
	double VCX_norm,VCY_norm,VCZ_norm;
	VCX_norm=sqrt(VCX[0]*VCX[0]+VCX[1]*VCX[1]+VCX[2]*VCX[2]);
	VCY_norm=sqrt(VCY[0]*VCY[0]+VCY[1]*VCY[1]+VCY[2]*VCY[2]);
	VCZ_norm=sqrt(VCZ[0]*VCZ[0]+VCZ[1]*VCZ[1]+VCZ[2]*VCZ[2]);
	for(int i=0;i<3;i++)
	{
		VCX[i]=VCX[i]/VCX_norm;	VCY[i]=VCY[i]/VCY_norm;	VCZ[i]=VCZ[i]/VCZ_norm;
	}
	
	//%Get ray in the world frame
	double Vla[3],Vlb[3],Vlc[3];
	for(int i=0;i<3;i++)
	{
		Vla[i]=ptw[i]-WL_test[i];	//Vla=WA-WL;
		Vlb[i]=ptw[i+3]-WL_test[i];	//Vlb=WB-WL;
		Vlc[i]=ptw[i+6]-WL_test[i];	//Vlc=WC-WL;
	}
	
	//Normalize
	double Vla_norm,Vlb_norm,Vlc_norm;
	Vla_norm=sqrt(Vla[0]*Vla[0]+Vla[1]*Vla[1]+Vla[2]*Vla[2]);
	Vlb_norm=sqrt(Vlb[0]*Vlb[0]+Vlb[1]*Vlb[1]+Vlb[2]*Vlb[2]);
	Vlc_norm=sqrt(Vlc[0]*Vlc[0]+Vlc[1]*Vlc[1]+Vlc[2]*Vlc[2]);
	for(int i=0;i<3;i++)
	{
		Vla[i]=Vla[i]/Vla_norm;	Vlb[i]=Vlb[i]/Vlb_norm;	Vlc[i]=Vlc[i]/Vlc_norm;
	}
	
	double WA1[3],WB1[3],WC1[3];
	for(int i=0;i<3;i++)
	{
		WA1[i]=WL_test[i]+Vla[i];	//WA1=WL+1*Vla;
		WB1[i]=WL_test[i]+Vlb[i];	//WB1=WL+1*Vlb;
		WC1[i]=WL_test[i]+Vlc[i];	//WC1=WL+1*Vlc;
	}
	
	double vcx[3],vcy[3],vcz[3];
	for(int i=0;i<3;i++)
	{
		vcx[i]=WB1[i]-WA1[i];	//vcx=WB1-WA1;
		vcy[i]=WC1[i]-WA1[i];	//vcy=WC1-WA1;
	}
	
	
	vcz[0]=vcx[1]*vcy[2]-vcx[2]*vcy[1];	//cvCrossProduct(vcx, vcy, vcz);
	vcz[1]=vcx[2]*vcy[0]-vcx[0]*vcy[2];
	vcz[2]=vcx[0]*vcy[1]-vcx[1]*vcy[0];
	
	vcy[0]=vcz[1]*vcx[2]-vcz[2]*vcx[1];	//cvCrossProduct(vcz, vcx, vcy);
	vcy[1]=vcz[2]*vcx[0]-vcz[0]*vcx[2];
	vcy[2]=vcz[0]*vcx[1]-vcz[1]*vcx[0];
	
	
	
	//Normalize
	//float vcx_norm_test,vcy_norm_test,vcz_norm_test;
	double vcx_norm,vcy_norm,vcz_norm;
	
	/* vcx_norm_test=vcx[0]*vcx[0]+vcx[1]*vcx[1]+vcx[2]*vcx[2];
	vcy_norm_test=vcy[0]*vcy[0]+vcy[1]*vcy[1]+vcy[2]*vcy[2];
	vcz_norm_test=vcz[0]*vcz[0]+vcz[1]*vcz[1]+vcz[2]*vcz[2];
	
	if(vcx_norm_test==0)
		vcx_norm=sqrt(10e-12);
	else	vcx_norm=sqrt(vcx_norm_test);
	
	if(vcy_norm_test==0)
		vcy_norm_test=sqrt(10e-12);
	else	vcy_norm=sqrt(vcy_norm_test);
	
	if(vcz_norm_test==0)
		vcz_norm_test=sqrt(10e-12);
	else	vcz_norm=sqrt(vcz_norm_test); */
	
	vcx_norm=sqrt(vcx[0]*vcx[0]+vcx[1]*vcx[1]+vcx[2]*vcx[2]);
	vcy_norm=sqrt(vcy[0]*vcy[0]+vcy[1]*vcy[1]+vcy[2]*vcy[2]);
	vcz_norm=sqrt(vcz[0]*vcz[0]+vcz[1]*vcz[1]+vcz[2]*vcz[2]);
	
	for(int i=0;i<3;i++)
	{
		vcx[i]=vcx[i]/vcx_norm;	vcy[i]=vcy[i]/vcy_norm;	vcz[i]=vcz[i]/vcz_norm;
	}
	
	//R=[VCx VCy VCz]*inv([vcx vcy vcz]);
	double R1[3][3],R2[3][3]/*,R3[3][3]*/;
	for(int i=0;i<3;i++)
	{
		R1[i][0]=VCX[i];	R1[i][1]=VCY[i];	R1[i][2]=VCZ[i];
		R2[0][i]=vcx[i];	R2[1][i]=vcy[i];	R2[2][i]=vcz[i];	//�]������ncvTranspose(R2, R2);�ҥH�񪺮ɭԪ���
	}
	
	//�x�}�ۭ�(3*3)R1*R2
	double Rext[3][3];
	for(int i=0;i<3;i++)
	{
		for(int j=0;j<3;j++)
		{
			double sum=0.0;
			for(int k=0;k<3;k++)
				sum+=R1[i][k]*R2[k][j];
			Rext[i][j]=sum;
		}
	}
		
	
	Rs[blockIdx.x*9*blockDim.x+tid*9]=Rext[0][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+1]=Rext[0][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+2]=Rext[0][2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+3]=Rext[1][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+4]=Rext[1][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+5]=Rext[1][2];
	Rs[blockIdx.x*9*blockDim.x+tid*9+6]=Rext[2][0];
	Rs[blockIdx.x*9*blockDim.x+tid*9+7]=Rext[2][1];
	Rs[blockIdx.x*9*blockDim.x+tid*9+8]=Rext[2][2]; 
		
	//for (int k = 0; k< 3; k++)
	//{
	//	cvmSet(text, k, 0, -cvmGet(WL, k, 0));
	//}
	//cvMatMul(Rext, text, text);			
	/* for(int i=0;i<3;i++)		text�ثe�S�Ψ�
	{
		float sum=0.0;
		for(int k=0;k<3;k++)
			sum+=Rext[i][k]*(-WL_test[k]);
		text[i]=sum;
	} */

	}
	
	Ransac_dub(mapft_info, WL ,Rs,parm, ransac_match_num);
}



void launchKernel(int map_num,int img_num,int descriptorDim,std::vector<int>&gpu_matcher,std::vector<feature>&feature_map,std::vector<feature>&feature_img,float* gpu_result,float* gpu_min,int* gpu_index,float* gpu_data_test)
{
		
	int blocknum = map_num;
	int thredperblock = NT;
	int couts = 0;
		
	couts=(img_num+thredperblock-1)/thredperblock;
    
    
    float *d_map, *d_img ;
	int  *l_map, *l_img;
	/////////////////// allocate memory on the cpu side//////////////////////
	d_map = (float*)malloc(map_num *descriptorDim* sizeof(float));
	l_map = (int*)malloc(map_num * sizeof(int));
	d_img = (float*)malloc(img_num *descriptorDim* sizeof(float));
	l_img = (int*)malloc(img_num * sizeof(int));
    ////////////////// put data in the host memory////////////////////////
    for (int i = 0; i < map_num; i++)
    {
        l_map[i]=feature_map[i].laplacian;
        for(int j=0;j<descriptorDim;j++)
		    d_map[i*descriptorDim+j]=feature_map[i].descriptor[j];
    }
        
	for (int i = 0; i < img_num; i++)
    {
        l_img[i]=feature_img[i].laplacian;
        for(int j=0;j<descriptorDim;j++)
		    d_img[i*descriptorDim+j]=feature_img[i].descriptor[j];
    }	
	////////////////// put data in the host memory////////////////////////
    
	float *dev_d_map, *dev_d_img;
	int *dev_l_map, *dev_l_img;
	float *dev_result;
	float *dev_min;
	int  *dev_index;
	float *dev_gpu_data_test;
	int *global_tem_index,*dev_tem_index;
	global_tem_index=(int*)malloc(map_num*NT/2* sizeof(int));
	////////////////// allocate the memory on the GPU ///////////////////////
	
	hipMalloc((void **)&dev_d_map, map_num *descriptorDim* sizeof(float));
	hipMalloc((void **)&dev_l_map, map_num * sizeof(int));
	hipMalloc((void **)&dev_d_img, img_num * descriptorDim*sizeof(float));
	hipMalloc((void **)&dev_l_img, img_num * sizeof(int));
	hipMalloc((void **)&dev_result, img_num*map_num * sizeof(float));
	//hipMalloc((void **)&dev_match, 2 * map_num * sizeof(int));
	hipMalloc((void **)&dev_min, couts*map_num * sizeof(float));
	hipMalloc((void **)&dev_index, couts * map_num * sizeof(int));
	hipMalloc((void **)&dev_gpu_data_test,map_num*img_num* sizeof(float));
	hipMalloc((void **)&dev_tem_index,map_num*NT/2* sizeof(int));
	////////////////// copy the arrays 's' and 'p' to the GPU/////////////////////////
	hipMemcpy(dev_d_map, d_map, map_num * descriptorDim*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_l_map, l_map, map_num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_d_img, d_img, img_num *descriptorDim* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_l_img, l_img, img_num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_result, gpu_result, img_num*map_num * sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(dev_match, match, 2 * map_num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_min, gpu_min, couts*map_num * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_index, gpu_index, couts*map_num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_gpu_data_test,gpu_data_test,map_num*img_num* sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_tem_index,global_tem_index,map_num*NT/2* sizeof(int),hipMemcpyHostToDevice);
	// Get start time event
	hipEvent_t start_kernel, stop_kernel;
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
	hipEventRecord(start_kernel, 0);

	matching << < blocknum, thredperblock >> > (dev_d_map, dev_l_map, dev_d_img, dev_l_img, dev_result, img_num, map_num,descriptorDim,dev_min, dev_index,dev_gpu_data_test,dev_tem_index);

	
	// Get stop time event
	hipEventRecord(stop_kernel, 0);
	hipEventSynchronize(stop_kernel);
	// Compute execution time
	float kernelTime;
	hipEventElapsedTime(&kernelTime, start_kernel, stop_kernel);
	printf("GPU time: %13f msec\n", kernelTime);
	hipEventDestroy(start_kernel);
	hipEventDestroy(stop_kernel);
	

	hipError_t cuda_err = hipGetLastError();
	if (hipSuccess != cuda_err) {
		cout << "before kernel call: error = %s\n" << hipGetErrorString(cuda_err) << endl;
		system("pause");
		exit(1);
	}

	hipEvent_t start_tra, stop_tra;
	hipEventCreate(&start_tra);
	hipEventCreate(&stop_tra);
	hipEventRecord(start_tra, 0);
	
	hipMemcpy(gpu_result, dev_result, img_num*map_num * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(gpu_min, dev_min, couts*map_num * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(gpu_index, dev_index, couts*map_num *sizeof(int), hipMemcpyDeviceToHost);
	
	hipEventRecord(stop_tra, 0);
	hipEventSynchronize(stop_tra);

	// Compute execution time
	float transferTime;
	hipEventElapsedTime(&transferTime, start_tra, stop_tra);
	printf("GPU transfer time: %13f msec\n", transferTime);
	hipEventDestroy(start_tra);
	hipEventDestroy(stop_tra);
	
	
		
	int min_couts=0;
	float tem_min=0;
	int tem_index=0;
	
	min_couts=(couts+thredperblock-1)/thredperblock;
	/* for(int j=0;j<map_num;j++)
		for(int i=0;i<min_couts;i++)
			cout<<gpu_min[j*couts+i]<<endl; */
	/* for(int j=0;j<12;j++)
			cout<<gpu_index[j]<<endl;
		cout<<endl; */
	/* for(int i=0;i<img_num;i++)
		cout<<gpu_data_test[(map_num-1)*img_num+i]<<endl;
	cout<<endl; */
	/* for(int j=0;j<couts;j++)
			cout<<gpu_min[couts*(map_num-2)+j]<<endl;
		cout<<endl; */
	
	/* for(int j=0;j<couts*map_num;j++)
		cout<<gpu_min[j]<<endl; */
		
	struct timeval startCPU, endCPU;
    gettimeofday(&startCPU, NULL);
	
	for(int i=0;i<map_num;i++)//////�^�Ǥ�̤p
	{
		tem_min=gpu_min[i*couts];tem_index=gpu_index[i*couts];
		for(int j=1;j<min_couts;j++)
		{
			if(tem_min>gpu_min[i*couts+j])
			{
				tem_min=gpu_min[i*couts+j];
				tem_index=gpu_index[i*couts+j];
			}	
		}
		gpu_min[i*couts]=tem_min;
		gpu_index[i*couts]=tem_index;
		if(gpu_min[i*couts]<0.09)//�p����e�ȧ�X���쪺
		{
			gpu_matcher.push_back(i);
			gpu_matcher.push_back(gpu_index[i*couts]);
		}
	}
	
	
		
	gettimeofday(&endCPU, NULL);
    double executime;
    executime = (endCPU.tv_sec - startCPU.tv_sec) * 1000.0;
    executime += (endCPU.tv_usec - startCPU.tv_usec) / 1000.0;
    printf("GPU_CPU find part min time: %13lf msec\n", executime);
	printf("All time: %13lf msec\n", executime+kernelTime+transferTime);
	/////////////////////////////////���զۤv�гy���Ⱥ�X�����G///////////////////////////////////////
	#ifdef GPU_Debug_Create
	int block_error=0,index_error=0;
	for(int i=0;i<map_num;i++)
	{
		 if(fabs(gpu_min[i*couts]-0.001)>0.00001)
		{
			//cout<<i<<setw(10)<<gpu_min[i*couts]<<endl;
			block_error++;
		} 
		//cout<<i<<setw(10)<<gpu_min[i]<<endl;
		//cout<<min[couts*i]<<endl;
		//cout<<index[i]<<endl;
	}
	for(int j=0;j<map_num;j++)
		if(gpu_index[j*couts]!=743)
		{
			//cout<<j<<setw(10)<<gpu_index[j*couts]<<endl;
			index_error++;
		}
			
	cout<<	"block_error"<<block_error<<endl;
	cout<<	"index_error"<<index_error<<endl;
	#endif
	/////////////////////////////////���զۤv�гy���Ⱥ�X�����G///////////////////////////////////////
    
	// Free device memory
	hipFree(dev_d_map);
	hipFree(dev_l_map);
	hipFree(dev_d_img);
	hipFree(dev_l_img);
	hipFree(dev_result);
	hipFree(dev_min);
	hipFree(dev_index);
	
	//return min;
	
}

void launchVOKernel(std::vector<keepfeature> matchfeature , std::vector<keepfeature> map_feature,std::vector<Ransac_Pos> &ransac_match,float *GPU_WL,float *GPU_Rs,int *ransac_match_num,double l_u0,double l_v0,double l_fu,double l_fv)
{
	int thredperblock = NT;
	int sol_num=(matchfeature.size()*(matchfeature.size()-1)*(matchfeature.size()-2))/6; //n��3
	int blocknum=(sol_num+thredperblock-1)/thredperblock;
	/////////////////// allocate memory on the cpu side//////////////////////
	int *matchfeature_NO,*map_feature_NO;//�s��
	float *matchfeature_info, *map_feature_info;
	int *P3P_NO;//��model���S�x�I�s��
	//float *WL, *Rs;
	float *parm;
	//int *ransac_match_num;//�ŦX�@�Ѷ��X���ƶq
	matchfeature_NO = (int*)malloc(matchfeature.size()* sizeof(int));
	map_feature_NO = (int*)malloc(map_feature.size()* sizeof(int));
	matchfeature_info = (float*)malloc(6 *matchfeature.size()* sizeof(float));//hx hy hz ix iy
	map_feature_info = (float*)malloc(6 *map_feature.size()* sizeof(float));//hx hy hz ix iy
	//WL=(float*)malloc(3 *sol_num* sizeof(float));//x y z
	//Rs=(float*)malloc(9 *sol_num* sizeof(float));//11 12 13 21 22 23 31 32 33
	P3P_NO=(int*)malloc(3 *sol_num* sizeof(int));
	parm=(float*)malloc(6 * sizeof(float));//l_u0,l_v0,l_fu,l_fv,map_feature.size()
	//ransac_match_num=(int*)malloc(sol_num* sizeof(int));
	////////////////// put data in the host memory////////////////////////
	int times=0;
	for(int g=0;g<matchfeature.size()-2;g++)
	{
		for(int g_two=g+1 ;  g_two< matchfeature.size()-1 ; g_two++ )
		{
			for(int g_three=g_two+1 ;  g_three< matchfeature.size() ; g_three++ )
			{
				P3P_NO[times]=g;
				P3P_NO[times+1]=g_two;
				P3P_NO[times+2]=g_three;
				times+=3;
			}
		}
	}
	/* for(int i=0;i<3*sol_num;i+=3)
	{
		cout<<P3P_NO[i]<<setw(5)<<P3P_NO[i+1]<<setw(5)<<P3P_NO[i+2]<<endl;
	} */
	parm[0]=l_u0;parm[1]=l_v0;parm[2]=l_fu;parm[3]=l_fv;parm[4]=sol_num;parm[5]=map_feature.size();
	
	for(int i=0;i<matchfeature.size();i++)
	{
		matchfeature_NO[i]=matchfeature[i].num;
		matchfeature_info[i*6]=matchfeature[i].hx;
		matchfeature_info[i*6+1]=matchfeature[i].hy;
		matchfeature_info[i*6+2]=matchfeature[i].hz;
		matchfeature_info[i*6+3]=matchfeature[i].l_ix;
		matchfeature_info[i*6+4]=matchfeature[i].l_iy;
		matchfeature_info[i*6+5]=matchfeature[i].depthvalue;
	}

	for(int j=0;j<map_feature.size();j++)
	{
		map_feature_NO[j]=map_feature[j].num;
		map_feature_info[j*6]=map_feature[j].hx;
		map_feature_info[j*6+1]=map_feature[j].hy;
		map_feature_info[j*6+2]=map_feature[j].hz;
		map_feature_info[j*6+3]=map_feature[j].l_ix;
		map_feature_info[j*6+4]=map_feature[j].l_iy;
		map_feature_info[j*6+5]=map_feature[j].depthvalue;
	}
		
	////////////////// allocate the memory on the GPU ///////////////////////
	
	int *dev_matchfeature_NO,*dev_map_feature_NO;//�s��
	float *dev_matchfeature_info, *dev_map_feature_info;
	float *dev_WL,*dev_Rs;
	int *dev_P3P_NO;//��model���S�x�I�s��
	float *dev_parm;
	int *dev_ransac_match_num;//�ŦX�@�Ѷ��X���ƶq
	hipMalloc((void **)&dev_matchfeature_NO, matchfeature.size()* sizeof(int));
	hipMalloc((void **)&dev_map_feature_NO, map_feature.size()* sizeof(int));
	hipMalloc((void **)&dev_matchfeature_info, 6 *matchfeature.size()* sizeof(float));
	hipMalloc((void **)&dev_map_feature_info, 6 *map_feature.size()* sizeof(float));
	hipMalloc((void **)&dev_WL, 3 *sol_num* sizeof(float));
	hipMalloc((void **)&dev_Rs, 9 *sol_num* sizeof(float));
	hipMalloc((void **)&dev_P3P_NO, 3 *sol_num* sizeof(int));
	hipMalloc((void **)&dev_parm, 6*sizeof(float));
	hipMalloc((void **)&dev_ransac_match_num, sol_num* sizeof(int));
	////////////////// copy the arrays to the GPU/////////////////////////
	hipMemcpy(dev_matchfeature_NO, matchfeature_NO, matchfeature.size()* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_map_feature_NO, map_feature_NO, map_feature.size()* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_matchfeature_info, matchfeature_info, 6 *matchfeature.size()* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_map_feature_info, map_feature_info, 6 *map_feature.size()* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_WL, GPU_WL, 3 *sol_num* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Rs, GPU_Rs, 9 *sol_num* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_P3P_NO, P3P_NO, 3 *sol_num* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_parm, parm, 6* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_ransac_match_num, ransac_match_num, sol_num* sizeof(int), hipMemcpyHostToDevice);
	
	cout<<"called launchVOKernel ok"<<endl;
	cout<<"blocknum"<<setw(10)<<blocknum<<endl;
	cout<<"thredperblock"<<setw(10)<<thredperblock<<endl;
	
	// Get start time event
	hipEvent_t start_kernel, stop_kernel;
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
	hipEventRecord(start_kernel, 0);
	
	GPU_P3P << < blocknum, thredperblock >> > (dev_matchfeature_NO,dev_map_feature_NO,dev_matchfeature_info,dev_map_feature_info,dev_P3P_NO,dev_parm,dev_WL,dev_Rs,dev_ransac_match_num);

	
	// Get stop time event
	hipEventRecord(stop_kernel, 0);
	hipEventSynchronize(stop_kernel);
	// Compute execution time
	float kernelTime;
	hipEventElapsedTime(&kernelTime, start_kernel, stop_kernel);
	printf("GPU float time: %13f msec\n", kernelTime);
	hipEventDestroy(start_kernel);
	hipEventDestroy(stop_kernel);
	
	
	hipError_t cuda_err = hipGetLastError();
	if (hipSuccess != cuda_err) {
		cout << "before kernel call: error = %s\n" << hipGetErrorString(cuda_err) << endl;
		system("pause");
		exit(1);
	}
	
	hipEvent_t start_tra, stop_tra;
	hipEventCreate(&start_tra);
	hipEventCreate(&stop_tra);
	hipEventRecord(start_tra, 0);
	
	hipMemcpy(GPU_WL, dev_WL, 3 *sol_num* sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(GPU_Rs, dev_Rs, 9 *sol_num* sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(ransac_match_num, dev_ransac_match_num, sol_num* sizeof(int), hipMemcpyDeviceToHost);
	
	hipEventRecord(stop_tra, 0);
	hipEventSynchronize(stop_tra);

	// Compute execution time
	float transferTime;
	hipEventElapsedTime(&transferTime, start_tra, stop_tra);
	printf("GPU transfer time: %13f msec\n", transferTime);
	cout<<"---------------------------------------------------"<<endl;
	hipEventDestroy(start_tra);
	hipEventDestroy(stop_tra);
	
	
	
	/* cout<<"-----------------------GPU_ransac_match--------------------------"<<endl;
	for(int i=0;i<sol_num;i++)
	{
		cout<<ransac_match_num[i]<<endl;
	}
	cout<<"-----------------------GPU_ransac_match--------------------------"<<endl;
	cout<<"sol num "<<setw(5)<<sol_num<<endl; */
	
	/* cout<<"-----------------------GPU sol camera pose----------------------------"<<endl; 
	for(int i=0;i<sol_num;i++)
	{
		cout<<GPU_WL[i*3]<<setw(15)<<GPU_WL[i*3+1]<<setw(15)<<GPU_WL[i*3+2]<<endl;
	}
	cout<<"-----------------------GPU sol rotation matrix----------------------------"<<endl; 
	for(int i=0;i<sol_num;i++)
	{
		cout<<Rs[i*9]<<setw(15)<<Rs[i*9+1]<<setw(15)<<Rs[i*9+2]<<endl;
		cout<<Rs[i*9+3]<<setw(15)<<Rs[i*9+4]<<setw(15)<<Rs[i*9+5]<<endl;
		cout<<Rs[i*9+6]<<setw(15)<<Rs[i*9+7]<<setw(15)<<Rs[i*9+8]<<endl;
	} */
	
	/* for(int i=4086;i<4095;i++)
		cout<<Rs[i]<<endl; */
	// Free device memory
	hipFree(dev_matchfeature_NO);
	hipFree(dev_map_feature_NO);
	hipFree(dev_matchfeature_info);
	hipFree(dev_map_feature_info);
	hipFree(dev_WL);
	hipFree(dev_Rs);
	hipFree(dev_parm);
	hipFree(dev_P3P_NO);
	hipFree(dev_ransac_match_num);
	
}

void launchVOKernel_dub(std::vector<keepfeature> matchfeature , std::vector<keepfeature> map_feature,std::vector<Ransac_Pos> &ransac_match,double *GPU_WL,double *GPU_Rs,int *ransac_match_num,double l_u0,double l_v0,double l_fu,double l_fv)
{
	int thredperblock = NT;
	int sol_num=(matchfeature.size()*(matchfeature.size()-1)*(matchfeature.size()-2))/6; //n��3
	int blocknum=(sol_num+thredperblock-1)/thredperblock;
	/////////////////// allocate memory on the cpu side//////////////////////
	int *matchfeature_NO,*map_feature_NO;//�s��
	double *matchfeature_info, *map_feature_info;
	int *P3P_NO;//��model���S�x�I�s��
	//float *WL, *Rs;
	double *parm;
	//int *ransac_match_num;//�ŦX�@�Ѷ��X���ƶq
	matchfeature_NO = (int*)malloc(matchfeature.size()* sizeof(int));
	map_feature_NO = (int*)malloc(map_feature.size()* sizeof(int));
	matchfeature_info = (double*)malloc(6 *matchfeature.size()* sizeof(double));//hx hy hz ix iy
	map_feature_info = (double*)malloc(6 *map_feature.size()* sizeof(double));//hx hy hz ix iy
	//WL=(float*)malloc(3 *sol_num* sizeof(float));//x y z
	//Rs=(float*)malloc(9 *sol_num* sizeof(float));//11 12 13 21 22 23 31 32 33
	P3P_NO=(int*)malloc(3 *sol_num* sizeof(int));
	parm=(double*)malloc(6 * sizeof(double));//l_u0,l_v0,l_fu,l_fv,map_feature.size()
	//ransac_match_num=(int*)malloc(sol_num* sizeof(int));
	////////////////// put data in the host memory////////////////////////
	int times=0;
	for(int g=0;g<matchfeature.size()-2;g++)
	{
		for(int g_two=g+1 ;  g_two< matchfeature.size()-1 ; g_two++ )
		{
			for(int g_three=g_two+1 ;  g_three< matchfeature.size() ; g_three++ )
			{
				P3P_NO[times]=g;
				P3P_NO[times+1]=g_two;
				P3P_NO[times+2]=g_three;
				times+=3;
			}
		}
	}
	/* for(int i=0;i<3*sol_num;i+=3)
	{
		cout<<P3P_NO[i]<<setw(5)<<P3P_NO[i+1]<<setw(5)<<P3P_NO[i+2]<<endl;
	} */
	parm[0]=l_u0;parm[1]=l_v0;parm[2]=l_fu;parm[3]=l_fv;parm[4]=sol_num;parm[5]=map_feature.size();
	
	for(int i=0;i<matchfeature.size();i++)
	{
		matchfeature_NO[i]=matchfeature[i].num;
		matchfeature_info[i*6]=matchfeature[i].hx;
		matchfeature_info[i*6+1]=matchfeature[i].hy;
		matchfeature_info[i*6+2]=matchfeature[i].hz;
		matchfeature_info[i*6+3]=matchfeature[i].l_ix;
		matchfeature_info[i*6+4]=matchfeature[i].l_iy;
		matchfeature_info[i*6+5]=matchfeature[i].depthvalue;
	}

	for(int j=0;j<map_feature.size();j++)
	{
		map_feature_NO[j]=map_feature[j].num;
		map_feature_info[j*6]=map_feature[j].hx;
		map_feature_info[j*6+1]=map_feature[j].hy;
		map_feature_info[j*6+2]=map_feature[j].hz;
		map_feature_info[j*6+3]=map_feature[j].l_ix;
		map_feature_info[j*6+4]=map_feature[j].l_iy;
		map_feature_info[j*6+5]=map_feature[j].depthvalue;
	}
		
	////////////////// allocate the memory on the GPU ///////////////////////
	
	int *dev_matchfeature_NO,*dev_map_feature_NO;//�s��
	double *dev_matchfeature_info, *dev_map_feature_info;
	double *dev_WL,*dev_Rs;
	int *dev_P3P_NO;//��model���S�x�I�s��
	double *dev_parm;
	int *dev_ransac_match_num;//�ŦX�@�Ѷ��X���ƶq
	hipMalloc((void **)&dev_matchfeature_NO, matchfeature.size()* sizeof(int));
	hipMalloc((void **)&dev_map_feature_NO, map_feature.size()* sizeof(int));
	hipMalloc((void **)&dev_matchfeature_info, 6 *matchfeature.size()* sizeof(double));
	hipMalloc((void **)&dev_map_feature_info, 6 *map_feature.size()* sizeof(double));
	hipMalloc((void **)&dev_WL, 3 *sol_num* sizeof(double));
	hipMalloc((void **)&dev_Rs, 9 *sol_num* sizeof(double));
	hipMalloc((void **)&dev_P3P_NO, 3 *sol_num* sizeof(int));
	hipMalloc((void **)&dev_parm, 6*sizeof(double));
	hipMalloc((void **)&dev_ransac_match_num, sol_num* sizeof(int));
	////////////////// copy the arrays to the GPU/////////////////////////
	hipMemcpy(dev_matchfeature_NO, matchfeature_NO, matchfeature.size()* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_map_feature_NO, map_feature_NO, map_feature.size()* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_matchfeature_info, matchfeature_info, 6 *matchfeature.size()* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_map_feature_info, map_feature_info, 6 *map_feature.size()* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_WL, GPU_WL, 3 *sol_num* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_Rs, GPU_Rs, 9 *sol_num* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_P3P_NO, P3P_NO, 3 *sol_num* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_parm, parm, 6* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_ransac_match_num, ransac_match_num, sol_num* sizeof(int), hipMemcpyHostToDevice);
	
		
	cout<<"called launchVOKernel_dub ok"<<endl;
	cout<<"blocknum"<<setw(10)<<blocknum<<endl;
	cout<<"thredperblock"<<setw(10)<<thredperblock<<endl;
	
	// Get start time event
	hipEvent_t start_kernel, stop_kernel;
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
	hipEventRecord(start_kernel, 0);
	
	GPU_P3P_dub << < blocknum, thredperblock >> > (dev_matchfeature_NO,dev_map_feature_NO,dev_matchfeature_info,dev_map_feature_info,dev_P3P_NO,dev_parm,dev_WL,dev_Rs,dev_ransac_match_num);

	
	// Get stop time event
	hipEventRecord(stop_kernel, 0);
	hipEventSynchronize(stop_kernel);
	// Compute execution time
	float kernelTime;
	hipEventElapsedTime(&kernelTime, start_kernel, stop_kernel);
	printf("GPU double time: %13f msec\n", kernelTime);
	hipEventDestroy(start_kernel);
	hipEventDestroy(stop_kernel);
	
	
	hipError_t cuda_err = hipGetLastError();
	if (hipSuccess != cuda_err) {
		cout << "before kernel call: error = %s\n" << hipGetErrorString(cuda_err) << endl;
		system("pause");
		exit(1);
	}
	
	hipEvent_t start_tra, stop_tra;
	hipEventCreate(&start_tra);
	hipEventCreate(&stop_tra);
	hipEventRecord(start_tra, 0);
	
	hipMemcpy(GPU_WL, dev_WL, 3 *sol_num* sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(GPU_Rs, dev_Rs, 9 *sol_num* sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(ransac_match_num, dev_ransac_match_num, sol_num* sizeof(int), hipMemcpyDeviceToHost);
	
	hipEventRecord(stop_tra, 0);
	hipEventSynchronize(stop_tra);

	// Compute execution time
	float transferTime;
	hipEventElapsedTime(&transferTime, start_tra, stop_tra);
	printf("GPU transfer time: %13f msec\n", transferTime);
	cout<<"---------------------------------------------------"<<endl;
	hipEventDestroy(start_tra);
	hipEventDestroy(stop_tra);
	
	
		
	/* cout<<"-----------------------GPU_ransac_match--------------------------"<<endl;
	for(int i=0;i<sol_num;i++)
	{
		cout<<ransac_match_num[i]<<endl;
	}
	cout<<"-----------------------GPU_ransac_match--------------------------"<<endl;
	cout<<"sol num "<<setw(5)<<sol_num<<endl; */
	
	/* cout<<"-----------------------GPU sol camera pose----------------------------"<<endl; 
	for(int i=0;i<sol_num;i++)
	{
		cout<<GPU_WL[i*3]<<setw(15)<<GPU_WL[i*3+1]<<setw(15)<<GPU_WL[i*3+2]<<endl;
	}
	cout<<"-----------------------GPU sol rotation matrix----------------------------"<<endl;  */
	/* for(int i=0;i<sol_num;i++)
	{
		cout<<Rs[i*9]<<setw(15)<<Rs[i*9+1]<<setw(15)<<Rs[i*9+2]<<endl;
		cout<<Rs[i*9+3]<<setw(15)<<Rs[i*9+4]<<setw(15)<<Rs[i*9+5]<<endl;
		cout<<Rs[i*9+6]<<setw(15)<<Rs[i*9+7]<<setw(15)<<Rs[i*9+8]<<endl;
	} */
	
	/* for(int i=4086;i<4095;i++)
		cout<<Rs[i]<<endl; */
	// Free device memory
	hipFree(dev_matchfeature_NO);
	hipFree(dev_map_feature_NO);
	hipFree(dev_matchfeature_info);
	hipFree(dev_map_feature_info);
	hipFree(dev_WL);
	hipFree(dev_Rs);
	hipFree(dev_parm);
	hipFree(dev_P3P_NO);
	hipFree(dev_ransac_match_num);
	
}


