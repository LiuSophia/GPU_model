#include "hip/hip_runtime.h"

#include "match.cuh"
#include <vector>
#include <stdio.h>
#include <stdlib.h>
using namespace std;



__device__ void findmin_global(float *data, int num, float* min, int* index,int* tem_index)
{
	int tid = threadIdx.x;
	int blockid = blockIdx.x;
	int couts =0;//一個block需要做的次數(從所有data)
	int min_couts=0;//一個block需要做的次數(從min)
	couts=(num+blockDim.x-1)/blockDim.x;
	int k=0,m=0;
	//__shared__ int tem_index[NT_match/2];//global
	
		while (k<couts)//判斷k<couts
		{
			if (num - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;
				m=0;
				while (i != 0)
				{
					if (tid < i) {
						if (data[blockid * num + k*blockDim.x + tid] > data[blockid * num + k*blockDim.x + tid + i])
						{
							data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid + i];
							if(m==0)	tem_index[blockid*NT_match/2+tid] = k*blockDim.x + tid + i;
							else	tem_index[blockid*NT_match/2+tid] = tem_index[blockid*NT_match/2+tid+i];
						}
						else
						{
							if(m==0)	tem_index[blockid*NT_match/2+tid] = k*blockDim.x + tid;
						}	
					}
					__syncthreads();
					i /= 2;
					m++;
				}
			}
			else//處理多餘
			{
				int remain = num - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				m=0;
				while (j != 0)
				{
					if (tid < j) 
					{
						
						if (data[blockid * num + k*blockDim.x + tid] > data[blockid * num + k*blockDim.x + tid + j])
						{
							data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid + j];
							/* if(m==0)	tem_index[tid] = k*blockDim.x + tid + j;
							else	tem_index[tid] = tem_index[tid+j]; */
							if(m==0)	tem_index[blockid*NT_match/2+tid] = k*blockDim.x + tid + j;
							else	tem_index[blockid*NT_match/2+tid] = tem_index[blockid*NT_match/2+tid+j];
						}
						else
						{
							if(m==0)	tem_index[blockid*NT_match/2+tid] = k*blockDim.x + tid;
						}
						if (tid + j * 2 < tem_j) {
							if (data[blockid * num + k*blockDim.x + tid + j * 2] < data[blockid * num + k*blockDim.x + tid])
							{
								data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid + j * 2];
								/* if(m==0)	tem_index[tid] = k*blockDim.x + remain - 1;
								else	tem_index[tid] = tem_index[tid+j*2]; */
								if(m==0)	tem_index[blockid*NT_match/2+tid] = k*blockDim.x + remain - 1;
								else	tem_index[blockid*NT_match/2+tid] = tem_index[blockid*NT_match/2+tid+j*2];
							}
						}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					m++;
				}
				if(remain==1)//如果剩一個
				{
					if (tid < remain)
					{
						data[blockid * num + k*blockDim.x + tid] = data[blockid * num + k*blockDim.x + tid ];
						tem_index[blockid*NT_match/2+tid] = k*blockDim.x + tid;
					}
				}
			}
			if(tid==0)
			{
				min[blockid * couts + k] = data[blockid * num + k*blockDim.x];
				index[blockid * couts + k] = tem_index[blockid*NT_match/2];
			}
			
			k++;
		}
		
		min_couts=(couts+blockDim.x-1)/blockDim.x;//3
		k=0;
		
		while(k<min_couts)
		{
			if(couts - blockDim.x*k >= blockDim.x)
			{
				int i = blockDim.x / 2;

				while (i != 0)
				{
					if (tid < i) {
							if (min[blockid*couts +k*blockDim.x+tid+ i ] < min[blockid*couts +k*blockDim.x+tid])
							{
								min[blockid*couts +k*blockDim.x+tid] = min[blockid*couts +k*blockDim.x+tid+ i ];
								index[blockid*couts +k*blockDim.x+tid]= index[blockid*couts +k*blockDim.x+tid + i];
							}
					}
					__syncthreads();
					i /= 2;
					//m++;
				}
			}
			else
			{
				int remain = couts - blockDim.x*k;
				int tem_j = remain;
				int j = remain / 2;
				
				while (j != 0)
				{
					if (tid < j) 
					{
							if (min[blockid*couts +k*blockDim.x+tid+j] < min[blockid*couts +k*blockDim.x+tid])
							{
								min[blockid*couts +k*blockDim.x+tid] = min[blockid*couts +k*blockDim.x+tid + j];
								index[blockid*couts +k*blockDim.x+tid] = index[blockid*couts +k*blockDim.x+tid + j];
							}
							if (tid + j * 2 < tem_j) {
								if (min[blockid*couts +k*blockDim.x+tid + j * 2] < min[blockid*couts +k*blockDim.x+tid])
								{
									min[blockid*couts +k*blockDim.x+tid] = min[blockid*couts +k*blockDim.x+tid + j * 2];
									index[blockid*couts +k*blockDim.x+tid] = index[blockid*couts +k*blockDim.x+tid + j * 2];
								}
							}
					}
					__syncthreads();
					tem_j /= 2;
					j /= 2;
					//m++;
				}
				if(remain==1)//如果剩一個
				{
					if (tid < remain)
					{
						min[blockid*couts +k*blockDim.x+tid] = min[blockid * couts + k*blockDim.x + tid ];
						index[blockid*couts +k*blockDim.x+tid] = index[blockid*couts + k*blockDim.x + tid];
					}
				}
				
			}
			
			if(tid==0)
			{
				min[blockid * couts + k] = min[blockid*couts +k*blockDim.x];
				index[blockid * couts + k] = index[blockid*couts +k*blockDim.x];
			}
			k++;

		}
		
}


__global__ void matchingGPU(float *d_map, int *l_map, float *d_img, int *l_img, float *result, int img_num,int map_num,int descriptorDim,float *min, int *index,int *tem_index)
{
	int tid = threadIdx.x;
	int temp_tid = 0;
	float SAD = 0.0;
	__shared__ float sum[NT_match] ;
	
		for (int j = 0; j < img_num; j++)
		{
			temp_tid=tid;
			SAD = 0.0;
			if (l_map[blockIdx.x] == l_img[j])
			{
				 for (int i = 0; i < descriptorDim/NT_match; i++)
				{ 
				//SAD = (d_map[blockIdx.x *descriptorDim+ tid] - d_img[j*descriptorDim + tid])*(d_map[blockIdx.x *descriptorDim+ tid] - d_img[j*descriptorDim + tid]);
					 SAD += (d_map[blockIdx.x *descriptorDim+ temp_tid] - d_img[j*descriptorDim + temp_tid])*(d_map[blockIdx.x *descriptorDim+ temp_tid] - d_img[j*descriptorDim + temp_tid]);
					temp_tid += blockDim.x;
				} 
				sum[tid] = SAD;
				__syncthreads();

				int i = blockDim.x / 2;
				while (i != 0)
				{
					if (tid < i) {
						sum[tid] += sum[tid + i];
					}
					__syncthreads();
					i /= 2;
				}
				
				if(tid==0)
					sum[0] = sqrt(sum[0]);
				__syncthreads();
				
				result[img_num*blockIdx.x + j] = sum[0];
			}
			else
			{
				result[img_num*blockIdx.x + j] = 10.0;
			}

		}

	findmin_global(result, img_num, min, index,tem_index);//找最小 global memory
	
		

}


void launchMatchKernel(int map_num,int img_num,int descriptorDim,std::vector<Keep_Feature>&feature_map,std::vector<PairFeature>&feature_img,std::vector<int>&gpu_matcher)
{
		
	int blocknum = map_num;
	int thredperblock = NT_match;
	int couts = 0;
		
	couts=(img_num+thredperblock-1)/thredperblock;
    
    
    float *d_map, *d_img ;
	int  *l_map, *l_img;
	float *gpu_result;
	float *gpu_min;
	int *gpu_index;int *global_tem_index;
	/////////////////// allocate memory on the cpu side//////////////////////
	d_map = (float*)malloc(map_num *descriptorDim* sizeof(float));
	l_map = (int*)malloc(map_num * sizeof(int));
	d_img = (float*)malloc(img_num *descriptorDim* sizeof(float));
	l_img = (int*)malloc(img_num * sizeof(int));
	gpu_result=(float*)malloc(map_num*img_num*sizeof(float));
	gpu_min=(float*)malloc(map_num * couts*sizeof(float));
	gpu_index=(int*)malloc(map_num * couts*sizeof(int));
	global_tem_index=(int*)malloc(map_num*NT_match/2* sizeof(int));
    ////////////////// put data in the host memory////////////////////////
    for (int i = 0; i < map_num; i++)
    {
        l_map[i]=feature_img[i].laplacian;
        for(int j=0;j<descriptorDim;j++)
		    d_map[i*descriptorDim+j]=feature_img[i].descriptor[j];
    }
        
	for (int i = 0; i < img_num; i++)
    {
        l_img[i]=feature_map[i].laplacian;
        for(int j=0;j<descriptorDim;j++)
		    d_img[i*descriptorDim+j]=feature_map[i].original_descriptor[j];
    }	
	////////////////// put data in the host memory////////////////////////
    
	float *dev_d_map, *dev_d_img;
	int *dev_l_map, *dev_l_img;
	float *dev_result;
	float *dev_min;
	int  *dev_index;
	//float *dev_gpu_data_test;
	int *dev_tem_index;
	////////////////// allocate the memory on the GPU ///////////////////////
	
	hipMalloc((void **)&dev_d_map, map_num *descriptorDim* sizeof(float));
	hipMalloc((void **)&dev_l_map, map_num * sizeof(int));
	hipMalloc((void **)&dev_d_img, img_num * descriptorDim*sizeof(float));
	hipMalloc((void **)&dev_l_img, img_num * sizeof(int));
	hipMalloc((void **)&dev_result, img_num*map_num * sizeof(float));
	//hipMalloc((void **)&dev_match, 2 * map_num * sizeof(int));
	hipMalloc((void **)&dev_min, couts*map_num * sizeof(float));
	hipMalloc((void **)&dev_index, couts * map_num * sizeof(int));
	//hipMalloc((void **)&dev_gpu_data_test,map_num*img_num* sizeof(float));
	hipMalloc((void **)&dev_tem_index,map_num*NT_match/2* sizeof(int));
	////////////////// copy the arrays 's' and 'p' to the GPU/////////////////////////
	hipMemcpy(dev_d_map, d_map, map_num * descriptorDim*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_l_map, l_map, map_num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_d_img, d_img, img_num *descriptorDim* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_l_img, l_img, img_num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_result, gpu_result, img_num*map_num * sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(dev_match, match, 2 * map_num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_min, gpu_min, couts*map_num * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_index, gpu_index, couts*map_num * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(dev_gpu_data_test,gpu_data_test,map_num*img_num* sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_tem_index,global_tem_index,map_num*NT_match/2* sizeof(int),hipMemcpyHostToDevice);
	// Get start time event
	hipEvent_t start_kernel, stop_kernel;
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
	hipEventRecord(start_kernel, 0);

	matchingGPU << < blocknum, thredperblock >> > (dev_d_map, dev_l_map, dev_d_img, dev_l_img, dev_result, img_num, map_num,descriptorDim,dev_min, dev_index,dev_tem_index);

	
	// Get stop time event
	hipEventRecord(stop_kernel, 0);
	hipEventSynchronize(stop_kernel);
	// Compute execution time
	float kernelTime;
	hipEventElapsedTime(&kernelTime, start_kernel, stop_kernel);
	printf("GPU time: %13f msec\n", kernelTime);
	hipEventDestroy(start_kernel);
	hipEventDestroy(stop_kernel);
	

	hipError_t cuda_err = hipGetLastError();
	if (hipSuccess != cuda_err) {
		cout << "before kernel call: error = %s\n" << hipGetErrorString(cuda_err) << endl;
		system("pause");
		exit(1);
	}

	hipEvent_t start_tra, stop_tra;
	hipEventCreate(&start_tra);
	hipEventCreate(&stop_tra);
	hipEventRecord(start_tra, 0);
	
	//hipMemcpy(gpu_result, dev_result, img_num*map_num * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(gpu_min, dev_min, couts*map_num * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(gpu_index, dev_index, couts*map_num *sizeof(int), hipMemcpyDeviceToHost);
	
	hipEventRecord(stop_tra, 0);
	hipEventSynchronize(stop_tra);

	// Compute execution time
	float transferTime;
	hipEventElapsedTime(&transferTime, start_tra, stop_tra);
	printf("GPU transfer time: %13f msec\n", transferTime);
	hipEventDestroy(start_tra);
	hipEventDestroy(stop_tra);
	
	
		
	int min_couts=0;
	float tem_min=0;
	int tem_index=0;
	
	min_couts=(couts+thredperblock-1)/thredperblock;
	

		
	struct timeval startCPU, endCPU;
    gettimeofday(&startCPU, NULL);
	
	for(int i=0;i<map_num;i++)//////回傳比最小
	{
		tem_min=gpu_min[i*couts];tem_index=gpu_index[i*couts];
		for(int j=1;j<min_couts;j++)
		{
			if(tem_min>gpu_min[i*couts+j])
			{
				tem_min=gpu_min[i*couts+j];
				tem_index=gpu_index[i*couts+j];
			}	
		}
		gpu_min[i*couts]=tem_min;
		gpu_index[i*couts]=tem_index;
		if(gpu_min[i*couts]<0.09)//小於門檻值找出比對到的
		{
			gpu_matcher.push_back(i);
			gpu_matcher.push_back(gpu_index[i*couts]);
		}
	}
	
	
		
	gettimeofday(&endCPU, NULL);
    double executime;
    executime = (endCPU.tv_sec - startCPU.tv_sec) * 1000.0;
    executime += (endCPU.tv_usec - startCPU.tv_usec) / 1000.0;
    printf("GPU_CPU find part min time: %13lf msec\n", executime);
	printf("All time: %13lf msec\n", executime+kernelTime+transferTime);
	/////////////////////////////////測試自己創造的值算出的結果///////////////////////////////////////
	#ifdef GPU_Debug_Create
	int block_error=0,index_error=0;
	for(int i=0;i<map_num;i++)
	{
		 if(fabs(gpu_min[i*couts]-0.001)>0.00001)
		{
			//cout<<i<<setw(10)<<gpu_min[i*couts]<<endl;
			block_error++;
		} 
		//cout<<i<<setw(10)<<gpu_min[i]<<endl;
		//cout<<min[couts*i]<<endl;
		//cout<<index[i]<<endl;
	}
	for(int j=0;j<map_num;j++)
		if(gpu_index[j*couts]!=743)
		{
			//cout<<j<<setw(10)<<gpu_index[j*couts]<<endl;
			index_error++;
		}
			
	cout<<	"block_error"<<block_error<<endl;
	cout<<	"index_error"<<index_error<<endl;
	#endif
	/////////////////////////////////測試自己創造的值算出的結果///////////////////////////////////////
    
	// Free device memory
	hipFree(dev_d_map);
	hipFree(dev_l_map);
	hipFree(dev_d_img);
	hipFree(dev_l_img);
	hipFree(dev_result);
	hipFree(dev_min);
	hipFree(dev_index);
	hipFree(dev_tem_index);
	
}


